#include "../NativeBlas.h"
#include <hipblas.h>



hipblasStatus_t convertStatus(int status) {
    switch (status) {
        case 0:
            return HIPBLAS_STATUS_SUCCESS;
        case 1:
            return HIPBLAS_STATUS_NOT_INITIALIZED;
        case 3:
            return HIPBLAS_STATUS_ALLOC_FAILED;
        case 7:
            return HIPBLAS_STATUS_INVALID_VALUE;
        case 8:
            return HIPBLAS_STATUS_ARCH_MISMATCH;
        case 11:
            return HIPBLAS_STATUS_MAPPING_ERROR;
        case 13:
            return HIPBLAS_STATUS_EXECUTION_FAILED;
        case 14:
            return HIPBLAS_STATUS_INTERNAL_ERROR;
        case 15:
            return HIPBLAS_STATUS_NOT_SUPPORTED;
        case 16:
            return HIPBLAS_STATUS_UNKNOWN;
        default:
            return HIPBLAS_STATUS_SUCCESS;
    }
}

hipblasFillMode_t convertUplo(int fillMode) {
    switch (fillMode) {
        case 0:
            return HIPBLAS_FILL_MODE_LOWER;
        case 1:
            return HIPBLAS_FILL_MODE_UPPER;
        default:
            return HIPBLAS_FILL_MODE_LOWER;
    }
}

hipblasDiagType_t convertDiag(int diag) {
    switch (diag) {
        case 0:
            return HIPBLAS_DIAG_NON_UNIT;
        case 1:
            return HIPBLAS_DIAG_UNIT;
        default:
            return HIPBLAS_DIAG_NON_UNIT;
    }
}

hipblasOperation_t convertTranspose(int op) {
    switch(op) {
        case 0: return HIPBLAS_OP_N;
        case 1: return HIPBLAS_OP_T;
        case 2: return HIPBLAS_OP_C;
        default: return HIPBLAS_OP_N;
    }
}

hipblasPointerMode_t convertPointerMode(int pointerMode) {
    switch(pointerMode) {
        case 0: return HIPBLAS_POINTER_MODE_HOST;
        case 1: return HIPBLAS_POINTER_MODE_DEVICE;
        default: return HIPBLAS_POINTER_MODE_HOST;
    }}

hipblasSideMode_t convertSideMode(int sideMode) {
    switch(sideMode) {
        case 0: return HIPBLAS_SIDE_LEFT;
        case 1: return HIPBLAS_SIDE_RIGHT;
        default: return HIPBLAS_SIDE_LEFT;
    }
}


/*
 * ======================================================
 * Level 1 BLAS functions
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * DOT
 * ------------------------------------------------------
 */

float Nd4jBlas::sdsdot(long *extraParams, int N, float alpha,
                       long X, int incX,
                       long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    //nothing found?
    return 0.0f;
}

double Nd4jBlas::dsdot(long *extraParams, int N,
                       long X, int incX,
                       long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    //nothing found?
    return 0.0f;
}

double Nd4jBlas::ddot(long *extraParams, int N,
                      long X, int incX,
                      long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasDdot(*handle, N, xPointer, incX, yPointer, incY, resultPointer);
    return 0.0;
}

float Nd4jBlas::sdot(long *extraParams, int N,
                     long X, int incX,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasSdot(*handle, N, xPointer, incX, yPointer, incY, resultPointer);
    return 0.0f;
}

/*
 * ------------------------------------------------------
 * NRM2
 * ------------------------------------------------------
 */

float Nd4jBlas::snrm2(long *extraParams, int N, long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasSnrm2(*handle, N, xPointer, incX, resultPointer);
    return 0.0f;


}

double Nd4jBlas::dnrm2(long *extraParams, int N, long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasDnrm2(*handle, N, xPointer, incX, resultPointer);
    return 0.0;
}

/*
 * ------------------------------------------------------
 * ASUM
 * ------------------------------------------------------
 */

float Nd4jBlas::sasum(long *extraParams, int N, long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasSasum(*handle, N, xPointer, incX, resultPointer);
    return 0.0f;


}

double Nd4jBlas::dasum(long *extraParams, int N, long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasDasum(*handle, N, xPointer, incX, resultPointer);
    return 0.0;

}

/*
 * ------------------------------------------------------
 * IAMAX
 * ------------------------------------------------------
 */

int Nd4jBlas::isamax(long *extraParams, int N, long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    int *resultPointer = reinterpret_cast<int *>(extraParams[1]);
    hipblasIsamax(*handle,N,xPointer,incX,resultPointer);
    return 0;

}

int Nd4jBlas::idamax(long *extraParams, int N, long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    int *resultPointer = reinterpret_cast<int *>(extraParams[1]);
    hipblasIdamax(*handle, N, xPointer, incX, resultPointer);
    return 0;


}

/*
 * ======================================================
 * Level 1 BLAS procedures
 * ======================================================
 */

/*
 * ------------------------------------------------------
 * ROT
 * ------------------------------------------------------
 */

void Nd4jBlas::srot(long *extraParams, int N,
                    long X, int incX,
                    long Y, int incY,
                    float c, float s) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSrot(*handle, N, xPointer, incX, yPointer, incY, &c, &s);
}

void Nd4jBlas::drot(long *extraParams, int N,
                    long X, int incX,
                    long Y, int incY,
                    double c, double s) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDrot(*handle, N, xPointer, incX, yPointer, incY, &c, &s);
}

/*
 * ------------------------------------------------------
 * ROTG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotg(long *extraParams, long args) {
    float *argsPointers = reinterpret_cast<float *>(args);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSrotg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3]);
}

void Nd4jBlas::drotg(long *extraParams, long args) {
    double *argsPointers = reinterpret_cast<double *>(args);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDrotg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3]);

}

/*
 * ------------------------------------------------------
 * ROTMG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotmg(long *extraParams, long args,
                      long P) {
    float *argsPointers = reinterpret_cast<float *>(args);
    float *pPointers = reinterpret_cast<float *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSrotmg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3], pPointers);

}

void Nd4jBlas::drotmg(long *extraParams, long args,
                      long P) {
    double *argsPointers = reinterpret_cast<double *>(args);
    double *pPointers = reinterpret_cast<double *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDrotmg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3], pPointers);

}

/*
 * ------------------------------------------------------
 * ROTM
 * ------------------------------------------------------
 */

void Nd4jBlas::srotm(long *extraParams, int N,
                     long X, int incX,
                     long Y, int incY,
                     long P) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    float *pPointer = reinterpret_cast<float *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSrotm(*handle, N, xPointer, incX, yPointer, incY, pPointer);

}

void Nd4jBlas::drotm(long *extraParams, int N,
                     long X, int incX,
                     long Y, int incY,
                     long P) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    double *pPointer = reinterpret_cast<double *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDrotm(*handle, N, xPointer, incX, yPointer, incY, pPointer);

}

/*
 * ------------------------------------------------------
 * SWAP
 * ------------------------------------------------------
 */

void Nd4jBlas::sswap(long *extraParams, int N,
                     long X, int incX,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSswap(*handle, N, xPointer, incX, yPointer, incY);
}

void Nd4jBlas::dswap(long *extraParams, int N,
                     long X, int incX,
                     long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDswap(*handle, N, xPointer, incX, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * SCAL
 * ------------------------------------------------------
 */

void Nd4jBlas::sscal(long *extraParams, int N, float alpha,
                     long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSscal(*handle, N, &alpha, xPointer, incX);

}

void Nd4jBlas::dscal(long *extraParams, int N, double alpha,
                     long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDscal(*handle, N, &alpha, xPointer, incX);
}

/*
 * ------------------------------------------------------
 * SCOPY
 * ------------------------------------------------------
 */

void Nd4jBlas::scopy(long *extraParams, int N,
                     long X, int incX,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasScopy(*handle, N, xPointer, incX, yPointer, incY);
}

void Nd4jBlas::dcopy(long *extraParams, int N,
                     long X, int incX,
                     long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDcopy(*handle, N, xPointer, incX, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * AXPY
 * ------------------------------------------------------
 */

void Nd4jBlas::saxpy(long *extraParams, int N, float alpha,
                     long X, int incX,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSaxpy(*handle, N, &alpha, xPointer, incX, yPointer, incY);
}

void Nd4jBlas::daxpy(long *extraParams, int N, double alpha,
                     long X, int incX,
                     long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDaxpy(*handle, N, &alpha, xPointer, incX, yPointer, incY);

}

/*
 * ======================================================
 * Level 2 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgemv(long *extraParams, int Order, int TransA,
                     int M, int N,
                     float alpha,
                     long A, int lda,
                     long X, int incX,
                     float beta,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    float *aPointer = reinterpret_cast<float *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSgemv(*handle, convertTranspose(TransA), M, N, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer,
                   incY);
}

void Nd4jBlas::dgemv(long *extraParams, int Order, int TransA,
                     int M, int N,
                     double alpha,
                     long A, int lda,
                     long X, int incX,
                     double beta,
                     long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    double *aPointer = reinterpret_cast<double *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDgemv(*handle,convertTranspose(TransA),M,N,&alpha,aPointer,lda,xPointer,incX,&beta,yPointer,incY);
}

/*
 * ------------------------------------------------------
 * GBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgbmv(long *extraParams, int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     float alpha,
                     long A, int lda,
                     long X, int incX,
                     float beta,
                     long Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSgbmv(*handle, convertTranspose(TransA), M, N, KL, KU, &alpha, aPointer, lda, xPointer, incX, &beta,
                   yPointer, incY);
}

void Nd4jBlas::dgbmv(long *extraParams, int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     double alpha,
                     long A, int lda,
                     long X, int incX,
                     double beta,
                     long Y, int incY) {

    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDgbmv(*handle, convertTranspose(TransA), M, N, KL, KU, &alpha, aPointer, lda, xPointer, incX, &beta,
                   yPointer, incY);
}

/*
 * ------------------------------------------------------
 * SYMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymv(long *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     long A, int lda,
                     long X, int incX,
                     float beta,
                     long Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsymv(*handle, convertUplo(Uplo), N, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);
}

void Nd4jBlas::dsymv(long *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     long A, int lda,
                     long X, int incX,
                     double beta,
                     long Y, int incY) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsymv(*handle, convertUplo(Uplo), N, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * SBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssbmv(long *extraParams, int Order, int Uplo,
                     int N, int K,
                     float alpha,
                     long A, int lda,
                     long X, int incX,
                     float beta,
                     long Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsbmv(*handle, convertUplo(Uplo), N, K, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);
}

void Nd4jBlas::dsbmv(long *extraParams, int Order, int Uplo,
                     int N, int K,
                     double alpha,
                     long A, int lda,
                     long X, int incX,
                     double beta,
                     long Y, int incY) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsbmv(*handle, convertUplo(Uplo), N, K, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * SPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sspmv(long *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     long Ap,
                     long X, int incX,
                     float beta,
                     long Y, int incY) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSspmv(*handle, convertUplo(Uplo), N, &alpha, apPointer, xPointer, incX, &beta, yPointer, incY);

}

void Nd4jBlas::dspmv(long *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     long Ap,
                     long X, int incX,
                     double beta,
                     long Y, int incY) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDspmv(*handle, convertUplo(Uplo), N, &alpha, apPointer, xPointer, incX, &beta, yPointer, incY);


}

/*
 * ------------------------------------------------------
 * TRMV
 * ------------------------------------------------------
 */

void Nd4jBlas::strmv(long *extraParams, int Order, int Uplo, int TransA,
                     int Diag,
                     int N, float alpha,
                     long A, int lda,
                     long X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStrmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda,
                   xPointer, incX);
}

void Nd4jBlas::dtrmv(long *extraParams, int Order, int Uplo, int TransA,
                     int Diag,
                     int N, double alpha,
                     long A, int lda,
                     long X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtrmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda,
                   xPointer, incX);

}

/*
 * ------------------------------------------------------
 * TBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbmv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long A, int lda,
                     long X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStbmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);
}

void Nd4jBlas::dtbmv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long A, int lda,
                     long X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtbmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);

}

/*
 * ------------------------------------------------------
 * TPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpmv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long Ap,
                     long X, int incX) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStpmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);

}

void Nd4jBlas::dtpmv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long Ap,
                     long X, int incX) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtpmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);

}

/*
 * ------------------------------------------------------
 * TRSV
 * ------------------------------------------------------
 */

void Nd4jBlas::strsv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long A, int lda,
                     long X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStrsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda, xPointer,
                   incX);
}

void Nd4jBlas::dtrsv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long A, int lda,
                     long X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtrsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda, xPointer,
                   incX);

}

/*
 * ------------------------------------------------------
 * TBSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbsv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long A, int lda,
                     long X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStbsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);

}

void Nd4jBlas::dtbsv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long A, int lda,
                     long X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtbsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);

}

/*
 * ------------------------------------------------------
 * TPSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpsv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long Ap,
                     long X, int incX) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStpsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);
}

void Nd4jBlas::dtpsv(long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long Ap,
                     long X, int incX) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtpsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);

}

/*
 * ------------------------------------------------------
 * GER
 * ------------------------------------------------------
 */

void Nd4jBlas::sger(long *extraParams, int Order,
                    int M, int N,
                    float alpha,
                    long X, int incX,
                    long Y, int incY,
                    long A, int lda) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSger(*handle, M, N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

void Nd4jBlas::dger(long *extraParams, int Order,
                    int M, int N,
                    double alpha,
                    long X, int incX,
                    long Y, int incY,
                    long A, int lda) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDger(*handle, M, N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

/*
 * ------------------------------------------------------
 * SYR
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr(long *extraParams, int Order, int Uplo,
                    int N,
                    float alpha,
                    long X, int incX,
                    long A, int lda) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *aPointer = reinterpret_cast<float *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsyr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, aPointer, lda);
}

void Nd4jBlas::dsyr(long *extraParams, int Order, int Uplo,
                    int N,
                    double alpha,
                    long X, int incX,
                    long A, int lda) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *aPointer = reinterpret_cast<double *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsyr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, aPointer, lda);

}

/*
 * ------------------------------------------------------
 * SPR
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr(long *extraParams, int Order, int Uplo,
                    int N,
                    float alpha,
                    long X, int incX,
                    long Ap) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *apPointer = reinterpret_cast<float *>(Ap);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSspr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, apPointer);
}

void Nd4jBlas::dspr(long *extraParams, int Order, int Uplo,
                    int N,
                    double alpha,
                    long X, int incX,
                    long Ap) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *apPointer = reinterpret_cast<double *>(Ap);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDspr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, apPointer);

}

/*
 * ------------------------------------------------------
 * SYR2
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2(long *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     long X, int incX,
                     long Y, int incY,
                     long A, int lda) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsyr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

void Nd4jBlas::dsyr2(long *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     long X, int incX,
                     long Y, int incY,
                     long A, int lda) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsyr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

/*
 * ------------------------------------------------------
 * SPR2
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr2(long *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     long X, int incX,
                     long Y, int incY,
                     long Ap) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSspr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, apPointer);
}

void Nd4jBlas::dspr2(long *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     long X, int incX,
                     long Y, int incY,
                     long Ap) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDspr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, apPointer);

}

/*
 * ======================================================
 * Level 3 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMM
 * ------------------------------------------------------
 */

void Nd4jBlas::sgemm(long *extraParams, int Order, int TransA, int TransB,
                     int M, int N, int K,
                     float alpha,
                     long A, int lda,
                     long B, int ldb,
                     float beta,
                     long C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSgemm(*handle, convertTranspose(TransA), convertTranspose(TransB), M, N, K, &alpha, aPointer, lda,
                   bPointer, ldb, &beta, cPointer, ldc);

}

void Nd4jBlas::dgemm(long *extraParams, int Order, int TransA, int TransB,
                     int M, int N, int K,
                     double alpha,
                     long A, int lda,
                     long B, int ldb,
                     double beta,
                     long C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDgemm(*handle, convertTranspose(TransA), convertTranspose(TransB), M, N, K, &alpha, aPointer, lda,
                   bPointer, ldb, &beta, cPointer, ldc);

}

/*
 * ------------------------------------------------------
 * SYMM
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymm(long *extraParams, int Order, int Side, int Uplo,
                     int M, int N,
                     float alpha,
                     long A, int lda,
                     long B, int ldb,
                     float beta,
                     long C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsymm(*handle, convertSideMode(Side), convertUplo(Uplo), M, N, &alpha, aPointer, lda, bPointer, ldb, &beta,
                   cPointer, ldc);

}

void Nd4jBlas::dsymm(long *extraParams, int Order, int Side, int Uplo,
                     int M, int N,
                     double alpha,
                     long A, int lda,
                     long B, int ldb,
                     double beta,
                     long C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsymm(*handle, convertSideMode(Side), convertUplo(Uplo), M, N, &alpha, aPointer, lda, bPointer, ldb, &beta,
                   cPointer, ldc);

}

/*
 * ------------------------------------------------------
 * SYRK
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyrk(long *extraParams, int Order, int Uplo, int Trans,
                     int N, int K,
                     float alpha,
                     long A, int lda,
                     float beta,
                     long C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsyrk(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, &beta, cPointer,
                   ldc);
}

void Nd4jBlas::dsyrk(long *extraParams, int Order, int Uplo, int Trans,
                     int N, int K,
                     double alpha,
                     long A, int lda,
                     double beta,
                     long C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsyrk(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, &beta, cPointer,
                   ldc);

}

/*
 * ------------------------------------------------------
 * SYR2K
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2k(long *extraParams, int Order, int Uplo, int Trans,
                      int N, int K,
                      float alpha,
                      long A, int lda,
                      long B, int ldb,
                      float beta,
                      long C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsyr2k(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, bPointer, ldb,
                    &beta, cPointer, ldc);

}

void Nd4jBlas::dsyr2k(long *extraParams, int Order, int Uplo, int Trans,
                      int N, int K,
                      double alpha,
                      long A, int lda,
                      long B, int ldb,
                      double beta,
                      long C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsyr2k(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, bPointer, ldb,
                    &beta, cPointer, ldc);

}

/*
 * ------------------------------------------------------
 * TRMM
 * ------------------------------------------------------
 */

void Nd4jBlas::strmm(long *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     long A, int lda,
                     long B, int ldb) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    float *cPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasStrmm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb, bPointer, ldb);

}

void Nd4jBlas::dtrmm(long *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     long A, int lda,
                     long B, int ldb) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtrmm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb, bPointer, ldb);


}

/*
 * ------------------------------------------------------
 * TRSM
 * ------------------------------------------------------
 */

void Nd4jBlas::strsm(long *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     long A, int lda,
                     long B, int ldb) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStrsm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb);

}

void Nd4jBlas::dtrsm(long *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     long A, int lda,
                     long B, int ldb) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtrsm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb);


}

