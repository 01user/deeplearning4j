#include "../NativeBlas.h"
#include <hipblas.h>
#include <pointercast.h>
#include <stdio.h>



hipblasStatus_t convertStatus(int status) {
    switch (status) {
        case 0:
            return HIPBLAS_STATUS_SUCCESS;
        case 1:
            return HIPBLAS_STATUS_NOT_INITIALIZED;
        case 3:
            return HIPBLAS_STATUS_ALLOC_FAILED;
        case 7:
            return HIPBLAS_STATUS_INVALID_VALUE;
        case 8:
            return HIPBLAS_STATUS_ARCH_MISMATCH;
        case 11:
            return HIPBLAS_STATUS_MAPPING_ERROR;
        case 13:
            return HIPBLAS_STATUS_EXECUTION_FAILED;
        case 14:
            return HIPBLAS_STATUS_INTERNAL_ERROR;
        case 15:
            return HIPBLAS_STATUS_NOT_SUPPORTED;
        case 16:
            return HIPBLAS_STATUS_UNKNOWN;
        default:
            return HIPBLAS_STATUS_SUCCESS;
    }
}

hipblasFillMode_t convertUplo(int fillMode) {
    switch (fillMode) {
        case 0:
            return HIPBLAS_FILL_MODE_LOWER;
        case 1:
            return HIPBLAS_FILL_MODE_UPPER;
        default:
            return HIPBLAS_FILL_MODE_LOWER;
    }
}

hipblasDiagType_t convertDiag(int diag) {
    switch (diag) {
        case 0:
            return HIPBLAS_DIAG_NON_UNIT;
        case 1:
            return HIPBLAS_DIAG_UNIT;
        default:
            return HIPBLAS_DIAG_NON_UNIT;
    }
}

hipblasOperation_t convertTranspose(int op) {
    switch(op) {
        case 78: return HIPBLAS_OP_N;
        case 84: return HIPBLAS_OP_T;
        case 67: return HIPBLAS_OP_C;
        default: return HIPBLAS_OP_N;
    }
}

hipblasPointerMode_t convertPointerMode(int pointerMode) {
    switch(pointerMode) {
        case 0: return HIPBLAS_POINTER_MODE_HOST;
        case 1: return HIPBLAS_POINTER_MODE_DEVICE;
        default: return HIPBLAS_POINTER_MODE_HOST;
    }}

hipblasSideMode_t convertSideMode(int sideMode) {
    switch(sideMode) {
        case 0: return HIPBLAS_SIDE_LEFT;
        case 1: return HIPBLAS_SIDE_RIGHT;
        default: return HIPBLAS_SIDE_LEFT;
    }
}


/*
 * ======================================================
 * Level 1 BLAS functions
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * DOT
 * ------------------------------------------------------
 */

float Nd4jBlas::sdsdot(Nd4jPointer *extraParams, int N, float alpha,
                       Nd4jPointer X, int incX,
                       Nd4jPointer Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    //nothing found?
    return 0.0f;
}

double Nd4jBlas::dsdot(Nd4jPointer *extraParams, int N,
                       Nd4jPointer X, int incX,
                       Nd4jPointer Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    //nothing found?
    return 0.0f;
}

double Nd4jBlas::ddot(Nd4jPointer *extraParams, int N,
                      Nd4jPointer X, int incX,
                      Nd4jPointer Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    double resultPointer = 0.0f;
    hipblasDdot(*handle, N, xPointer, incX, yPointer, incY, &resultPointer);
    return resultPointer;
}

float Nd4jBlas::sdot(Nd4jPointer *extraParams, int N,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    float resultPointer = 0.0f;
    hipblasSdot(*handle, N, xPointer, incX, yPointer, incY, &resultPointer);
    return resultPointer;
}

/*
 * ------------------------------------------------------
 * NRM2
 * ------------------------------------------------------
 */

float Nd4jBlas::snrm2(Nd4jPointer *extraParams, int N, Nd4jPointer X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    float resultPointer = 0.0f;
    hipblasSnrm2(*handle, N, xPointer, incX, &resultPointer);
    return resultPointer;


}

double Nd4jBlas::dnrm2(Nd4jPointer *extraParams, int N, Nd4jPointer X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    double resultPointer = 0.0;
    hipblasDnrm2(*handle, N, xPointer, incX, &resultPointer);
    return resultPointer;
}

/*
 * ------------------------------------------------------
 * ASUM
 * ------------------------------------------------------
 */

float Nd4jBlas::sasum(Nd4jPointer *extraParams, int N, Nd4jPointer X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    float resultPointer = 0.0f;
    hipblasSasum(*handle, N, xPointer, incX, &resultPointer);
    return resultPointer;


}

double Nd4jBlas::dasum(Nd4jPointer *extraParams, int N, Nd4jPointer X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    double resultPointer = 0.0f;
    hipblasDasum(*handle, N, xPointer, incX, &resultPointer);
    return resultPointer;

}

/*
 * ------------------------------------------------------
 * IAMAX
 * ------------------------------------------------------
 */

int Nd4jBlas::isamax(Nd4jPointer *extraParams, int N, Nd4jPointer X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    int resultPointer = 0;
    hipblasIsamax(*handle,N,xPointer,incX,&resultPointer);
    return resultPointer;

}

int Nd4jBlas::idamax(Nd4jPointer *extraParams, int N, Nd4jPointer X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    int resultPointer = 0;
    hipblasIdamax(*handle, N, xPointer, incX, &resultPointer);
    return resultPointer;


}

/*
 * ======================================================
 * Level 1 BLAS procedures
 * ======================================================
 */

/*
 * ------------------------------------------------------
 * ROT
 * ------------------------------------------------------
 */

void Nd4jBlas::srot(Nd4jPointer *extraParams, int N,
                    Nd4jPointer X, int incX,
                    Nd4jPointer Y, int incY,
                    float c, float s) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSrot(*handle, N, xPointer, incX, yPointer, incY, &c, &s);
}

void Nd4jBlas::drot(Nd4jPointer *extraParams, int N,
                    Nd4jPointer X, int incX,
                    Nd4jPointer Y, int incY,
                    double c, double s) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDrot(*handle, N, xPointer, incX, yPointer, incY, &c, &s);
}

/*
 * ------------------------------------------------------
 * ROTG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotg(Nd4jPointer *extraParams, Nd4jPointer args) {
    float *argsPointers = reinterpret_cast<float *>(args);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSrotg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3]);
}

void Nd4jBlas::drotg(Nd4jPointer *extraParams, Nd4jPointer args) {
    double *argsPointers = reinterpret_cast<double *>(args);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDrotg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3]);

}

/*
 * ------------------------------------------------------
 * ROTMG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotmg(Nd4jPointer *extraParams, Nd4jPointer args,
                      Nd4jPointer P) {
    float *argsPointers = reinterpret_cast<float *>(args);
    float *pPointers = reinterpret_cast<float *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSrotmg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3], pPointers);

}

void Nd4jBlas::drotmg(Nd4jPointer *extraParams, Nd4jPointer args,
                      Nd4jPointer P) {
    double *argsPointers = reinterpret_cast<double *>(args);
    double *pPointers = reinterpret_cast<double *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDrotmg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3], pPointers);

}

/*
 * ------------------------------------------------------
 * ROTM
 * ------------------------------------------------------
 */

void Nd4jBlas::srotm(Nd4jPointer *extraParams, int N,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY,
                     Nd4jPointer P) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    float *pPointer = reinterpret_cast<float *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSrotm(*handle, N, xPointer, incX, yPointer, incY, pPointer);

}

void Nd4jBlas::drotm(Nd4jPointer *extraParams, int N,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY,
                     Nd4jPointer P) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    double *pPointer = reinterpret_cast<double *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDrotm(*handle, N, xPointer, incX, yPointer, incY, pPointer);

}

/*
 * ------------------------------------------------------
 * SWAP
 * ------------------------------------------------------
 */

void Nd4jBlas::sswap(Nd4jPointer *extraParams, int N,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSswap(*handle, N, xPointer, incX, yPointer, incY);
}

void Nd4jBlas::dswap(Nd4jPointer *extraParams, int N,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDswap(*handle, N, xPointer, incX, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * SCAL
 * ------------------------------------------------------
 */

void Nd4jBlas::sscal(Nd4jPointer *extraParams, int N, float alpha,
                     Nd4jPointer X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSscal(*handle, N, &alpha, xPointer, incX);

}

void Nd4jBlas::dscal(Nd4jPointer *extraParams, int N, double alpha,
                     Nd4jPointer X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDscal(*handle, N, &alpha, xPointer, incX);
}

/*
 * ------------------------------------------------------
 * SCOPY
 * ------------------------------------------------------
 */

void Nd4jBlas::scopy(Nd4jPointer *extraParams, int N,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasScopy(*handle, N, xPointer, incX, yPointer, incY);
}

void Nd4jBlas::dcopy(Nd4jPointer *extraParams, int N,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDcopy(*handle, N, xPointer, incX, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * AXPY
 * ------------------------------------------------------
 */

void Nd4jBlas::saxpy(Nd4jPointer *extraParams, int N, float alpha,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSaxpy(*handle, N, &alpha, xPointer, incX, yPointer, incY);
}

void Nd4jBlas::daxpy(Nd4jPointer *extraParams, int N, double alpha,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDaxpy(*handle, N, &alpha, xPointer, incX, yPointer, incY);

}

/*
 * ======================================================
 * Level 2 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgemv(Nd4jPointer *extraParams, int Order, int TransA,
                     int M, int N,
                     float alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX,
                     float beta,
                     Nd4jPointer Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    float *aPointer = reinterpret_cast<float *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSgemv(*handle, convertTranspose(TransA), M, N, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer,
                   incY);
}

void Nd4jBlas::dgemv(Nd4jPointer *extraParams, int Order, int TransA,
                     int M, int N,
                     double alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX,
                     double beta,
                     Nd4jPointer Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    double *aPointer = reinterpret_cast<double *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDgemv(*handle,convertTranspose(TransA),M,N,&alpha,aPointer,lda,xPointer,incX,&beta,yPointer,incY);
}

/*
 * ------------------------------------------------------
 * GBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgbmv(Nd4jPointer *extraParams, int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     float alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX,
                     float beta,
                     Nd4jPointer Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSgbmv(*handle, convertTranspose(TransA), M, N, KL, KU, &alpha, aPointer, lda, xPointer, incX, &beta,
                   yPointer, incY);
}

void Nd4jBlas::dgbmv(Nd4jPointer *extraParams, int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     double alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX,
                     double beta,
                     Nd4jPointer Y, int incY) {

    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDgbmv(*handle, convertTranspose(TransA), M, N, KL, KU, &alpha, aPointer, lda, xPointer, incX, &beta,
                   yPointer, incY);
}

/*
 * ------------------------------------------------------
 * SYMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX,
                     float beta,
                     Nd4jPointer Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsymv(*handle, convertUplo(Uplo), N, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);
}

void Nd4jBlas::dsymv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX,
                     double beta,
                     Nd4jPointer Y, int incY) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsymv(*handle, convertUplo(Uplo), N, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * SBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssbmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N, int K,
                     float alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX,
                     float beta,
                     Nd4jPointer Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsbmv(*handle, convertUplo(Uplo), N, K, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);
}

void Nd4jBlas::dsbmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N, int K,
                     double alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX,
                     double beta,
                     Nd4jPointer Y, int incY) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsbmv(*handle, convertUplo(Uplo), N, K, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * SPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sspmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     Nd4jPointer Ap,
                     Nd4jPointer X, int incX,
                     float beta,
                     Nd4jPointer Y, int incY) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSspmv(*handle, convertUplo(Uplo), N, &alpha, apPointer, xPointer, incX, &beta, yPointer, incY);

}

void Nd4jBlas::dspmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     Nd4jPointer Ap,
                     Nd4jPointer X, int incX,
                     double beta,
                     Nd4jPointer Y, int incY) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDspmv(*handle, convertUplo(Uplo), N, &alpha, apPointer, xPointer, incX, &beta, yPointer, incY);


}

/*
 * ------------------------------------------------------
 * TRMV
 * ------------------------------------------------------
 */

void Nd4jBlas::strmv(Nd4jPointer *extraParams, int Order, int Uplo, int TransA,
                     int Diag,
                     int N, float alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStrmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda,
                   xPointer, incX);
}

void Nd4jBlas::dtrmv(Nd4jPointer *extraParams, int Order, int Uplo, int TransA,
                     int Diag,
                     int N, double alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtrmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda,
                   xPointer, incX);

}

/*
 * ------------------------------------------------------
 * TBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStbmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);
}

void Nd4jBlas::dtbmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtbmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);

}

/*
 * ------------------------------------------------------
 * TPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     Nd4jPointer Ap,
                     Nd4jPointer X, int incX) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStpmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);

}

void Nd4jBlas::dtpmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     Nd4jPointer Ap,
                     Nd4jPointer X, int incX) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtpmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);

}

/*
 * ------------------------------------------------------
 * TRSV
 * ------------------------------------------------------
 */

void Nd4jBlas::strsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStrsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda, xPointer,
                   incX);
}

void Nd4jBlas::dtrsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtrsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda, xPointer,
                   incX);

}

/*
 * ------------------------------------------------------
 * TBSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStbsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);

}

void Nd4jBlas::dtbsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     Nd4jPointer A, int lda,
                     Nd4jPointer X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtbsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);

}

/*
 * ------------------------------------------------------
 * TPSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     Nd4jPointer Ap,
                     Nd4jPointer X, int incX) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStpsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);
}

void Nd4jBlas::dtpsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     Nd4jPointer Ap,
                     Nd4jPointer X, int incX) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtpsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);

}

/*
 * ------------------------------------------------------
 * GER
 * ------------------------------------------------------
 */

void Nd4jBlas::sger(Nd4jPointer *extraParams, int Order,
                    int M, int N,
                    float alpha,
                    Nd4jPointer X, int incX,
                    Nd4jPointer Y, int incY,
                    Nd4jPointer A, int lda) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSger(*handle, M, N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

void Nd4jBlas::dger(Nd4jPointer *extraParams, int Order,
                    int M, int N,
                    double alpha,
                    Nd4jPointer X, int incX,
                    Nd4jPointer Y, int incY,
                    Nd4jPointer A, int lda) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDger(*handle, M, N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

/*
 * ------------------------------------------------------
 * SYR
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr(Nd4jPointer *extraParams, int Order, int Uplo,
                    int N,
                    float alpha,
                    Nd4jPointer X, int incX,
                    Nd4jPointer A, int lda) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *aPointer = reinterpret_cast<float *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsyr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, aPointer, lda);
}

void Nd4jBlas::dsyr(Nd4jPointer *extraParams, int Order, int Uplo,
                    int N,
                    double alpha,
                    Nd4jPointer X, int incX,
                    Nd4jPointer A, int lda) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *aPointer = reinterpret_cast<double *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsyr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, aPointer, lda);

}

/*
 * ------------------------------------------------------
 * SPR
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr(Nd4jPointer *extraParams, int Order, int Uplo,
                    int N,
                    float alpha,
                    Nd4jPointer X, int incX,
                    Nd4jPointer Ap) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *apPointer = reinterpret_cast<float *>(Ap);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSspr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, apPointer);
}

void Nd4jBlas::dspr(Nd4jPointer *extraParams, int Order, int Uplo,
                    int N,
                    double alpha,
                    Nd4jPointer X, int incX,
                    Nd4jPointer Ap) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *apPointer = reinterpret_cast<double *>(Ap);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDspr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, apPointer);

}

/*
 * ------------------------------------------------------
 * SYR2
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY,
                     Nd4jPointer A, int lda) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsyr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

void Nd4jBlas::dsyr2(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY,
                     Nd4jPointer A, int lda) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsyr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

/*
 * ------------------------------------------------------
 * SPR2
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr2(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY,
                     Nd4jPointer Ap) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSspr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, apPointer);
}

void Nd4jBlas::dspr2(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     Nd4jPointer X, int incX,
                     Nd4jPointer Y, int incY,
                     Nd4jPointer Ap) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDspr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, apPointer);

}

/*
 * ======================================================
 * Level 3 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMM
 * ------------------------------------------------------
 */

void Nd4jBlas::sgemm(Nd4jPointer *extraParams, int Order, int TransA, int TransB,
                     int M, int N, int K,
                     float alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer B, int ldb,
                     float beta,
                     Nd4jPointer C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSgemm(*handle, convertTranspose(TransA), convertTranspose(TransB), M, N, K, &alpha, aPointer, lda,
                   bPointer, ldb, &beta, cPointer, ldc);

}

void Nd4jBlas::dgemm(Nd4jPointer *extraParams, int Order, int TransA, int TransB,
                     int M, int N, int K,
                     double alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer B, int ldb,
                     double beta,
                     Nd4jPointer C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDgemm(*handle, convertTranspose(TransA), convertTranspose(TransB), M, N, K, &alpha, aPointer, lda,
                   bPointer, ldb, &beta, cPointer, ldc);

}

/*
 * ------------------------------------------------------
 * SYMM
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymm(Nd4jPointer *extraParams, int Order, int Side, int Uplo,
                     int M, int N,
                     float alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer B, int ldb,
                     float beta,
                     Nd4jPointer C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsymm(*handle, convertSideMode(Side), convertUplo(Uplo), M, N, &alpha, aPointer, lda, bPointer, ldb, &beta,
                   cPointer, ldc);

}

void Nd4jBlas::dsymm(Nd4jPointer *extraParams, int Order, int Side, int Uplo,
                     int M, int N,
                     double alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer B, int ldb,
                     double beta,
                     Nd4jPointer C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsymm(*handle, convertSideMode(Side), convertUplo(Uplo), M, N, &alpha, aPointer, lda, bPointer, ldb, &beta,
                   cPointer, ldc);

}

/*
 * ------------------------------------------------------
 * SYRK
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyrk(Nd4jPointer *extraParams, int Order, int Uplo, int Trans,
                     int N, int K,
                     float alpha,
                     Nd4jPointer A, int lda,
                     float beta,
                     Nd4jPointer C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsyrk(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, &beta, cPointer,
                   ldc);
}

void Nd4jBlas::dsyrk(Nd4jPointer *extraParams, int Order, int Uplo, int Trans,
                     int N, int K,
                     double alpha,
                     Nd4jPointer A, int lda,
                     double beta,
                     Nd4jPointer C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsyrk(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, &beta, cPointer,
                   ldc);

}

/*
 * ------------------------------------------------------
 * SYR2K
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2k(Nd4jPointer *extraParams, int Order, int Uplo, int Trans,
                      int N, int K,
                      float alpha,
                      Nd4jPointer A, int lda,
                      Nd4jPointer B, int ldb,
                      float beta,
                      Nd4jPointer C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsyr2k(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, bPointer, ldb,
                    &beta, cPointer, ldc);

}

void Nd4jBlas::dsyr2k(Nd4jPointer *extraParams, int Order, int Uplo, int Trans,
                      int N, int K,
                      double alpha,
                      Nd4jPointer A, int lda,
                      Nd4jPointer B, int ldb,
                      double beta,
                      Nd4jPointer C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsyr2k(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, bPointer, ldb,
                    &beta, cPointer, ldc);

}

/*
 * ------------------------------------------------------
 * TRMM
 * ------------------------------------------------------
 */

void Nd4jBlas::strmm(Nd4jPointer *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer B, int ldb) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    float *cPointer = reinterpret_cast<float *>(&extraParams[1]);
    hipblasStrmm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb, bPointer, ldb);

}

void Nd4jBlas::dtrmm(Nd4jPointer *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer B, int ldb) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtrmm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb, bPointer, ldb);


}

/*
 * ------------------------------------------------------
 * TRSM
 * ------------------------------------------------------
 */

void Nd4jBlas::strsm(Nd4jPointer *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer B, int ldb) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStrsm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb);

}

void Nd4jBlas::dtrsm(Nd4jPointer *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     Nd4jPointer A, int lda,
                     Nd4jPointer B, int ldb) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtrsm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb);


}

