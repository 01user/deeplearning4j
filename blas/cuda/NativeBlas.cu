#include "../NativeBlas.h"
#include <hipblas.h>


hipblasStatus_t convertStatus(int status) {
    return reinterpret_cast<hipblasStatus_t>(status);
}

hipblasFillMode_t convertFillMode(int fillMode) {
    return reinterpret_cast<hipblasStatus_t>(fillMode);
}

hipblasDiagType_t convertDiag(int diag) {
    return reinterpret_cast<hipblasDiagType_t>(diag);
}

hipblasOperation_t convertTranspose(int op) {
    return reinterpret_cast<hipblasOperation_t>(op);
}
hipblasPointerMode_t convertPointerMode(int pointerMode) {
    return reinterpret_cast<hipblasPointerMode_t>(pointerMode);
}

hipblasSideMode_t convertSideMode(int sideMode) {
    return reinterpret_cast<hipblasSideMode_t>(sideMode);
}


/*
 *
 * /* CUBLAS status type returns
typedef enum{
    HIPBLAS_STATUS_SUCCESS         =0,
    HIPBLAS_STATUS_NOT_INITIALIZED =1,
    HIPBLAS_STATUS_ALLOC_FAILED    =3,
    HIPBLAS_STATUS_INVALID_VALUE   =7,
    HIPBLAS_STATUS_ARCH_MISMATCH   =8,
    HIPBLAS_STATUS_MAPPING_ERROR   =11,
    HIPBLAS_STATUS_EXECUTION_FAILED=13,
    HIPBLAS_STATUS_INTERNAL_ERROR  =14,
    HIPBLAS_STATUS_NOT_SUPPORTED   =15,
    HIPBLAS_STATUS_UNKNOWN   =16
} hipblasStatus_t;


typedef enum {
    HIPBLAS_FILL_MODE_LOWER=0,
    HIPBLAS_FILL_MODE_UPPER=1
} hipblasFillMode_t;

typedef enum {
    HIPBLAS_DIAG_NON_UNIT=0,
    HIPBLAS_DIAG_UNIT=1
} hipblasDiagType_t;

typedef enum {
    HIPBLAS_SIDE_LEFT =0,
    HIPBLAS_SIDE_RIGHT=1
} hipblasSideMode_t;


typedef enum {
    HIPBLAS_OP_N=0,
    HIPBLAS_OP_T=1,
    HIPBLAS_OP_C=2
} hipblasOperation_t;


typedef enum {
    HIPBLAS_POINTER_MODE_HOST   = 0,
    HIPBLAS_POINTER_MODE_DEVICE = 1
} hipblasPointerMode_t;

typedef enum {
    HIPBLAS_ATOMICS_NOT_ALLOWED   = 0,
    HIPBLAS_ATOMICS_ALLOWED       = 1
} hipblasAtomicsMode_t;
 //Used by cublasSgemmEx
typedef enum
{
    CUBLAS_DATA_FLOAT    = 0,
    CUBLAS_DATA_DOUBLE   = 1,
    CUBLAS_DATA_HALF     = 2,
    CUBLAS_DATA_INT8     = 3
} hipDataType;

 * */
/*
 * ======================================================
 * Level 1 BLAS functions
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * DOT
 * ------------------------------------------------------
 */

float Nd4jBlas::sdsdot(long *extraParams,int N, float alpha,
                       long X, int incX,
                       long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    //nothing found?

}

double Nd4jBlas::dsdot(long *extraParams,int N,
                       long X, int incX,
                       long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    //nothing found?
}

double Nd4jBlas::ddot(long *extraParams,int N,
                      long X, int incX,
                      long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasDdot(*handle,n,xPointer,incX,yPointer,incY,resultPointer);
    return 0.0;
}

float Nd4jBlas::sdot(long *extraParams,int N,
                     long X, int incX,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasSdot(*handle,n,xPointer,incX,yPointer,incY,resultPointer);
    return 0.0f;
}

/*
 * ------------------------------------------------------
 * NRM2
 * ------------------------------------------------------
 */

float Nd4jBlas::snrm2(long *extraParams,int N, long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasSnrm2(*handle,n,xPointer,incX,resultPointer);
    return 0.0f;


}

double Nd4jBlas::dnrm2(long *extraParams,int N, long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasDnrm2(*handle,n,xPointer,incX,resultPointer);
    return 0.0;
}

/*
 * ------------------------------------------------------
 * ASUM
 * ------------------------------------------------------
 */

float Nd4jBlas::sasum(long *extraParams,int N, long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasSasum(*handle,n,xPointer,incX,resultPointer);
    return 0.0f;


}

double Nd4jBlas::dasum(long *extraParams,int N, long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasDasum(*handle,n,xPointer,incX,resultPointer);
    return 0.0;

}

/*
 * ------------------------------------------------------
 * IAMAX
 * ------------------------------------------------------
 */

int Nd4jBlas::isamax(long *extraParams,int N, long X, int incX){
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasIsamax(*handle,n,xPointer,incX,resultPointer);
    return 0;

}

int Nd4jBlas::idamax(long *extraParams,int N, long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasIdamax(*handle,n,xPointer,incX,resultPointer);
    return 0;


}

/*
 * ======================================================
 * Level 1 BLAS procedures
 * ======================================================
 */

/*
 * ------------------------------------------------------
 * ROT
 * ------------------------------------------------------
 */

void Nd4jBlas::srot(long *extraParams,int N,
                    long X, int incX,
                    long Y, int incY,
                    float c, float s) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSrot(*handle,n,xPointer,incX,yPointer,incY,&c,&s);
}

void Nd4jBlas::drot(long *extraParams,int N,
                    long X, int incX,
                    long Y, int incY,
                    double c, double s) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDrot(*handle,n,xPointer,incX,yPointer,incY,&c,&s);
}

/*
 * ------------------------------------------------------
 * ROTG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotg(long *extraParams,long args) {
    float *argsPointers = reinterpret_cast<float *>(args);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSrotg(*handle,&argsPointers[0],&argsPointers[1],&argsPointers[2],&argsPointers[3]);
}

void Nd4jBlas::drotg(long *extraParams,long args) {
    double *argsPointers = reinterpret_cast<double *>(args);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDrotg(*handle,&argsPointers[0],&argsPointers[1],&argsPointers[2],&argsPointers[3]);

}

/*
 * ------------------------------------------------------
 * ROTMG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotmg(long *extraParams,long args,
                      long P) {
    float *argsPointers = reinterpret_cast<float *>(args);
    float *pPointers = reinterpret_cast<float *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSrotmg(*handle,&argsPointers[0],&argsPointers[1],&argsPointers[2],&argsPointers[3],pPointers);

}

void Nd4jBlas::drotmg(long *extraParams,long args,
                      long P) {
    double *argsPointers = reinterpret_cast<double *>(args);
    double *pPointers = reinterpret_cast<double *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDrotmg(*handle,&argsPointers[0],&argsPointers[1],&argsPointers[2],&argsPointers[3],pPointers);

}

/*
 * ------------------------------------------------------
 * ROTM
 * ------------------------------------------------------
 */

void Nd4jBlas::srotm(long *extraParams,int N,
                     long X, int incX,
                     long Y, int incY,
                     long P) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    float *pPointer = reinterpret_cast<float *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSrotm(*handle,n,xPointer,incX,yPointer,incY,pPointer);

}

void Nd4jBlas::drotm(long *extraParams,int N,
                     long X, int incX,
                     long Y, int incY,
                     long P) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    double *pPointer = reinterpret_cast<double *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDrotm(*handle,n,xPointer,incX,yPointer,incY,pPointer);

}

/*
 * ------------------------------------------------------
 * SWAP
 * ------------------------------------------------------
 */

void Nd4jBlas::sswap(long *extraParams,int N,
                     long X, int incX,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSswap(*handle,n,xPointer,incX,yPointer,incY);
}

void Nd4jBlas::dswap(long *extraParams,int N,
                     long X, int incX,
                     long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDswap(*handle,n,xPointer,incX,yPointer,incY);

}

/*
 * ------------------------------------------------------
 * SCAL
 * ------------------------------------------------------
 */

void Nd4jBlas::sscal(long *extraParams,int N, float alpha,
                     long X, int incX){
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSscal(*handle,n,&alpha,xPointer,incX);

}

void Nd4jBlas::dscal(long *extraParams,int N, double alpha,
                     long X, int incX){
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDscal(*handle,n,&alpha,xPointer,incX);
}

/*
 * ------------------------------------------------------
 * SCOPY
 * ------------------------------------------------------
 */

void Nd4jBlas::scopy(long *extraParams,int N,
                     long X, int incX,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasScopy(*handle,n,xPointer,incX,yPointer,incY);
}

void Nd4jBlas::dcopy(long *extraParams,int N,
                     long X, int incX,
                     long Y, int incY){
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDcopy(*handle,n,xPointer,incX,yPointer,incY);

}

/*
 * ------------------------------------------------------
 * AXPY
 * ------------------------------------------------------
 */

void Nd4jBlas::saxpy(long *extraParams,int N, float alpha,
                     long X, int incX,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSaxpy(*handle,n,&alpha,xPointer,incX,yPointer,incY);
}

void Nd4jBlas::daxpy(long *extraParams,int N, double alpha,
                     long X, int incX,
                     long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDaxpy(*handle,n,&alpha,xPointer,incX,yPointer,incY);

}

/*
 * ======================================================
 * Level 2 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgemv(long *extraParams,int Order, int TransA,
                     int M, int N,
                     float alpha,
                     long A, int lda,
                     long X, int incX,
                     float beta,
                     long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    float *aPointer = reinterpret_cast<float *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSgemv(*handle,convertTranspose(TransA),m,n,&alpha,aPointer,lda,xPointer,incX,&beta,yPointer,incY);
}

void Nd4jBlas::dgemv(long *extraParams,int Order, int TransA,
                     int M, int N,
                     double alpha,
                     long A, int lda,
                     long X, int incX,
                     double beta,
                     long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    double *aPointer = reinterpret_cast<double *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDgbmv(*handle,convertTranspose(TransA),m,n,kl,ku,&alpha,aPointer,lda,xPointer,incX,&beta,yPointer,incY);
}

/*
 * ------------------------------------------------------
 * GBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgbmv(long *extraParams,int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     float alpha,
                     long A, int lda,
                     long X, int incX,
                     float beta,
                     long Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSgbmv(*handle,convertTranspose(TransA),m,n,kl,ku,&alpha,aPointer,lda,xPointer,incX,&beta,yPointer,incY);
}

void Nd4jBlas::dgbmv(long *extraParams,int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     double alpha,
                     long A, int lda,
                     long X, int incX,
                     double beta,
                     long Y, int incY) {

    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDgbmv(*handle,convertTranspose(TransA),m,n,kl,ku,&alpha,aPointer,lda,xPointer,incX,&beta,yPointer,incY);
}

/*
 * ------------------------------------------------------
 * SYMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymv(long *extraParams,int Order, int Uplo,
                     int N,
                     float alpha,
                     long A, int lda,
                     long X, int incX,
                     float beta,
                     long Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSsymv(*handle,convertUplo(Uplo),n,&alpha,aPointer,lda,xPointer,incX,&beta,yPointer,incY);
}

void Nd4jBlas::dsymv(long *extraParams,int Order, int Uplo,
                     int N,
                     double alpha,
                     long A, int lda,
                     long X, int incX,
                     double beta,
                     long Y, int incY) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDsymv(*handle,convertUplo(Uplo),n,&alpha,aPointer,lda,xPointer,incX,&beta,yPointer,incY);

}

/*
 * ------------------------------------------------------
 * SBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssbmv(long *extraParams,int Order, int Uplo,
                     int N, int K,
                     float alpha,
                     long A, int lda,
                     long X, int incX,
                     float beta,
                     long Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSsbmv(*handle,convertUplo(Uplo),n,k,&alpha,aPointer,lda,xPointer,incx,&beta,yPointer,incY);
}

void Nd4jBlas::dsbmv(long *extraParams,int Order, int Uplo,
                     int N, int K,
                     double alpha,
                     long A, int lda,
                     long X, int incX,
                     double beta,
                     long Y, int incY){
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDsbmv(*handle,convertUplo(Uplo),n,k,&alpha,aPointer,lda,xPointer,incx,&beta,yPointer,incY);

}

/*
 * ------------------------------------------------------
 * SPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sspmv(long *extraParams,int Order, int Uplo,
                     int N,
                     float alpha,
                     long Ap,
                     long X, int incX,
                     float beta,
                     long Y, int incY){
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSspmv(*handle,convertUplo(Uplo),n,&alpha,apPointer,xPointer,incX,&beta,yPointer,incY);

}

void Nd4jBlas::dspmv(long *extraParams,int Order, int Uplo,
                     int N,
                     double alpha,
                     long Ap,
                     long X, int incX,
                     double beta,
                     long Y, int incY){
    double *apPointer = reinterpret_cast<double *>(p);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDspmv(*handle,convertUplo(Uplo),n,&alpha,apPointer,xPointer,incX,&beta,yPointer,incY);


}

/*
 * ------------------------------------------------------
 * TRMV
 * ------------------------------------------------------
 */

void Nd4jBlas::strmv(long *extraParams,int Order, int Uplo, int TransA,
                     int Diag,
                     int N, float alpha,
                     long A, int lda,
                     long X, int incX){
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasStrmv(*handle,convertFillMode(Uplo),convertTranspose(TransA),convertDiag(Diag),n,aPointer,lda,xPointer,incX);
}

void Nd4jBlas::dtrmv(long *extraParams,int Order, int Uplo, int TransA,
                     int Diag,
                     int N, double alpha,
                     long A, int lda,
                     long X, int incX){
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDtrmv(*handle,convertFillMode(Uplo),convertTranspose(TransA),convertDiag(Diag),n,aPointer,lda,xPointer,incX);

}

/*
 * ------------------------------------------------------
 * TBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbmv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long A, int lda,
                     long X, int incX){
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasStbmv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,k,aPointer,lda,xPointer,incX);
}

void Nd4jBlas::dtbmv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long A, int lda,
                     long X, int incX){
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDtbmv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,k,aPointer,lda,xPointer,incX);

}

/*
 * ------------------------------------------------------
 * TPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpmv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long Ap,
                     long X, int incX){
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasStpmv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,apPointer,xPointer,incx);

}

void Nd4jBlas::dtpmv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long Ap,
                     long X, int incX) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDtpmv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,apPointer,xPointer,incx);

}

/*
 * ------------------------------------------------------
 * TRSV
 * ------------------------------------------------------
 */

void Nd4jBlas::strsv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long A, int lda,
                     long X, int incX){
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasStrsv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,aPointer,lda,xPointer,incX);
}

void Nd4jBlas::dtrsv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long A, int lda,
                     long X, int incX){
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDtrsv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,aPointer,lda,xPointer,incX);

}

/*
 * ------------------------------------------------------
 * TBSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbsv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long A, int lda,
                     long X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasStbsv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,k,aPointer,lda,xPointer,incX);

}

void Nd4jBlas::dtbsv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long A, int lda,
                     long X, int incX){
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDtbsv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,k,aPointer,lda,xPointer,incX);

}

/*
 * ------------------------------------------------------
 * TPSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpsv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long Ap,
                     long X, int incX){
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasStpsv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,apPointer,xPointer,incX);
}

void Nd4jBlas::dtpsv(long *extraParams,int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long Ap,
                     long X, int incX){
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDtpsv(*handle,convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),n,apPointer,xPointer,incX);

}

/*
 * ------------------------------------------------------
 * GER
 * ------------------------------------------------------
 */

void Nd4jBlas::sger(long *extraParams,int Order,
                    int M, int N,
                    float alpha,
                    long X, int incX,
                    long Y, int incY,
                    long A, int lda){
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSger(*handle,m,n,&alpha,xPointer,incx,yPointer,incY,aPointer,lda);

}

void Nd4jBlas::dger(long *extraParams,int Order,
                    int M, int N,
                    double alpha,
                    long X, int incX,
                    long Y, int incY,
                    long A, int lda){
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDger(*handle,m,n,&alpha,xPointer,incx,yPointer,incY,aPointer,lda);

}

/*
 * ------------------------------------------------------
 * SYR
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr(long *extraParams,int Order, int Uplo,
                    int N,
                    float alpha,
                    long X, int incX,
                    long A, int lda){
    float *xPointer = reinterpret_cast<float *>(X);
    float *aPointer = reinterpret_cast<float *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSsyr(*handle,convertUplo(Uplo),n,&alpha,xPointer,incX,aPointer,lda);
}

void Nd4jBlas::dsyr(long *extraParams,int Order, int Uplo,
                    int N,
                    double alpha,
                    long X, int incX,
                    long A, int lda){
    double *xPointer = reinterpret_cast<double *>(X);
    double *aPointer = reinterpret_cast<double *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDsyr(*handle,convertUplo(Uplo),n,&alpha,xPointer,incX,aPointer,lda);

}

/*
 * ------------------------------------------------------
 * SPR
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr(long *extraParams,int Order, int Uplo,
                    int N,
                    float alpha,
                    long X, int incX,
                    long Ap){
    float *xPointer = reinterpret_cast<float *>(X);
    float *apPointer = reinterpret_cast<float *>(Ap);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSspr(*handle,convertUplo(Uplo),n,&alpha,xPointer,incX,apPointer);
}

void Nd4jBlas::dspr(long *extraParams,int Order, int Uplo,
                    int N,
                    double alpha,
                    long X, int incX,
                    long Ap){
    double *xPointer = reinterpret_cast<double *>(X);
    double *apPointer = reinterpret_cast<double *>(Ap);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDspr(*handle,convertUplo(Uplo),n,&alpha,xPointer,incX,apPointer);

}

/*
 * ------------------------------------------------------
 * SYR2
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2(long *extraParams,int Order, int Uplo,
                     int N,
                     float alpha,
                     long X, int incX,
                     long Y, int incY,
                     long A, int lda){
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSsyr2(*handle,convertUplo(Uplo),n,&alpha,xPointer,incX,yPointer,incY,aPointer,lda);

}

void Nd4jBlas::dsyr2(long *extraParams,int Order, int Uplo,
                     int N,
                     double alpha,
                     long X, int incX,
                     long Y, int incY,
                     long A, int lda){
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDsyr2(*handle,convertUplo(Uplo),n,&alpha,xPointer,incX,yPointer,incY,aPointer,lda);

}

/*
 * ------------------------------------------------------
 * SPR2
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr2(long *extraParams,int Order, int Uplo,
                     int N,
                     float alpha,
                     long X, int incX,
                     long Y, int incY,
                     long Ap){
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSspr2(*handle,convertUplo(Uplo),n,&alpha,xPointer,incX,yPointer,incY,apPointer);
}

void Nd4jBlas::dspr2(long *extraParams,int Order, int Uplo,
                     int N,
                     double alpha,
                     long X, int incX,
                     long Y, int incY,
                     long Ap){
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDspr2(*handle,convertUplo(Uplo),n,&alpha,xPointer,incX,yPointer,incY,apPointer);

}

/*
 * ======================================================
 * Level 3 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMM
 * ------------------------------------------------------
 */

void Nd4jBlas::sgemm(long *extraParams,int Order, int TransA, int TransB,
                     int M, int N, int K,
                     float alpha,
                     long A, int lda,
                     long B, int ldb,
                     float beta,
                     long C, int ldc){
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSgemm(*handle,convertTranspose(TransA),convertTranspose(TransB),m,n,k,&alpha,aPointer,lda,bPointer,ldb,&beta,cPointer,ldc);

}

void Nd4jBlas::dgemm(long *extraParams,int Order, int TransA, int TransB,
                     int M, int N, int K,
                     double alpha,
                     long A, int lda,
                     long B, int ldb,
                     double beta,
                     long C, int ldc){
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDgemm(*handle,convertTranspose(TransA),convertTranspose(TransB),m,n,k,&alpha,aPointer,lda,bPointer,ldb,&beta,cPointer,ldc);

}

/*
 * ------------------------------------------------------
 * SYMM
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymm(long *extraParams,int Order, int Side, int Uplo,
                     int M, int N,
                     float alpha,
                     long A, int lda,
                     long B, int ldb,
                     float beta,
                     long C, int ldc){
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSsymm(*handle,convertSideMode(Side),convertUplo(Uplo),m,n,&alpha,aPointer,lda,bPointer,ldb,&beta,cPointer,ldc);

}

void Nd4jBlas::dsymm(long *extraParams,int Order, int Side, int Uplo,
                     int M, int N,
                     double alpha,
                     long A, int lda,
                     long B, int ldb,
                     double beta,
                     long C, int ldc){
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDsymm(*handle,convertSideMode(Side),convertUplo(Uplo),m,n,&alpha,aPointer,lda,bPointer,ldb,&beta,cPointer,ldc);

}

/*
 * ------------------------------------------------------
 * SYRK
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyrk(long *extraParams,int Order, int Uplo, int Trans,
                     int N, int K,
                     float alpha,
                     long A, int lda,
                     float beta,
                     long C, int ldc){
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSsyrk(*handle,convertUplo(Uplo),convertTranspose(Trans),n,k,&alpha,aPointer,lda,&beta,cPointer,ldc);
}

void Nd4jBlas::dsyrk(long *extraParams,int Order, int Uplo, int Trans,
                     int N, int K,
                     double alpha,
                     long A, int lda,
                     double beta,
                     long C, int ldc){
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDsyrk(*handle,convertUplo(Uplo),convertTranspose(Trans),n,k,&alpha,aPointer,lda,&beta,cPointer,ldc);

}

/*
 * ------------------------------------------------------
 * SYR2K
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2k(long *extraParams,int Order, int Uplo, int Trans,
                      int N, int K,
                      float alpha,
                      long A, int lda,
                      long B, int ldb,
                      float beta,
                      long C, int ldc){
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasSsyr2k(*handle,convertUplo(Uplo),convertTranspose(Trans),n,k,&alpha,aPointer,lda,bPointer,ldb,&beta,cPointer,ldc);

}

void Nd4jBlas::dsyr2k(long *extraParams,int Order, int Uplo, int Trans,
                      int N, int K,
                      double alpha,
                      long A, int lda,
                      long B, int ldb,
                      double beta,
                      long C, int ldc){
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDsyr2k(*handle,convertUplo(Uplo),convertTranspose(Trans),n,k,&alpha,aPointer,lda,bPointer,ldb,&beta,cPointer,ldc);

}

/*
 * ------------------------------------------------------
 * TRMM
 * ------------------------------------------------------
 */

void Nd4jBlas::strmm(long *extraParams,int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     long A, int lda,
                     long B, int ldb){
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    float *cPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasStrmm(*handle,convertSideMode(Side),convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),m,n,&alpha,aPointer,lda,bPointer,ldb,bPointer,ldb);

}

void Nd4jBlas::dtrmm(long *extraParams,int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     long A, int lda,
                     long B, int ldb){
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDtrmm(*handle,convertSideMode(Side),convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),m,n,&alpha,aPointer,lda,bPointer,ldb,bPointer,ldb);


}

/*
 * ------------------------------------------------------
 * TRSM
 * ------------------------------------------------------
 */

void Nd4jBlas::strsm(long *extraParams,int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     long A, int lda,
                     long B, int ldb){
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasStrsm(*handle,convertSideMode(Side),convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),m,n,&alpha,aPointer,lda,bPointer,ldb);

}

void Nd4jBlas::dtrsm(long *extraParams,int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     long A, int lda,
                     long B, int ldb){
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t  *>(extraParams[0]);
    hipblasDtrsm(*handle,convertSideMode(Side),convertUplo(Uplo),convertTranspose(TransA),convertDiag(Diag),m,n,&alpha,aPointer,lda,bPointer,ldb);


}

