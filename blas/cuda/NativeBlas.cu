#include "../NativeBlas.h"
#include <hipblas.h>
#include <pointercast.h>



hipblasStatus_t convertStatus(int status) {
    switch (status) {
        case 0:
            return HIPBLAS_STATUS_SUCCESS;
        case 1:
            return HIPBLAS_STATUS_NOT_INITIALIZED;
        case 3:
            return HIPBLAS_STATUS_ALLOC_FAILED;
        case 7:
            return HIPBLAS_STATUS_INVALID_VALUE;
        case 8:
            return HIPBLAS_STATUS_ARCH_MISMATCH;
        case 11:
            return HIPBLAS_STATUS_MAPPING_ERROR;
        case 13:
            return HIPBLAS_STATUS_EXECUTION_FAILED;
        case 14:
            return HIPBLAS_STATUS_INTERNAL_ERROR;
        case 15:
            return HIPBLAS_STATUS_NOT_SUPPORTED;
        case 16:
            return HIPBLAS_STATUS_UNKNOWN;
        default:
            return HIPBLAS_STATUS_SUCCESS;
    }
}

hipblasFillMode_t convertUplo(int fillMode) {
    switch (fillMode) {
        case 0:
            return HIPBLAS_FILL_MODE_LOWER;
        case 1:
            return HIPBLAS_FILL_MODE_UPPER;
        default:
            return HIPBLAS_FILL_MODE_LOWER;
    }
}

hipblasDiagType_t convertDiag(int diag) {
    switch (diag) {
        case 0:
            return HIPBLAS_DIAG_NON_UNIT;
        case 1:
            return HIPBLAS_DIAG_UNIT;
        default:
            return HIPBLAS_DIAG_NON_UNIT;
    }
}

hipblasOperation_t convertTranspose(int op) {
    switch(op) {
        case 0: return HIPBLAS_OP_N;
        case 1: return HIPBLAS_OP_T;
        case 2: return HIPBLAS_OP_C;
        default: return HIPBLAS_OP_N;
    }
}

hipblasPointerMode_t convertPointerMode(int pointerMode) {
    switch(pointerMode) {
        case 0: return HIPBLAS_POINTER_MODE_HOST;
        case 1: return HIPBLAS_POINTER_MODE_DEVICE;
        default: return HIPBLAS_POINTER_MODE_HOST;
    }}

hipblasSideMode_t convertSideMode(int sideMode) {
    switch(sideMode) {
        case 0: return HIPBLAS_SIDE_LEFT;
        case 1: return HIPBLAS_SIDE_RIGHT;
        default: return HIPBLAS_SIDE_LEFT;
    }
}


/*
 * ======================================================
 * Level 1 BLAS functions
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * DOT
 * ------------------------------------------------------
 */

float Nd4jBlas::sdsdot(long long *extraParams, int N, float alpha,
                       long long X, int incX,
                       long long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    //nothing found?
    return 0.0f;
}

double Nd4jBlas::dsdot(long long *extraParams, int N,
                       long long X, int incX,
                       long long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    //nothing found?
    return 0.0f;
}

double Nd4jBlas::ddot(long long *extraParams, int N,
                      long long X, int incX,
                      long long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasDdot(*handle, N, xPointer, incX, yPointer, incY, resultPointer);
    return 0.0;
}

float Nd4jBlas::sdot(long long *extraParams, int N,
                     long long X, int incX,
                     long long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasSdot(*handle, N, xPointer, incX, yPointer, incY, resultPointer);
    return 0.0f;
}

/*
 * ------------------------------------------------------
 * NRM2
 * ------------------------------------------------------
 */

float Nd4jBlas::snrm2(long long *extraParams, int N, long long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasSnrm2(*handle, N, xPointer, incX, resultPointer);
    return 0.0f;


}

double Nd4jBlas::dnrm2(long long *extraParams, int N, long long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasDnrm2(*handle, N, xPointer, incX, resultPointer);
    return 0.0;
}

/*
 * ------------------------------------------------------
 * ASUM
 * ------------------------------------------------------
 */

float Nd4jBlas::sasum(long long *extraParams, int N, long long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    float *resultPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasSasum(*handle, N, xPointer, incX, resultPointer);
    return 0.0f;


}

double Nd4jBlas::dasum(long long *extraParams, int N, long long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    double *resultPointer = reinterpret_cast<double *>(extraParams[1]);
    hipblasDasum(*handle, N, xPointer, incX, resultPointer);
    return 0.0;

}

/*
 * ------------------------------------------------------
 * IAMAX
 * ------------------------------------------------------
 */

int Nd4jBlas::isamax(long long *extraParams, int N, long long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    int *resultPointer = reinterpret_cast<int *>(extraParams[1]);
    hipblasIsamax(*handle,N,xPointer,incX,resultPointer);
    return 0;

}

int Nd4jBlas::idamax(long long *extraParams, int N, long long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    int *resultPointer = reinterpret_cast<int *>(extraParams[1]);
    hipblasIdamax(*handle, N, xPointer, incX, resultPointer);
    return 0;


}

/*
 * ======================================================
 * Level 1 BLAS procedures
 * ======================================================
 */

/*
 * ------------------------------------------------------
 * ROT
 * ------------------------------------------------------
 */

void Nd4jBlas::srot(long long *extraParams, int N,
                    long long X, int incX,
                    long long Y, int incY,
                    float c, float s) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSrot(*handle, N, xPointer, incX, yPointer, incY, &c, &s);
}

void Nd4jBlas::drot(long long *extraParams, int N,
                    long long X, int incX,
                    long long Y, int incY,
                    double c, double s) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDrot(*handle, N, xPointer, incX, yPointer, incY, &c, &s);
}

/*
 * ------------------------------------------------------
 * ROTG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotg(long long *extraParams, long long args) {
    float *argsPointers = reinterpret_cast<float *>(args);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSrotg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3]);
}

void Nd4jBlas::drotg(long long *extraParams, long long args) {
    double *argsPointers = reinterpret_cast<double *>(args);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDrotg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3]);

}

/*
 * ------------------------------------------------------
 * ROTMG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotmg(long long *extraParams, long long args,
                      long long P) {
    float *argsPointers = reinterpret_cast<float *>(args);
    float *pPointers = reinterpret_cast<float *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSrotmg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3], pPointers);

}

void Nd4jBlas::drotmg(long long *extraParams, long long args,
                      long long P) {
    double *argsPointers = reinterpret_cast<double *>(args);
    double *pPointers = reinterpret_cast<double *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDrotmg(*handle, &argsPointers[0], &argsPointers[1], &argsPointers[2], &argsPointers[3], pPointers);

}

/*
 * ------------------------------------------------------
 * ROTM
 * ------------------------------------------------------
 */

void Nd4jBlas::srotm(long long *extraParams, int N,
                     long long X, int incX,
                     long long Y, int incY,
                     long long P) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    float *pPointer = reinterpret_cast<float *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSrotm(*handle, N, xPointer, incX, yPointer, incY, pPointer);

}

void Nd4jBlas::drotm(long long *extraParams, int N,
                     long long X, int incX,
                     long long Y, int incY,
                     long long P) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    double *pPointer = reinterpret_cast<double *>(P);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDrotm(*handle, N, xPointer, incX, yPointer, incY, pPointer);

}

/*
 * ------------------------------------------------------
 * SWAP
 * ------------------------------------------------------
 */

void Nd4jBlas::sswap(long long *extraParams, int N,
                     long long X, int incX,
                     long long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSswap(*handle, N, xPointer, incX, yPointer, incY);
}

void Nd4jBlas::dswap(long long *extraParams, int N,
                     long long X, int incX,
                     long long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDswap(*handle, N, xPointer, incX, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * SCAL
 * ------------------------------------------------------
 */

void Nd4jBlas::sscal(long long *extraParams, int N, float alpha,
                     long long X, int incX) {
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSscal(*handle, N, &alpha, xPointer, incX);

}

void Nd4jBlas::dscal(long long *extraParams, int N, double alpha,
                     long long X, int incX) {
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDscal(*handle, N, &alpha, xPointer, incX);
}

/*
 * ------------------------------------------------------
 * SCOPY
 * ------------------------------------------------------
 */

void Nd4jBlas::scopy(long long *extraParams, int N,
                     long long X, int incX,
                     long long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasScopy(*handle, N, xPointer, incX, yPointer, incY);
}

void Nd4jBlas::dcopy(long long *extraParams, int N,
                     long long X, int incX,
                     long long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDcopy(*handle, N, xPointer, incX, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * AXPY
 * ------------------------------------------------------
 */

void Nd4jBlas::saxpy(long long *extraParams, int N, float alpha,
                     long long X, int incX,
                     long long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSaxpy(*handle, N, &alpha, xPointer, incX, yPointer, incY);
}

void Nd4jBlas::daxpy(long long *extraParams, int N, double alpha,
                     long long X, int incX,
                     long long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDaxpy(*handle, N, &alpha, xPointer, incX, yPointer, incY);

}

/*
 * ======================================================
 * Level 2 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgemv(long long *extraParams, int Order, int TransA,
                     int M, int N,
                     float alpha,
                     long long A, int lda,
                     long long X, int incX,
                     float beta,
                     long long Y, int incY) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    float *aPointer = reinterpret_cast<float *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSgemv(*handle, convertTranspose(TransA), M, N, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer,
                   incY);
}

void Nd4jBlas::dgemv(long long *extraParams, int Order, int TransA,
                     int M, int N,
                     double alpha,
                     long long A, int lda,
                     long long X, int incX,
                     double beta,
                     long long Y, int incY) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    double *aPointer = reinterpret_cast<double *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDgemv(*handle,convertTranspose(TransA),M,N,&alpha,aPointer,lda,xPointer,incX,&beta,yPointer,incY);
}

/*
 * ------------------------------------------------------
 * GBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgbmv(long long *extraParams, int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     float alpha,
                     long long A, int lda,
                     long long X, int incX,
                     float beta,
                     long long Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSgbmv(*handle, convertTranspose(TransA), M, N, KL, KU, &alpha, aPointer, lda, xPointer, incX, &beta,
                   yPointer, incY);
}

void Nd4jBlas::dgbmv(long long *extraParams, int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     double alpha,
                     long long A, int lda,
                     long long X, int incX,
                     double beta,
                     long long Y, int incY) {

    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDgbmv(*handle, convertTranspose(TransA), M, N, KL, KU, &alpha, aPointer, lda, xPointer, incX, &beta,
                   yPointer, incY);
}

/*
 * ------------------------------------------------------
 * SYMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymv(long long *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     long long A, int lda,
                     long long X, int incX,
                     float beta,
                     long long Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsymv(*handle, convertUplo(Uplo), N, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);
}

void Nd4jBlas::dsymv(long long *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     long long A, int lda,
                     long long X, int incX,
                     double beta,
                     long long Y, int incY) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsymv(*handle, convertUplo(Uplo), N, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * SBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssbmv(long long *extraParams, int Order, int Uplo,
                     int N, int K,
                     float alpha,
                     long long A, int lda,
                     long long X, int incX,
                     float beta,
                     long long Y, int incY) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsbmv(*handle, convertUplo(Uplo), N, K, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);
}

void Nd4jBlas::dsbmv(long long *extraParams, int Order, int Uplo,
                     int N, int K,
                     double alpha,
                     long long A, int lda,
                     long long X, int incX,
                     double beta,
                     long long Y, int incY) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsbmv(*handle, convertUplo(Uplo), N, K, &alpha, aPointer, lda, xPointer, incX, &beta, yPointer, incY);

}

/*
 * ------------------------------------------------------
 * SPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sspmv(long long *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     long long Ap,
                     long long X, int incX,
                     float beta,
                     long long Y, int incY) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSspmv(*handle, convertUplo(Uplo), N, &alpha, apPointer, xPointer, incX, &beta, yPointer, incY);

}

void Nd4jBlas::dspmv(long long *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     long long Ap,
                     long long X, int incX,
                     double beta,
                     long long Y, int incY) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDspmv(*handle, convertUplo(Uplo), N, &alpha, apPointer, xPointer, incX, &beta, yPointer, incY);


}

/*
 * ------------------------------------------------------
 * TRMV
 * ------------------------------------------------------
 */

void Nd4jBlas::strmv(long long *extraParams, int Order, int Uplo, int TransA,
                     int Diag,
                     int N, float alpha,
                     long long A, int lda,
                     long long X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStrmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda,
                   xPointer, incX);
}

void Nd4jBlas::dtrmv(long long *extraParams, int Order, int Uplo, int TransA,
                     int Diag,
                     int N, double alpha,
                     long long A, int lda,
                     long long X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtrmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda,
                   xPointer, incX);

}

/*
 * ------------------------------------------------------
 * TBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbmv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long long A, int lda,
                     long long X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStbmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);
}

void Nd4jBlas::dtbmv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long long A, int lda,
                     long long X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtbmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);

}

/*
 * ------------------------------------------------------
 * TPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpmv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long long Ap,
                     long long X, int incX) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStpmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);

}

void Nd4jBlas::dtpmv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long long Ap,
                     long long X, int incX) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtpmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);

}

/*
 * ------------------------------------------------------
 * TRSV
 * ------------------------------------------------------
 */

void Nd4jBlas::strsv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long long A, int lda,
                     long long X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStrsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda, xPointer,
                   incX);
}

void Nd4jBlas::dtrsv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long long A, int lda,
                     long long X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtrsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, aPointer, lda, xPointer,
                   incX);

}

/*
 * ------------------------------------------------------
 * TBSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbsv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long long A, int lda,
                     long long X, int incX) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStbsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);

}

void Nd4jBlas::dtbsv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     long long A, int lda,
                     long long X, int incX) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtbsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, aPointer, lda,
                   xPointer, incX);

}

/*
 * ------------------------------------------------------
 * TPSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpsv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long long Ap,
                     long long X, int incX) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStpsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);
}

void Nd4jBlas::dtpsv(long long *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     long long Ap,
                     long long X, int incX) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtpsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, apPointer, xPointer,
                   incX);

}

/*
 * ------------------------------------------------------
 * GER
 * ------------------------------------------------------
 */

void Nd4jBlas::sger(long long *extraParams, int Order,
                    int M, int N,
                    float alpha,
                    long long X, int incX,
                    long long Y, int incY,
                    long long A, int lda) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSger(*handle, M, N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

void Nd4jBlas::dger(long long *extraParams, int Order,
                    int M, int N,
                    double alpha,
                    long long X, int incX,
                    long long Y, int incY,
                    long long A, int lda) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDger(*handle, M, N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

/*
 * ------------------------------------------------------
 * SYR
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr(long long *extraParams, int Order, int Uplo,
                    int N,
                    float alpha,
                    long long X, int incX,
                    long long A, int lda) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *aPointer = reinterpret_cast<float *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsyr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, aPointer, lda);
}

void Nd4jBlas::dsyr(long long *extraParams, int Order, int Uplo,
                    int N,
                    double alpha,
                    long long X, int incX,
                    long long A, int lda) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *aPointer = reinterpret_cast<double *>(A);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsyr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, aPointer, lda);

}

/*
 * ------------------------------------------------------
 * SPR
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr(long long *extraParams, int Order, int Uplo,
                    int N,
                    float alpha,
                    long long X, int incX,
                    long long Ap) {
    float *xPointer = reinterpret_cast<float *>(X);
    float *apPointer = reinterpret_cast<float *>(Ap);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSspr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, apPointer);
}

void Nd4jBlas::dspr(long long *extraParams, int Order, int Uplo,
                    int N,
                    double alpha,
                    long long X, int incX,
                    long long Ap) {
    double *xPointer = reinterpret_cast<double *>(X);
    double *apPointer = reinterpret_cast<double *>(Ap);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDspr(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, apPointer);

}

/*
 * ------------------------------------------------------
 * SYR2
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2(long long *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     long long X, int incX,
                     long long Y, int incY,
                     long long A, int lda) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsyr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

void Nd4jBlas::dsyr2(long long *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     long long X, int incX,
                     long long Y, int incY,
                     long long A, int lda) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsyr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, aPointer, lda);

}

/*
 * ------------------------------------------------------
 * SPR2
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr2(long long *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     long long X, int incX,
                     long long Y, int incY,
                     long long Ap) {
    float *apPointer = reinterpret_cast<float *>(Ap);
    float *xPointer = reinterpret_cast<float *>(X);
    float *yPointer = reinterpret_cast<float *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSspr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, apPointer);
}

void Nd4jBlas::dspr2(long long *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     long long X, int incX,
                     long long Y, int incY,
                     long long Ap) {
    double *apPointer = reinterpret_cast<double *>(Ap);
    double *xPointer = reinterpret_cast<double *>(X);
    double *yPointer = reinterpret_cast<double *>(Y);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDspr2(*handle, convertUplo(Uplo), N, &alpha, xPointer, incX, yPointer, incY, apPointer);

}

/*
 * ======================================================
 * Level 3 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMM
 * ------------------------------------------------------
 */

void Nd4jBlas::sgemm(long long *extraParams, int Order, int TransA, int TransB,
                     int M, int N, int K,
                     float alpha,
                     long long A, int lda,
                     long long B, int ldb,
                     float beta,
                     long long C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSgemm(*handle, convertTranspose(TransA), convertTranspose(TransB), M, N, K, &alpha, aPointer, lda,
                   bPointer, ldb, &beta, cPointer, ldc);

}

void Nd4jBlas::dgemm(long long *extraParams, int Order, int TransA, int TransB,
                     int M, int N, int K,
                     double alpha,
                     long long A, int lda,
                     long long B, int ldb,
                     double beta,
                     long long C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDgemm(*handle, convertTranspose(TransA), convertTranspose(TransB), M, N, K, &alpha, aPointer, lda,
                   bPointer, ldb, &beta, cPointer, ldc);

}

/*
 * ------------------------------------------------------
 * SYMM
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymm(long long *extraParams, int Order, int Side, int Uplo,
                     int M, int N,
                     float alpha,
                     long long A, int lda,
                     long long B, int ldb,
                     float beta,
                     long long C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsymm(*handle, convertSideMode(Side), convertUplo(Uplo), M, N, &alpha, aPointer, lda, bPointer, ldb, &beta,
                   cPointer, ldc);

}

void Nd4jBlas::dsymm(long long *extraParams, int Order, int Side, int Uplo,
                     int M, int N,
                     double alpha,
                     long long A, int lda,
                     long long B, int ldb,
                     double beta,
                     long long C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsymm(*handle, convertSideMode(Side), convertUplo(Uplo), M, N, &alpha, aPointer, lda, bPointer, ldb, &beta,
                   cPointer, ldc);

}

/*
 * ------------------------------------------------------
 * SYRK
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyrk(long long *extraParams, int Order, int Uplo, int Trans,
                     int N, int K,
                     float alpha,
                     long long A, int lda,
                     float beta,
                     long long C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsyrk(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, &beta, cPointer,
                   ldc);
}

void Nd4jBlas::dsyrk(long long *extraParams, int Order, int Uplo, int Trans,
                     int N, int K,
                     double alpha,
                     long long A, int lda,
                     double beta,
                     long long C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsyrk(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, &beta, cPointer,
                   ldc);

}

/*
 * ------------------------------------------------------
 * SYR2K
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2k(long long *extraParams, int Order, int Uplo, int Trans,
                      int N, int K,
                      float alpha,
                      long long A, int lda,
                      long long B, int ldb,
                      float beta,
                      long long C, int ldc) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    float *cPointer = reinterpret_cast<float *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasSsyr2k(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, bPointer, ldb,
                    &beta, cPointer, ldc);

}

void Nd4jBlas::dsyr2k(long long *extraParams, int Order, int Uplo, int Trans,
                      int N, int K,
                      double alpha,
                      long long A, int lda,
                      long long B, int ldb,
                      double beta,
                      long long C, int ldc) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    double *cPointer = reinterpret_cast<double *>(C);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDsyr2k(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, aPointer, lda, bPointer, ldb,
                    &beta, cPointer, ldc);

}

/*
 * ------------------------------------------------------
 * TRMM
 * ------------------------------------------------------
 */

void Nd4jBlas::strmm(long long *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     long long A, int lda,
                     long long B, int ldb) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    float *cPointer = reinterpret_cast<float *>(extraParams[1]);
    hipblasStrmm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb, bPointer, ldb);

}

void Nd4jBlas::dtrmm(long long *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     long long A, int lda,
                     long long B, int ldb) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtrmm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb, bPointer, ldb);


}

/*
 * ------------------------------------------------------
 * TRSM
 * ------------------------------------------------------
 */

void Nd4jBlas::strsm(long long *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     long long A, int lda,
                     long long B, int ldb) {
    float *aPointer = reinterpret_cast<float *>(A);
    float *bPointer = reinterpret_cast<float *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasStrsm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb);

}

void Nd4jBlas::dtrsm(long long *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     long long A, int lda,
                     long long B, int ldb) {
    double *aPointer = reinterpret_cast<double *>(A);
    double *bPointer = reinterpret_cast<double *>(B);
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(extraParams[0]);
    hipblasDtrsm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, aPointer, lda, bPointer, ldb);


}

