#include "../NativeOps.h"
#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

#include <buffer.h>
#include <shape.h>

#include <reduce3.h>
#include <reduce.h>
#include <indexreduce.h>
#include <pairwise_transform.h>
#include <transform.h>
#include <scalar.h>
#include <broadcasting.h>
#include <summarystatsreduce.h>


dim3 getOptimalDimensions(int n,hipFuncAttributes attributes) {
    // next, get the hipDeviceProp_t object corresponding to the current device
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);

    // we can combine the two to compute a block size
    size_t num_threads = block_size_with_maximum_potential_occupancy(attributes, properties);

    // compute the number of blocks of size num_threads to launch
    size_t num_blocks = n / num_threads;

    // check for partial block at the end
    if(n % num_threads) ++num_blocks;

    return dim3(num_blocks,num_threads,1);
}

nd4j::buffer::Buffer<int> * createScalarBuffer() {
    int *scalarShapeInfo = shape::createScalarShapeInfo();
    nd4j::buffer::Buffer<int> *buff = nd4j::buffer::createBuffer(scalarShapeInfo,shape::shapeInfoLength(2));
    nd4j::buffer::copyDataToGpu(&buff);
    return buff;
}

template <typename T>
class ScalarInfo {
    nd4j::buffer::Buffer<T> *scalarData;
    nd4j::buffer::Buffer<int> scalarDimension;
    nd4j::buffer::Buffer<int> *scalarShapeInfo;
    T finalResult;
public:
    ScalarInfo() {
        scalarShapeInfo = createScalarBuffer();
        T *scalarResult = malloc(sizeof(T));
        scalarData = nd4j::buffer::createBuffer(scalarResult,1);
        nd4j::buffer::copyDataToGpu(&scalarData);
        int *scalarDimensionBuff = malloc(sizeof(int));
        scalarDimension[0] = shape::MAX_DIMENSION;
        scalarDimension = nd4j::buffer::createBuffer(scalarDimensionBuff,1);
        nd4j::buffer::copyDataToGpu(&scalarDimension);
    }

    T getFinalResultFromDevice() {
        nd4j::buffer::copyDataFromGpu(&scalarData);
        return scalarData[0];
    }

    /**
     * Get the device shape information
     * representinga scalar
     */
    int *getDeviceShapeInfo() {
        return scalarShapeInfo->gData;
    }

    /**
     * Get the result pointers
     */
    T *getDevicePointer() {
        return scalarData->gData;
    }

    /**
     * Get the infinite dimension device pointer
     */
    int *getDimensionDevicePointer() {
        return scalarDimension.gData;
    }

    ~ScalarInfo() {
        nd4j::buffer::freeBuffer(&scalarShapeInfo);
        nd4j::buffer::freeBuffer(&scalarData);
        nd4j::buffer::freeBuffer(&scalarDimension);
    }
};

/**
     *
     * @param opNum
     * @param x
     * @param xShapeInfo
     * @param extraParams
     */
double   NativeOps::execIndexReduceScalarDouble(int opNum,
                                                long x,
                                                long xShapeInfo,
                                                long extraParams) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    hipFuncAttributes attributes;
    hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(indexReduceDouble));
    dim3 launchDims = getOptimalDimensions(1,attributes);

    ScalarInfo<double> *scalarInfo = new ScalarInfo<double>();
    indexReduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
                    opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    NULL,
                    scalarInfo->getDevicePointer(),
                    scalarInfo->getDimensionDevicePointer(),
                    scalarInfo->getDimensionDevicePointer(),
                    1,
                    1);
    hipDeviceSynchronize();

    return scalarInfo->getFinalResultFromDevice();

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceDouble(int opNum,
                                        long x,
                                        long xShapeInfo,
                                        long extraParams,
                                        long result,
                                        long resultShapeInfoBuffer,
                                        long dimension, int dimensionLength) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execIndexReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);


}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastDouble(int opNum,
                                      long x,
                                      long xShapeInfo,
                                      long y,
                                      long yShapeInfo,
                                      long result,
                                      long resultShapeInfo,
                                      long dimension, int dimensionLength){
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execBroadcast(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformDouble(int opNum,
                                              long dx,
                                              int xStride,
                                              long y,
                                              int yStride,
                                              long result,
                                              int resultStride,
                                              long extraParams, int n){
    double *xPointer = reinterpret_cast<double *>(dx);
    double *yPointer = reinterpret_cast<double *>(y);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xStride,yPointer,yStride,resultPointer,resultStride,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformDouble(int opNum,
                                            long dx,
                                            long xShapeInfo,
                                            long y,
                                            long yShapeInfo,
                                            long result,
                                            long resultShapeInfo,
                                            long extraParams,
                                            int n,
                                            long xIndexes,
                                            long yIndexes,
                                            long resultIndexes){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return DoubleNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n,xIndexesPointer,yIndexesPointer,resultIndexesPointer);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformDouble(int opNum,
                                            long dx,
                                            long  xShapeInfo,
                                            long y,
                                            long  yShapeInfo,
                                            long result,
                                            long  resultShapeInfo,
                                            long extraParams, int n) {
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(int opNum,
                                   long x,
                                   long xShapeInfo,
                                   long extraParams,
                                   long result,
                                   long resultShapeInfo){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(int opNum,
                                   long x,
                                   long xShapeInfo,
                                   long extraParams,
                                   long result,
                                   long resultShapeInfo,
                                   long dimension,int dimensionLength) {
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
double NativeOps::execReduceScalarDouble(int opNum,
                                         long x,
                                         long xShapeInfo,
                                         long extraParams){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execReduceScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Double(int opNum,
                                    long x,
                                    long xShapeInfo,
                                    long extraParamsVals,
                                    long y,
                                    long yShapeInfo,
                                    long result,
                                    long resultShapeInfo){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execReduce3(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
double   NativeOps::execReduce3ScalarDouble(int opNum,
                                            long x,
                                            long xShapeInfo,
                                            long extraParamsVals,
                                            long y,
                                            long yShapeInfo){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execReduce3Scalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Double(int opNum,
                                    long x,
                                    long xShapeInfo,
                                    long extraParamsVals,
                                    long y,
                                    long yShapeInfo,
                                    long result,
                                    long resultShapeInfoBuffer,
                                    long dimension,
                                    int dimensionLength){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execReduce3(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer);

}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarDouble(int opNum,
                                   long x,
                                   int xStride,
                                   long result,
                                   int resultStride,
                                   double scalar,
                                   long extraParams,
                                   int n) {
    double *xPointer = reinterpret_cast<double *>(dx);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xStride,resultPointer,resultStride,scalar,extraParamsPointer,n);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarDouble(int opNum,
                                 long x,
                                 long xShapeInfo,
                                 long result,
                                 long resultShapeInfo,
                                 double scalar,
                                 long extraParams,
                                 int n){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,scalar,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarDouble(int opNum,
                                 long x,
                                 long xShapeInfo,
                                 long result,
                                 long resultShapeInfo,
                                 double scalar,
                                 long extraParams,
                                 int n,
                                 long xIndexes,
                                 long resultIndexes){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return DoubleNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,scalar,extraParamsPointer,n,xIndexesPointer,resultIndexesPointer);

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execSummaryStatsScalarDouble(int opNum,long x,
                                                 long xShapeInfo,
                                                 long extraParams){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execSummaryStatsScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsDouble(int opNum,
                                         long x,
                                         long xShapeInfo,
                                         long extraParams,
                                         long result,
                                         long resultShapeInfo){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execSummaryStats(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsDouble(int opNum,long x,
                                         long xShapeInfo,
                                         long extraParams,
                                         long result,
                                         long resultShapeInfoBuffer,
                                         long dimension, int dimensionLength){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execSummaryStats(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(int opNum,
                                      long dx,
                                      int xStride,
                                      long result,
                                      int resultStride,
                                      long extraParams, int n){
    double *xPointer = reinterpret_cast<double *>(dx);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xStride,resultPointer,resultStride,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(int opNum,
                                      long dx,
                                      long xShapeInfo,
                                      long result,
                                      long resultShapeInfo,
                                      long extraParams, int n){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(int opNum,
                                      long dx,
                                      long xShapeInfo,
                                      long result,
                                      long resultShapeInfo,
                                      long extraParams,
                                      int n,
                                      long xIndexes,
                                      long resultIndexes){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return DoubleNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n,xIndexesPointer,resultIndexesPointer);

}

/**
*
* @param opNum
* @param x
* @param xShapeInfo
* @param extraParams
*/
double   NativeOps::execIndexReduceScalarFloat(int opNum,
                                               long x,
                                               long xShapeInfo,
                                               long extraParams){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execIndexReduceScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceFloat(int opNum,
                                       long x,
                                       long xShapeInfo,
                                       long extraParams,
                                       long result,
                                       long resultShapeInfoBuffer,
                                       long dimension, int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return FloatNativeOpExecutioner::getInstance()->execIndexReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);


}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastFloat(int opNum,
                                     long x,
                                     long xShapeInfo,
                                     long y,
                                     long yShapeInfo,
                                     long result,
                                     long resultShapeInfo,
                                     long dimension, int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    float *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return FloatNativeOpExecutioner::getInstance()->execBroadcast(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformFloat(int opNum,
                                             long dx,
                                             int xStride,
                                             long y,
                                             int yStride,
                                             long result,
                                             int resultStride,
                                             long extraParams, int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    float *yPointer = reinterpret_cast<float *>(y);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FLoatNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xStride,yPointer,yStride,resultPointer,resultStride,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformFloat(int opNum,
                                           long dx,
                                           long xShapeInfo,
                                           long y,
                                           long yShapeInfo,
                                           long result,
                                           long resultShapeInfo,
                                           long extraParams,
                                           int n,
                                           long xIndexes,
                                           long yIndexes,
                                           long resultIndexes){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return FloatNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n,xIndexesPointer,yIndexesPointer,resultIndexesPointer);

}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformFloat(int opNum,
                                           long dx,
                                           long  xShapeInfo,
                                           long y,
                                           long  yShapeInfo,
                                           long result,
                                           long  resultShapeInfo,
                                           long extraParams, int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(int opNum,
                                  long x,
                                  long xShapeInfo,
                                  long extraParams,
                                  long result,
                                  long resultShapeInfo){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(int opNum,
                                  long x,
                                  long xShapeInfo,
                                  long extraParams,
                                  long result,
                                  long resultShapeInfo,
                                  long dimension,int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
double NativeOps::execReduceScalarFloat(int opNum,
                                        long x,
                                        long xShapeInfo,
                                        long extraParams){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execReduceScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Float(int opNum,
                                   long x,
                                   long xShapeInfo,
                                   long extraParamsVals,
                                   long y,
                                   long yShapeInfo,
                                   long result,
                                   long resultShapeInfo){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execReduce3(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
double   NativeOps::execReduce3ScalarFloat(int opNum,
                                           long x,
                                           long xShapeInfo,
                                           long extraParamsVals,
                                           long y,
                                           long yShapeInfo){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execReduce3Scalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Float(int opNum,
                                   long x,
                                   long xShapeInfo,
                                   long extraParamsVals,
                                   long y,
                                   long yShapeInfo,
                                   long result,
                                   long resultShapeInfoBuffer,
                                   long dimension,
                                   int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return FloatNativeOpExecutioner::getInstance()->execReduce3(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer);

}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarFloat(int opNum,
                                  long x,
                                  int xStride,
                                  long result,
                                  int resultStride,
                                  double scalar,
                                  long extraParams,
                                  int n){
    float *xPointer = reinterpret_cast<double *>(dx);
    float *resultPointer = reinterpret_cast<double *>(result);
    float *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xStride,resultPointer,resultStride,scalar,extraParamsPointer,n);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarFloat(int opNum,
                                long x,
                                long xShapeInfo,
                                long result,
                                long resultShapeInfo,
                                float scalar,
                                long extraParams,
                                int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,scalar,extraParamsPointer,n);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarFloat(int opNum,
                                long x,
                                long xShapeInfo,
                                long result,
                                long resultShapeInfo,
                                double scalar,
                                long extraParams,
                                int n,
                                long xIndexes,
                                long resultIndexes){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return FloatNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,scalar,extraParamsPointer,n,xIndexesPointer,resultIndexesPointer);

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execSummaryStatsScalarFloat(int opNum,long x,
                                                long xShapeInfo,
                                                long extraParams){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execSummaryStatsScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsFloat(int opNum,
                                        long x,
                                        long xShapeInfo,
                                        long extraParams,
                                        long result,
                                        long resultShapeInfo){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execSummaryStats(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsFloat(int opNum,long x,
                                        long xShapeInfo,
                                        long extraParams,
                                        long result,
                                        long resultShapeInfoBuffer,
                                        long dimension, int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return FloatNativeOpExecutioner::getInstance()->execSummaryStats(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(int opNum,
                                     long dx,
                                     int xStride,
                                     long result,
                                     int resultStride,
                                     long extraParams, int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xStride,resultPointer,resultStride,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(int opNum,
                                     long dx,
                                     long xShapeInfo,
                                     long result,
                                     long resultShapeInfo,
                                     long extraParams, int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(int opNum,
                                     long dx,
                                     long xShapeInfo,
                                     long result,
                                     long resultShapeInfo,
                                     long extraParams,
                                     int n,
                                     long xIndexes,
                                     long resultIndexes){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return FloatNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n,xIndexesPointer,resultIndexesPointer);

}