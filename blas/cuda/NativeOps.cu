#include "../NativeOps.h"
#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

#include <buffer.h>
#include <shape.h>

#include <reduce3.h>
#include <reduce.h>
#include <indexreduce.h>
#include <pairwise_transform.h>
#include <transform.h>
#include <scalar.h>
#include <broadcasting.h>
#include <summarystatsreduce.h>


dim3 getOptimalDimensions(int n,hipFuncAttributes attributes) {
    // next, get the hipDeviceProp_t object corresponding to the current device
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);

    // we can combine the two to compute a block size
    size_t num_threads = block_size_with_maximum_potential_occupancy(attributes, properties);

    // compute the number of blocks of size num_threads to launch
    size_t num_blocks = n / num_threads;

    // check for partial block at the end
    if(n % num_threads) ++num_blocks;

    return dim3(num_blocks,num_threads,1);
}

nd4j::buffer::Buffer<int> * createScalarBuffer() {
    int *scalarShapeInfo = shape::createScalarShapeInfo();
    nd4j::buffer::Buffer<int> *buff = nd4j::buffer::createBuffer(scalarShapeInfo,shape::shapeInfoLength(2));
    nd4j::buffer::copyDataToGpu(&buff);
    return buff;
}

template <typename T>
class ScalarInfo {
    nd4j::buffer::Buffer<T> *scalarData;
    nd4j::buffer::Buffer<int> scalarDimension;
    nd4j::buffer::Buffer<int> *scalarShapeInfo;
    T finalResult;
public:
    ScalarInfo() {
        scalarShapeInfo = createScalarBuffer();
        T *scalarResult = malloc(sizeof(T));
        scalarData = nd4j::buffer::createBuffer(scalarResult,1);
        nd4j::buffer::copyDataToGpu(&scalarData);
        int *scalarDimensionBuff = malloc(sizeof(int));
        scalarDimension[0] = shape::MAX_DIMENSION;
        scalarDimension = nd4j::buffer::createBuffer(scalarDimensionBuff,1);
        nd4j::buffer::copyDataToGpu(&scalarDimension);
    }

    T getFinalResultFromDevice() {
        nd4j::buffer::copyDataFromGpu(&scalarData);
        return scalarData[0];
    }

    /**
     * Get the device shape information
     * representinga scalar
     */
    int *getDeviceShapeInfo() {
        return scalarShapeInfo->gData;
    }

    /**
     * Get the result pointers
     */
    T *getDevicePointer() {
        return scalarData->gData;
    }

    /**
     * Get the infinite dimension device pointer
     */
    int *getDimensionDevicePointer() {
        return scalarDimension.gData;
    }

    ~ScalarInfo() {
        nd4j::buffer::freeBuffer(&scalarShapeInfo);
        nd4j::buffer::freeBuffer(&scalarData);
        nd4j::buffer::freeBuffer(&scalarDimension);
    }
};

/**
     *
     * @param opNum
     * @param x
     * @param xShapeInfo
     * @param extraParams
     */
double   NativeOps::execIndexReduceScalarDouble(long *extraPointers,int opNum,
                                                long x,
                                                long xShapeInfo,
                                                long extraParams) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    hipFuncAttributes attributes;
    hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(indexReduceDouble));
    int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);
    int n = shape::length(hostXShapeInfo);
    dim3 launchDims = getOptimalDimensions(n,attributes);

    ScalarInfo<double> *scalarInfo = new ScalarInfo<double>();
    indexReduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
                    opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    NULL,
                    scalarInfo->getDevicePointer(),
                    scalarInfo->getDimensionDevicePointer(),
                    scalarInfo->getDimensionDevicePointer(),
                    1,
                    1);
    hipDeviceSynchronize();

    return scalarInfo->getFinalResultFromDevice();

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceDouble(long *extraPointers,int opNum,
                                        long x,
                                        long xShapeInfo,
                                        long extraParams,
                                        long result,
                                        long resultShapeInfoBuffer,
                                        long dimension, int dimensionLength) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execIndexReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);


}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastDouble(long *extraPointers,int opNum,
                                      long x,
                                      long xShapeInfo,
                                      long y,
                                      long yShapeInfo,
                                      long result,
                                      long resultShapeInfo,
                                      long dimension, int dimensionLength){
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execBroadcast(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformDouble(long *extraPointers,int opNum,
                                              long dx,
                                              int xStride,
                                              long y,
                                              int yStride,
                                              long result,
                                              int resultStride,
                                              long extraParams, int n){
    double *xPointer = reinterpret_cast<double *>(dx);
    double *yPointer = reinterpret_cast<double *>(y);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xStride,yPointer,yStride,resultPointer,resultStride,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformDouble(long *extraPointers,int opNum,
                                            long dx,
                                            long xShapeInfo,
                                            long y,
                                            long yShapeInfo,
                                            long result,
                                            long resultShapeInfo,
                                            long extraParams,
                                            int n,
                                            long xIndexes,
                                            long yIndexes,
                                            long resultIndexes){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return DoubleNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n,xIndexesPointer,yIndexesPointer,resultIndexesPointer);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformDouble(long *extraPointers,int opNum,
                                            long dx,
                                            long  xShapeInfo,
                                            long y,
                                            long  yShapeInfo,
                                            long result,
                                            long  resultShapeInfo,
                                            long extraParams, int n) {
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(long *extraPointers,int opNum,
                                   long x,
                                   long xShapeInfo,
                                   long extraParams,
                                   long result,
                                   long resultShapeInfo){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(long *extraPointers,int opNum,
                                   long x,
                                   long xShapeInfo,
                                   long extraParams,
                                   long result,
                                   long resultShapeInfo,
                                   long dimension,int dimensionLength) {
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
double NativeOps::execReduceScalarDouble(long *extraPointers,int opNum,
                                         long x,
                                         long xShapeInfo,
                                         long extraParams){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execReduceScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Double(long *extraPointers,int opNum,
                                    long x,
                                    long xShapeInfo,
                                    long extraParamsVals,
                                    long y,
                                    long yShapeInfo,
                                    long result,
                                    long resultShapeInfo){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execReduce3(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
double   NativeOps::execReduce3ScalarDouble(long *extraPointers,int opNum,
                                            long x,
                                            long xShapeInfo,
                                            long extraParamsVals,
                                            long y,
                                            long yShapeInfo){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execReduce3Scalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Double(long *extraPointers,int opNum,
                                    long x,
                                    long xShapeInfo,
                                    long extraParamsVals,
                                    long y,
                                    long yShapeInfo,
                                    long result,
                                    long resultShapeInfoBuffer,
                                    long dimension,
                                    int dimensionLength){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execReduce3(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer);

}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarDouble(long *extraPointers,int opNum,
                                   long x,
                                   int xStride,
                                   long result,
                                   int resultStride,
                                   double scalar,
                                   long extraParams,
                                   int n) {
    double *xPointer = reinterpret_cast<double *>(dx);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xStride,resultPointer,resultStride,scalar,extraParamsPointer,n);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarDouble(long *extraPointers,int opNum,
                                 long x,
                                 long xShapeInfo,
                                 long result,
                                 long resultShapeInfo,
                                 double scalar,
                                 long extraParams,
                                 int n){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,scalar,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarDouble(long *extraPointers,int opNum,
                                 long x,
                                 long xShapeInfo,
                                 long result,
                                 long resultShapeInfo,
                                 double scalar,
                                 long extraParams,
                                 int n,
                                 long xIndexes,
                                 long resultIndexes){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return DoubleNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,scalar,extraParamsPointer,n,xIndexesPointer,resultIndexesPointer);

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execSummaryStatsScalarDouble(long *extraPointers,int opNum,long x,
                                                 long xShapeInfo,
                                                 long extraParams){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execSummaryStatsScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsDouble(long *extraPointers,int opNum,
                                         long x,
                                         long xShapeInfo,
                                         long extraParams,
                                         long result,
                                         long resultShapeInfo){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execSummaryStats(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsDouble(long *extraPointers,int opNum,long x,
                                         long xShapeInfo,
                                         long extraParams,
                                         long result,
                                         long resultShapeInfoBuffer,
                                         long dimension, int dimensionLength){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execSummaryStats(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(long *extraPointers,int opNum,
                                      long dx,
                                      int xStride,
                                      long result,
                                      int resultStride,
                                      long extraParams, int n){
    double *xPointer = reinterpret_cast<double *>(dx);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return DoubleNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xStride,resultPointer,resultStride,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(long *extraPointers,int opNum,
                                      long dx,
                                      long xShapeInfo,
                                      long result,
                                      long resultShapeInfo,
                                      long extraParams, int n){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    return DoubleNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(long *extraPointers,int opNum,
                                      long dx,
                                      long xShapeInfo,
                                      long result,
                                      long resultShapeInfo,
                                      long extraParams,
                                      int n,
                                      long xIndexes,
                                      long resultIndexes){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return DoubleNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n,xIndexesPointer,resultIndexesPointer);

}

/**
*
* @param opNum
* @param x
* @param xShapeInfo
* @param extraParams
*/
double   NativeOps::execIndexReduceScalarFloat(long *extraPointers,int opNum,
                                               long x,
                                               long xShapeInfo,
                                               long extraParams){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execIndexReduceScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceFloat(long *extraPointers,int opNum,
                                       long x,
                                       long xShapeInfo,
                                       long extraParams,
                                       long result,
                                       long resultShapeInfoBuffer,
                                       long dimension, int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return FloatNativeOpExecutioner::getInstance()->execIndexReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);


}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastFloat(long *extraPointers,int opNum,
                                     long x,
                                     long xShapeInfo,
                                     long y,
                                     long yShapeInfo,
                                     long result,
                                     long resultShapeInfo,
                                     long dimension, int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    float *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return FloatNativeOpExecutioner::getInstance()->execBroadcast(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformFloat(long *extraPointers,int opNum,
                                             long dx,
                                             int xStride,
                                             long y,
                                             int yStride,
                                             long result,
                                             int resultStride,
                                             long extraParams, int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    float *yPointer = reinterpret_cast<float *>(y);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FLoatNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xStride,yPointer,yStride,resultPointer,resultStride,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformFloat(long *extraPointers,int opNum,
                                           long dx,
                                           long xShapeInfo,
                                           long y,
                                           long yShapeInfo,
                                           long result,
                                           long resultShapeInfo,
                                           long extraParams,
                                           int n,
                                           long xIndexes,
                                           long yIndexes,
                                           long resultIndexes){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return FloatNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n,xIndexesPointer,yIndexesPointer,resultIndexesPointer);

}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformFloat(long *extraPointers,int opNum,
                                           long dx,
                                           long  xShapeInfo,
                                           long y,
                                           long  yShapeInfo,
                                           long result,
                                           long  resultShapeInfo,
                                           long extraParams, int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execPairwiseTransform(opNum,xPointer,xShapeInfoPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(long *extraPointers,int opNum,
                                  long x,
                                  long xShapeInfo,
                                  long extraParams,
                                  long result,
                                  long resultShapeInfo){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(long *extraPointers,int opNum,
                                  long x,
                                  long xShapeInfo,
                                  long extraParams,
                                  long result,
                                  long resultShapeInfo,
                                  long dimension,int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return DoubleNativeOpExecutioner::getInstance()->execReduce(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
double NativeOps::execReduceScalarFloat(long *extraPointers,int opNum,
                                        long x,
                                        long xShapeInfo,
                                        long extraParams){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execReduceScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Float(long *extraPointers,int opNum,
                                   long x,
                                   long xShapeInfo,
                                   long extraParamsVals,
                                   long y,
                                   long yShapeInfo,
                                   long result,
                                   long resultShapeInfo){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execReduce3(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
double   NativeOps::execReduce3ScalarFloat(long *extraPointers,int opNum,
                                           long x,
                                           long xShapeInfo,
                                           long extraParamsVals,
                                           long y,
                                           long yShapeInfo){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execReduce3Scalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Float(long *extraPointers,int opNum,
                                   long x,
                                   long xShapeInfo,
                                   long extraParamsVals,
                                   long y,
                                   long yShapeInfo,
                                   long result,
                                   long resultShapeInfoBuffer,
                                   long dimension,
                                   int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return FloatNativeOpExecutioner::getInstance()->execReduce3(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,yPointer,yShapeInfoPointer,resultPointer,resultShapeInfoPointer);

}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarFloat(long *extraPointers,int opNum,
                                  long x,
                                  int xStride,
                                  long result,
                                  int resultStride,
                                  double scalar,
                                  long extraParams,
                                  int n){
    float *xPointer = reinterpret_cast<double *>(dx);
    float *resultPointer = reinterpret_cast<double *>(result);
    float *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xStride,resultPointer,resultStride,scalar,extraParamsPointer,n);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarFloat(long *extraPointers,int opNum,
                                long x,
                                long xShapeInfo,
                                long result,
                                long resultShapeInfo,
                                float scalar,
                                long extraParams,
                                int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,scalar,extraParamsPointer,n);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarFloat(long *extraPointers,int opNum,
                                long x,
                                long xShapeInfo,
                                long result,
                                long resultShapeInfo,
                                double scalar,
                                long extraParams,
                                int n,
                                long xIndexes,
                                long resultIndexes){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return FloatNativeOpExecutioner::getInstance()->execScalar(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,scalar,extraParamsPointer,n,xIndexesPointer,resultIndexesPointer);

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execSummaryStatsScalarFloat(long *extraPointers,int opNum,long x,
                                                long xShapeInfo,
                                                long extraParams){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execSummaryStatsScalar(opNum,xPointer,xShapeInfoPointer,extraParamsPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsFloat(long *extraPointers,int opNum,
                                        long x,
                                        long xShapeInfo,
                                        long extraParams,
                                        long result,
                                        long resultShapeInfo){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execSummaryStats(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer);
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsFloat(long *extraPointers,int opNum,long x,
                                        long xShapeInfo,
                                        long extraParams,
                                        long result,
                                        long resultShapeInfoBuffer,
                                        long dimension, int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    return FloatNativeOpExecutioner::getInstance()->execSummaryStats(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultShapeInfoPointer,dimensionPointer,dimensionLength);

}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(long *extraPointers,int opNum,
                                     long dx,
                                     int xStride,
                                     long result,
                                     int resultStride,
                                     long extraParams, int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    return FloatNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xStride,resultPointer,resultStride,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(long *extraPointers,int opNum,
                                     long dx,
                                     long xShapeInfo,
                                     long result,
                                     long resultShapeInfo,
                                     long extraParams, int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    return FloatNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(long *extraPointers,int opNum,
                                     long dx,
                                     long xShapeInfo,
                                     long result,
                                     long resultShapeInfo,
                                     long extraParams,
                                     int n,
                                     long xIndexes,
                                     long resultIndexes){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    return FloatNativeOpExecutioner::getInstance()->execTransform(opNum,xPointer,xShapeInfoPointer,resultPointer,resultShapeInfoPointer,extraParamsPointer,n,xIndexesPointer,resultIndexesPointer);

}