#include "../NativeOps.h"
#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

#include <buffer.h>
#include <shape.h>

#include <reduce3.h>
#include <reduce.h>
#include <indexreduce.h>
#include <pairwise_transform.h>
#include <transform.h>
#include <scalar.h>
#include <broadcasting.h>
#include <summarystatsreduce.h>
#include <thread>
#include <map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <pointercast.h>

template <typename T>
dim3 getOptimalDimensions(int n,hipFuncAttributes attributes) {
    // next, get the hipDeviceProp_t object corresponding to the current device
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);

    // we can combine the two to compute a block size
    int num_threads = block_size_with_maximum_potential_occupancy(attributes, properties);

    // compute the number of blocks of size num_threads to launch
    int num_blocks = n / num_threads;

    // check for partial block at the end
    if(n % num_threads) ++num_blocks;

    return dim3(num_blocks,num_threads,num_threads * sizeof(T));
}

/**
 * Returns optimal launch parameters
 * given the extra pointers passed in.
 * The extra pointer should be
 * the host pointer for the shape information
 * associated with the data.
 * From there it is used to obtain the length
 * from which we can derive the optimal launch parameters.
 *
 */
template <typename T>
dim3 getOptimalLaunchParameters(Nd4jPointer *extraPointers) {
    hipFuncAttributes attributes;
    hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(indexReduceDouble));
    int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);
    int n = shape::length(hostXShapeInfo);
    dim3 launchDims = getOptimalDimensions<T>(n,attributes);
    return launchDims;
}

nd4j::buffer::Buffer<int> * createScalarBuffer() {
    int *scalarShapeInfo = shape::createScalarShapeInfo();
    nd4j::buffer::Buffer<int> *buff = nd4j::buffer::createBuffer(scalarShapeInfo,shape::shapeInfoLength(2));
    nd4j::buffer::copyDataToGpu(&buff);
    return buff;
}


class ScalarShapeInformation {
private:
    nd4j::buffer::Buffer<int> *scalarDimension;
    nd4j::buffer::Buffer<int> *scalarShapeInfo;
    std::thread::id threadId;

public:
    ScalarShapeInformation() {
        int *scalarDimensionBuff = (int *) malloc(sizeof(int));
        scalarDimensionBuff[0] = shape::MAX_DIMENSION;
        scalarDimension = nd4j::buffer::createBuffer(scalarDimensionBuff,1);
        scalarShapeInfo = createScalarBuffer();
        threadId = std::this_thread::get_id();

    }
    ~ScalarShapeInformation() {
        nd4j::buffer::freeBuffer(&scalarShapeInfo);
        nd4j::buffer::freeBuffer(&scalarDimension);
    }


    int *getShapeInfoHostPointer() {
        return scalarShapeInfo->data;
    }

    int * getShapeInfoGpuPointer() {
        return scalarShapeInfo->gData;
    }

    int * getDimensionHostPointer() {
        return scalarDimension->data;
    }

    int  * getDimensionGpuPointer() {
        return scalarDimension->gData;
    }

};





template <typename T>
class ScalarInfo {
    nd4j::buffer::Buffer<T> *scalarData;
#ifdef R__WIN32
    static thread_local ScalarShapeInformation shapeInfo;
#else
    static  ScalarShapeInformation shapeInfo;

#endif
    T finalResult;
public:
    ScalarInfo() {
        T *scalarResult = (T*)malloc(sizeof(T));
        scalarData = nd4j::buffer::createBuffer(scalarResult,1);
        nd4j::buffer::copyDataToGpu(&scalarData);
    }

    T getFinalResultFromDevice() {
        nd4j::buffer::copyDataFromGpu(&scalarData);
        return scalarData->data[0];
    }

    /**
     * Get the device shape information
     * representing a scalar
     */
    int *getDeviceShapeInfo() {
        return shapeInfo.getShapeInfoGpuPointer();
    }

    /**
     * Get the result pointers
     */
    T *getDevicePointer() {
        return scalarData->gData;
    }

    /**
     * Get the infinite dimension device pointer
     */
    int *getDimensionDevicePointer() {
        return shapeInfo.getDimensionGpuPointer();
    }

    ~ScalarInfo() {
        nd4j::buffer::freeBuffer(&scalarData);
    }
};


/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execIndexReduceScalarDouble(Nd4jPointer *extraPointers,int opNum,
                                                Nd4jPointer x,
                                                Nd4jPointer xShapeInfo,
                                                Nd4jPointer extraParams) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    ScalarInfo<double> *scalarInfo = new ScalarInfo<double>();

    indexReduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    scalarInfo->getDevicePointer(),
                    scalarInfo->getDeviceShapeInfo(),
                    scalarInfo->getDimensionDevicePointer(),
                    1,
                    1);
    hipDeviceSynchronize();

    double result =  scalarInfo->getFinalResultFromDevice();
    delete scalarInfo;
    return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfoBuffer,
        Nd4jPointer dimension, int dimensionLength) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);

    indexReduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    dimensionPointer,
                    dimensionLength,
                    1);


}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastDouble(Nd4jPointer *extraPointers,
                                      int opNum,
                                      Nd4jPointer x,
                                      Nd4jPointer xShapeInfo,
                                      Nd4jPointer y,
                                      Nd4jPointer yShapeInfo,
                                      Nd4jPointer result,
                                      Nd4jPointer resultShapeInfo,
                                      Nd4jPointer dimension, int dimensionLength){
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);

    broadcastDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    yPointer,
                    yShapeInfoPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    dimensionPointer,
                    dimensionLength);

}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        int xStride,
        Nd4jPointer y,
        int yStride,
        Nd4jPointer result,
        int resultStride,
        Nd4jPointer extraParams, int n) {
    double *xPointer = reinterpret_cast<double *>(dx);
    double *yPointer = reinterpret_cast<double *>(y);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    pairWiseTransformStridedDouble<<<launchDims.x,launchDims.y,launchDims.z>>>
                                                               (
                                                                       opNum,
                                                                               n,
                                                                               xPointer,
                                                                               yPointer,
                                                                               xStride,
                                                                               yStride,
                                                                               extraParamsPointer,
                                                                               resultPointer,
                                                                               resultStride);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        Nd4jPointer xShapeInfo,
        Nd4jPointer y,
        Nd4jPointer yShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer xIndexes,
        Nd4jPointer yIndexes,
        Nd4jPointer resultIndexes) {
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    pairWiseTransformDoubleIndex <<<launchDims.x, launchDims.y, launchDims.z>>>(
            opNum,
                    xPointer,
                    yPointer,
                    extraParamsPointer,
                    resultPointer,
                    xShapeInfoPointer,
                    yShapeInfoPointer,
                    resultShapeInfoPointer,
                    xIndexesPointer,
                    yIndexesPointer,
                    resultIndexesPointer);
}
/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        Nd4jPointer  xShapeInfo,
        Nd4jPointer y,
        Nd4jPointer  yShapeInfo,
        Nd4jPointer result,
        Nd4jPointer  resultShapeInfo,
        Nd4jPointer extraParams) {
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    pairWiseTransformDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    yPointer,
                    extraParamsPointer,
                    resultPointer,
                    xShapeInfoPointer,
                    yShapeInfoPointer,
                    resultShapeInfoPointer);


}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    ScalarInfo<double> *scalarInfo = new ScalarInfo<double>();

    reduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer
                    ,extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    scalarInfo->getDimensionDevicePointer(),
                    1,
                    1);

    delete scalarInfo;


}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(
        Nd4jPointer *extraPointers
        ,int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        Nd4jPointer dimension,
        int dimensionLength) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    reduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer
                    ,extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    dimensionPointer,
                    dimensionLength,
                    1);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
double NativeOps::execReduceScalarDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams){
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    ScalarInfo<double> *scalarInfo = new ScalarInfo<double>();
    reduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer
                    ,extraParamsPointer,
                    scalarInfo->getDevicePointer(),
                    scalarInfo->getDeviceShapeInfo(),
                    scalarInfo->getDimensionDevicePointer(),
                    1,
                    1);
    hipDeviceSynchronize();
    double result =  scalarInfo->getFinalResultFromDevice();
    delete scalarInfo;
    return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Double(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParamsVals,
        Nd4jPointer y,
        Nd4jPointer yShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    ScalarInfo<double> *scalarInfo = new ScalarInfo<double>();
    reduce3Double<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,
            xPointer,
            xShapeInfoPointer,
            yPointer,
            yShapeInfoPointer,
            extraParamsPointer,
            resultPointer,
            resultShapeInfoPointer,
            scalarInfo->getDimensionDevicePointer(),
            1,
            1);
    delete scalarInfo;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
double   NativeOps::execReduce3ScalarDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParamsVals,
        Nd4jPointer y,
        Nd4jPointer yShapeInfo){
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    ScalarInfo<double> *scalarInfo = new ScalarInfo<double>();
    reduce3Double<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,
            xPointer,
            xShapeInfoPointer,
            yPointer,
            yShapeInfoPointer,
            extraParamsPointer,
            scalarInfo->getDevicePointer(),
            scalarInfo->getDeviceShapeInfo(),
            scalarInfo->getDimensionDevicePointer(),
            1,
            1);
    hipDeviceSynchronize();
    double result  = scalarInfo->getFinalResultFromDevice();
    delete scalarInfo;
    return result;
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Double(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParamsVals,
        Nd4jPointer y,
        Nd4jPointer yShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfoBuffer,
        Nd4jPointer dimension,
        int dimensionLength){
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *yPointer = reinterpret_cast<double *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    reduce3Double<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,
            xPointer,
            xShapeInfoPointer,
            yPointer,
            yShapeInfoPointer,
            extraParamsPointer,
            resultPointer,
            resultShapeInfoPointer,
            dimensionPointer,
            dimensionLength,
            1);

}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        int xStride,
        Nd4jPointer result,
        int resultStride,
        double scalar,
        Nd4jPointer extraParams,
        int n) {
    double *xPointer = reinterpret_cast<double *>(x);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    scalarDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    n,
                    scalar,
                    xPointer,
                    xStride,
                    extraParamsPointer,
                    resultPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        double scalar,
        Nd4jPointer extraParams){
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    scalarDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    scalar,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    resultPointer);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        double scalar,
        Nd4jPointer extraParams,
        int n,
        Nd4jPointer xIndexes,
        Nd4jPointer resultIndexes){
    double *xPointer = reinterpret_cast<double *>(x);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    scalarDoubleIndexes<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum, n,scalar,xPointer,extraParamsPointer,resultPointer,resultIndexesPointer);


}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execSummaryStatsScalarDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,bool biasCorrected){
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    ScalarInfo<double> *scalarShapeInformation = new ScalarInfo<double>();
    summaryStatsReduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,
            xPointer,
            xShapeInfoPointer,
            extraParamsPointer,
            scalarShapeInformation->getDevicePointer(),
            scalarShapeInformation->getDeviceShapeInfo(),
            scalarShapeInformation->getDimensionDevicePointer(),
            1,
            1,biasCorrected);
    hipDeviceSynchronize();
    double result = scalarShapeInformation->getFinalResultFromDevice();
    delete scalarShapeInformation;
    return result;

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,bool biasCorrected) {
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    ScalarInfo<double> *scalarShapeInformation = new ScalarInfo<double>();
    summaryStatsReduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    scalarShapeInformation->getDimensionDevicePointer(),
                    1,
                    1,biasCorrected);
    delete scalarShapeInformation;
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfoBuffer,
        Nd4jPointer dimension, int dimensionLength,bool biasCorrected){
    double *xPointer = reinterpret_cast<double *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    summaryStatsReduceDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    dimensionPointer,
                    dimensionLength,
                    1,biasCorrected);

}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        int xStride,
        Nd4jPointer result,
        int resultStride,
        Nd4jPointer extraParams,
        int n) {
    double *xPointer = reinterpret_cast<double *>(dx);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    transformDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,n,xPointer,xStride,extraParamsPointer,resultPointer);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        Nd4jPointer xShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        Nd4jPointer extraParams){
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    transformDouble<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        Nd4jPointer xShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer xIndexes,
        Nd4jPointer resultIndexes) {
    double *xPointer = reinterpret_cast<double *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    double *resultPointer = reinterpret_cast<double *>(result);
    double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    dim3 launchDims = getOptimalLaunchParameters<double>(extraPointers);
    transformDoubleIndexes<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultIndexesPointer);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
float   NativeOps::execIndexReduceScalarFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    ScalarInfo<float> *scalarInfo = new ScalarInfo<float>();

    indexReduceFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    scalarInfo->getDevicePointer(),
                    scalarInfo->getDeviceShapeInfo(),
                    scalarInfo->getDimensionDevicePointer(),
                    1,
                    1);
    hipDeviceSynchronize();

    float result =  scalarInfo->getFinalResultFromDevice();
    delete scalarInfo;
    return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfoBuffer,
        Nd4jPointer dimension,
        int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    indexReduceFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    dimensionPointer,
                    dimensionLength,
                    1);

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer y,
        Nd4jPointer yShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        Nd4jPointer dimension, int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);

    broadcastFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    yPointer,
                    yShapeInfoPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    dimensionPointer,
                    dimensionLength);
}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        int xStride,
        Nd4jPointer y,
        int yStride,
        Nd4jPointer result,
        int resultStride,
        Nd4jPointer extraParams, int n){
    float *xPointer = reinterpret_cast<float *>(dx);
    float *yPointer = reinterpret_cast<float *>(y);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    pairWiseTransformStridedFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,n,xPointer,yPointer,xStride,yStride,extraParamsPointer,resultPointer,resultStride);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        Nd4jPointer xShapeInfo,
        Nd4jPointer y,
        Nd4jPointer yShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer xIndexes,
        Nd4jPointer yIndexes,
        Nd4jPointer resultIndexes){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
    int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    pairWiseTransformFloatIndex<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    yPointer,
                    extraParamsPointer,
                    resultPointer,
                    xShapeInfoPointer,
                    yShapeInfoPointer,
                    resultShapeInfoPointer,
                    xIndexesPointer,
                    yIndexesPointer,
                    resultIndexesPointer);
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        Nd4jPointer  xShapeInfo,
        Nd4jPointer y,
        Nd4jPointer  yShapeInfo,
        Nd4jPointer result,
        Nd4jPointer  resultShapeInfo,
        Nd4jPointer extraParams){
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    pairWiseTransformFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    yPointer,
                    extraParamsPointer,
                    resultPointer,
                    xShapeInfoPointer,
                    yShapeInfoPointer,
                    resultShapeInfoPointer);
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo) {
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    ScalarInfo<float> *scalarInfo = new ScalarInfo<float>();
    reduceFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer
                    ,extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    scalarInfo->getDimensionDevicePointer(),
                    1,
                    1);

    delete scalarInfo;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        Nd4jPointer dimension,int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    reduceFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer
                    ,extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    dimensionPointer,
                    dimensionLength,
                    1);


}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
float NativeOps::execReduceScalarFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    ScalarInfo<float> *scalarInfo = new ScalarInfo<float>();
    reduceFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer
                    ,extraParamsPointer,
                    scalarInfo->getDevicePointer(),
                    scalarInfo->getDeviceShapeInfo(),
                    scalarInfo->getDimensionDevicePointer(),
                    1,
                    1);
    hipDeviceSynchronize();
    double result =  scalarInfo->getFinalResultFromDevice();
    delete scalarInfo;
    return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Float(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParamsVals,
        Nd4jPointer y,
        Nd4jPointer yShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    ScalarInfo<float> *scalarInfo = new ScalarInfo<float>();
    reduce3Float<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,
            xPointer,
            xShapeInfoPointer,
            yPointer,
            yShapeInfoPointer,
            extraParamsPointer,
            resultPointer,
            resultShapeInfoPointer,
            scalarInfo->getDimensionDevicePointer(),
            1,
            1);
    delete scalarInfo;

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
float   NativeOps::execReduce3ScalarFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParamsVals,
        Nd4jPointer y,
        Nd4jPointer yShapeInfo) {
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    ScalarInfo<float> *scalarInfo = new ScalarInfo<float>();
    reduce3Float<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,
            xPointer,
            xShapeInfoPointer,
            yPointer,
            yShapeInfoPointer,
            extraParamsPointer,
            scalarInfo->getDevicePointer(),
            scalarInfo->getDeviceShapeInfo(),
            scalarInfo->getDimensionDevicePointer(),
            1,
            1);
    hipDeviceSynchronize();
    double result  = scalarInfo->getFinalResultFromDevice();
    delete scalarInfo;
    return result;

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Float(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParamsVals,
        Nd4jPointer y,
        Nd4jPointer yShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfoBuffer,
        Nd4jPointer dimension,
        int dimensionLength){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *yPointer = reinterpret_cast<float *>(y);
    int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    reduce3Float<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,
            xPointer,
            xShapeInfoPointer,
            yPointer,
            yShapeInfoPointer,
            extraParamsPointer,
            resultPointer,
            resultShapeInfoPointer,
            dimensionPointer,
            dimensionLength,
            1);

}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        int xStride,
        Nd4jPointer result,
        int resultStride,
        double scalar,
        Nd4jPointer extraParams,
        int n){
    float *xPointer = reinterpret_cast<float *>(x);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    scalarFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    n,
                    scalar,
                    xPointer,
                    xStride,
                    extraParamsPointer,
                    resultPointer);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        float scalar,
        Nd4jPointer extraParams){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    scalarFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    scalar,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    resultPointer);

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        double scalar,
        Nd4jPointer extraParams,
        Nd4jPointer xIndexes,
        Nd4jPointer resultIndexes){
    float *xPointer = reinterpret_cast<float *>(x);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    int *hostShapeInfo = reinterpret_cast<int *>(extraPointers[0]);
    int n = shape::length(hostShapeInfo);
    scalarFloatIndexes<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    n,
                    scalar,
                    xPointer,
                    extraParamsPointer,
                    resultPointer,
                    resultIndexesPointer);

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
float   NativeOps::execSummaryStatsScalarFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,bool biasCorrected){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    ScalarInfo<float> *scalarShapeInformation = new ScalarInfo<float>();
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    summaryStatsReduceFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,
            xPointer,
            xShapeInfoPointer,
            extraParamsPointer,
            xPointer,
            xShapeInfoPointer,
            scalarShapeInformation->getDimensionDevicePointer(),
            1,
            1,biasCorrected);
    hipDeviceSynchronize();
    float result = scalarShapeInformation->getFinalResultFromDevice();
    delete scalarShapeInformation;
    return result;
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,bool biasCorrected){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    ScalarInfo<float> *scalarShapeInformation = new ScalarInfo<float>();
    summaryStatsReduceFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    scalarShapeInformation->getDimensionDevicePointer(),
                    1,
                    1,biasCorrected);
    delete scalarShapeInformation;
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer x,
        Nd4jPointer xShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfoBuffer,
        Nd4jPointer dimension,
        int dimensionLength,bool biasCorrected){
    float *xPointer = reinterpret_cast<float *>(x);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *dimensionPointer = reinterpret_cast<int *>(dimension);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    summaryStatsReduceFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(
            opNum,
                    xPointer,
                    xShapeInfoPointer,
                    extraParamsPointer,
                    resultPointer,
                    resultShapeInfoPointer,
                    dimensionPointer,
                    dimensionLength,
                    1,biasCorrected);

}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        int xStride,
        Nd4jPointer result,
        int resultStride,
        Nd4jPointer extraParams,
        int n) {
    float *xPointer = reinterpret_cast<float *>(dx);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    transformFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,n,xPointer,xStride,extraParamsPointer,resultPointer);

}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(Nd4jPointer *extraPointers,int opNum,
                                     Nd4jPointer dx,
                                     Nd4jPointer xShapeInfo,
                                     Nd4jPointer result,
                                     Nd4jPointer resultShapeInfo,
                                     Nd4jPointer extraParams) {
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    transformFloat<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer);

}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(
        Nd4jPointer *extraPointers,
        int opNum,
        Nd4jPointer dx,
        Nd4jPointer xShapeInfo,
        Nd4jPointer result,
        Nd4jPointer resultShapeInfo,
        Nd4jPointer extraParams,
        Nd4jPointer xIndexes,
        Nd4jPointer resultIndexes) {
    float *xPointer = reinterpret_cast<float *>(dx);
    int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
    float *resultPointer = reinterpret_cast<float *>(result);
    float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
    int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
    dim3 launchDims = getOptimalLaunchParameters<float>(extraPointers);
    transformFloatIndexes<<<launchDims.x,launchDims.y,launchDims.z>>>(opNum,xPointer,xShapeInfoPointer,extraParamsPointer,resultPointer,resultIndexesPointer);


}
