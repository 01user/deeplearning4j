#include "hip/hip_runtime.h"
#include "../NativeOps.h"
#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

#include <buffer.h>
#include <shape.h>

#include <hipblas.h>
#include <reduce3.h>
#include <reduce.h>
#include <indexreduce.h>
#include <pairwise_transform.h>
#include <transform.h>
#include <scalar.h>
#include <broadcasting.h>
#include <summarystatsreduce.h>
#include <thread>
#include <map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <pointercast.h>
#include <stdio.h>

hipDeviceProp_t *deviceProperties;
hipFuncAttributes *funcAttributes = new hipFuncAttributes[28];
int blockLimit = 128;
int maxThreads = -1;
bool debug = false;

template <typename T>
dim3 getOptimalDimensions(Nd4jIndex n,hipFuncAttributes attributes, hipDeviceProp_t properties) {

	// we can combine the two to compute a block size
	int num_threads = block_size_with_maximum_potential_occupancy(attributes, properties);

	// no real sense launching more threads, then number of elements we have
	if (num_threads > n) num_threads = n;

	if (maxThreads > 0 && num_threads > maxThreads) num_threads = maxThreads;

	// compute the number of blocks of size num_threads to launch
	int num_blocks = n / num_threads;

	// check for partial block at the end

	if (num_blocks > blockLimit) num_blocks = blockLimit;

	if (num_blocks < 4 && n > 128) {
		num_blocks = 4;
		num_threads = n / num_blocks;
	}

	if (num_threads >= 768) {
		num_blocks = num_blocks * 2;
		num_threads = num_threads / 2;
	}

	if(n % num_threads && num_blocks < blockLimit) ++num_blocks;
    //(num_threads * sizeof(T)) + attributes.sharedSizeBytes);
	return dim3(num_blocks,num_threads, 5000);
}

/**
 * Returns optimal launch parameters
 * given the extra pointers passed in.
 * The extra pointer should be
 * the host pointer for the shape information
 * associated with the data.
 * From there it is used to obtain the length
 * from which we can derive the optimal launch parameters.
 *
 */
template <typename T>
dim3 getOptimalLaunchParameters(Nd4jPointer *extraPointers, hipFuncAttributes attributes, hipDeviceProp_t properties) {
	int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);
	Nd4jIndex n = shape::length(hostXShapeInfo);

	dim3 launchDims = getOptimalDimensions<T>(n,attributes, properties);

	if (debug)
		printf("Params: gridSize: [%i], blockSize: [%i], shMem: [%i], problemLength: [%i], totalThreads:[%i]\n", launchDims.x, launchDims.y, launchDims.z, n, (launchDims.x * launchDims.y));

	return launchDims;
}


nd4j::buffer::Buffer<int> * createScalarBuffer(hipStream_t stream) {
	int *scalarShapeInfo = shape::createScalarShapeInfo();
	nd4j::buffer::Buffer<int> *buff = nd4j::buffer::createBuffer(scalarShapeInfo,shape::shapeInfoLength(2), stream);
	nd4j::buffer::copyDataToGpu(&buff, stream);
	return buff;
}


class ScalarShapeInformation {
private:
	nd4j::buffer::Buffer<int> *scalarDimension;
	nd4j::buffer::Buffer<int> *scalarShapeInfo;
	std::thread::id threadId;

public:
	ScalarShapeInformation(hipStream_t stream) {
		int *scalarDimensionBuff = (int *) malloc(sizeof(int));
		scalarDimensionBuff[0] = MAX_DIMENSION;
		scalarDimension = nd4j::buffer::createBuffer(scalarDimensionBuff,1, stream);
		scalarShapeInfo = createScalarBuffer(stream);
		threadId = std::this_thread::get_id();

	}
	~ScalarShapeInformation() {
		nd4j::buffer::freeBuffer(&scalarShapeInfo);
		nd4j::buffer::freeBuffer(&scalarDimension);
	}


	int *getShapeInfoHostPointer() {
		return scalarShapeInfo->data;
	}

	int * getShapeInfoGpuPointer() {
		return scalarShapeInfo->gData;
	}

	int * getDimensionHostPointer() {
		return scalarDimension->data;
	}

	int  * getDimensionGpuPointer() {
		return scalarDimension->gData;
	}

};





template <typename T>
class ScalarInfo {
	nd4j::buffer::Buffer<T> *scalarData;
	ScalarShapeInformation *shapeInfo;
	T finalResult;
	hipStream_t streamRef;
public:
	ScalarInfo(hipStream_t stream) {
		T *scalarResult = (T*)malloc(sizeof(T));
		shapeInfo = new ScalarShapeInformation(stream);
		scalarData = nd4j::buffer::createBuffer(scalarResult,1, stream);
		streamRef = stream;
		nd4j::buffer::copyDataToGpu(&scalarData, stream);
	}

	T getFinalResultFromDevice() {
		nd4j::buffer::copyDataFromGpu(&scalarData, streamRef);
		return scalarData->data[0];
	}

	/**
	 * Get the device shape information
	 * representing a scalar
	 */
	 int *getDeviceShapeInfo() {
		return shapeInfo->getShapeInfoGpuPointer();
	}

	/**
	 * Get the result pointers
	 */
	 T *getDevicePointer() {
		 return scalarData->gData;
	 }

	 /**
	  * Get the infinite dimension device pointer
	  */
	 int *getDimensionDevicePointer() {
		 return shapeInfo->getDimensionGpuPointer();
	 }

	 ~ScalarInfo() {
		 nd4j::buffer::freeBuffer(&scalarData);
		 delete shapeInfo;
	 }
};


/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execIndexReduceScalarDouble(Nd4jPointer *extraPointers,int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[27], deviceProperties[(int) extraPointers[2]]);

	double *resultPointer = reinterpret_cast<double *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	indexReduceDouble<<<launchDims.x,launchDims.y,launchDims.z * 4, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1, allocationPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result = resultPointer[0];
	return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension, int dimensionLength) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);


	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[27], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	indexReduceDouble<<<launchDims.x,launchDims.y,launchDims.z * 2, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastDouble(Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer dimension, int dimensionLength){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[26], deviceProperties[(int) extraPointers[2]]);

	broadcastDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		int xStride,
		Nd4jPointer y,
		int yStride,
		Nd4jPointer result,
		int resultStride,
		Nd4jPointer extraParams, Nd4jIndex n) {
	double *xPointer = reinterpret_cast<double *>(dx);
	double *yPointer = reinterpret_cast<double *>(y);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[25], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	pairWiseTransformStridedDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>> (
			opNum,
			n,
			xPointer,
			yPointer,
			xStride,
			yStride,
			extraParamsPointer,
			resultPointer,
			resultStride, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer yIndexes,
		Nd4jPointer resultIndexes) {
	double *xPointer = reinterpret_cast<double *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
	int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[24], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	pairWiseTransformDoubleIndex <<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(
			opNum,
			xPointer,
			yPointer,
			extraParamsPointer,
			resultPointer,
			xShapeInfoPointer,
			yShapeInfoPointer,
			resultShapeInfoPointer,
			xIndexesPointer,
			yIndexesPointer,
			resultIndexesPointer, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer  xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer  yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer  resultShapeInfo,
		Nd4jPointer extraParams) {
	double *xPointer = reinterpret_cast<double *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[23], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	pairWiseTransformDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			yPointer,
			extraParamsPointer,
			resultPointer,
			xShapeInfoPointer,
			yShapeInfoPointer,
			resultShapeInfoPointer, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[22], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	reduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer
			,extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			nullptr,
			1,
			1,
			allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(
		Nd4jPointer *extraPointers
		,int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer dimension,
		int dimensionLength) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[22], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	reduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer
			,extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1,
			allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
double NativeOps::execReduceScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[22], deviceProperties[(int) extraPointers[2]]);

	double *resultPointer = reinterpret_cast<double *>(extraPointers[5]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	reduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer
			,extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1,
			allocPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result = resultPointer[0];
	return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Double(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[21], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	reduce3Double<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			nullptr,
			1,
			1, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
double   NativeOps::execReduce3ScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[21], deviceProperties[(int) extraPointers[2]]);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	double *resultPointer = reinterpret_cast<double *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	reduce3Double<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1, allocationPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result  = resultPointer[0];
	return result;
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Double(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension,
		int dimensionLength){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[21], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	reduce3Double<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		int xStride,
		Nd4jPointer result,
		int resultStride,
		double scalar,
		Nd4jPointer extraParams,
		Nd4jIndex n) {
	double *xPointer = reinterpret_cast<double *>(x);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[20], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	scalarDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			scalar,
			xPointer,
			xStride,
			extraParamsPointer,
			resultPointer,resultStride, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		double scalar,
		Nd4jPointer extraParams){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[19], deviceProperties[(int) extraPointers[2]]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	scalarDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			scalar,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,resultShapeInfoPointer, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		double scalar,
		Nd4jPointer extraParams,
		Nd4jIndex n,
		Nd4jPointer xIndexes,
		Nd4jPointer resultIndexes){
	double *xPointer = reinterpret_cast<double *>(x);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[18], deviceProperties[(int) extraPointers[2]]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	scalarDoubleIndexes<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			scalar,
			xPointer,
			extraParamsPointer,
			resultPointer,
			resultIndexesPointer, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execSummaryStatsScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,bool biasCorrected){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[17], deviceProperties[(int) extraPointers[2]]);

	double *resultPointer = reinterpret_cast<double *>(extraPointers[5]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	summaryStatsReduceDouble<<<launchDims.x,launchDims.y,launchDims.z * 10, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1,biasCorrected, allocationPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result = resultPointer[0];
	return result;

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,bool biasCorrected) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[17], deviceProperties[(int) extraPointers[2]]);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	summaryStatsReduceDouble<<<launchDims.x,launchDims.y,launchDims.z * 10, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			nullptr,
			1,
			1,biasCorrected, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension, int dimensionLength,bool biasCorrected){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[17], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	summaryStatsReduceDouble<<<launchDims.x,launchDims.y,launchDims.z * 10, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1,biasCorrected, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		int xStride,
		Nd4jPointer result,
		int resultStride,
		Nd4jPointer extraParams,
		Nd4jIndex n) {
	double *xPointer = reinterpret_cast<double *>(dx);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[16], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	transformDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			xPointer,
			xStride,
			extraParamsPointer,
			resultPointer,resultStride, allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams){
	double *xPointer = reinterpret_cast<double *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *resultShapeInfoPointer =  reinterpret_cast<int *>(resultShapeInfo);
	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[1], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	// special pointer for special buffer for special ops
	double *specialPointer = reinterpret_cast<double *>(extraPointers[6]);

	int *dimension = (int *) specialPointer;
	int *maxDimension = dimension + 1;
	int *maxShapeBuffer = (int *) maxDimension + 1;
	double * special = (double *) maxShapeBuffer + 8;

	// simple trick to get workaround over reductions into scalar
	if (opNum >= 38 && opNum <= 41) {
		if (shape::isVector(hostXShapeInfo) && opNum != 41) {
			// if that's vector, we just go directly to op in 1 block
			transformDouble<<< 1, launchDims.y, launchDims.z * 3, *stream >> > (
					opNum,
							xPointer,
							xShapeInfoPointer,
							extraParamsPointer,
							resultPointer, resultShapeInfoPointer, allocPointer, reductionPointer);
		} else {
			// going for blockwise specials
			//float *xpf = reinterpret_cast<float *>(dx);

			int *shape = shape::shapeOf(hostXShapeInfo);
			//printf("Rows num: %i\n", shape[0]);
			switch (opNum) {
				case 40: // LogSoftMax
				case 39: // SoftMax Derivative
				case 38: {// softmax
					prepareShapeBuffer << < 1, 1, 128, *stream >> > (dimension, maxDimension, maxShapeBuffer, shape[0]);

					//checkCudaErrors(hipStreamSynchronize(*stream));

					// max 3
					execReduceDouble(extraPointers, 3, dx, xShapeInfo, extraParams, (Nd4jPointer) special,
									(Nd4jPointer) maxShapeBuffer, (Nd4jPointer) maxDimension, 1);

					// sub 1
					execBroadcastDouble(extraPointers, 1, dx, xShapeInfo, (Nd4jPointer) special,
									   (Nd4jPointer) maxShapeBuffer, dx, xShapeInfo, (Nd4jPointer) dimension, 1);

					// exp 3
					execTransformDouble(extraPointers, 3, dx, xShapeInfo, dx, xShapeInfo, extraParams);

					//sum 1
					execReduceDouble(extraPointers, 1, dx, xShapeInfo, extraParams, (Nd4jPointer) special,
									(Nd4jPointer) maxShapeBuffer, (Nd4jPointer) maxDimension, 1);

					// divide 3
					execBroadcastDouble(extraPointers, 3, dx, xShapeInfo, (Nd4jPointer) special,
									   (Nd4jPointer) maxShapeBuffer, dx, xShapeInfo, (Nd4jPointer) dimension, 1);

					// log 3
					if (opNum == 40)
						execTransformDouble(extraPointers, 5, dx, xShapeInfo, dx, xShapeInfo, extraParams);
					else if (opNum == 39)
						execTransformDouble(extraPointers, 42, dx, xShapeInfo, dx, xShapeInfo, extraParams);

					break;
				}
				case 41: {
					// IsMax along all dimensions
					if (extraParamsPointer == nullptr) {
						int maxIdx = (int) execIndexReduceScalarDouble(extraPointers, 0, dx, xShapeInfo, extraParams);
						int targetIdx = 0;

						if (shape::order(hostXShapeInfo) == 'c' || shape::order(hostXShapeInfo) == 'f' && maxIdx * shape::stride(hostXShapeInfo)[shape::rank(hostXShapeInfo) - 1] >= shape::length(hostXShapeInfo))
							targetIdx = maxIdx;
						else
							targetIdx = maxIdx * shape::stride(hostXShapeInfo)[shape::rank(hostXShapeInfo) - 1];

						fillIsMaxDouble<<< 256, 256, 0, *stream >>>(resultPointer, shape::length(hostXShapeInfo), targetIdx);
					} else {
						// going for dimension-based IsMax
						execIndexReduceDouble(extraPointers,0, dx, xShapeInfo, extraParams, result, resultShapeInfo, (Nd4jPointer) dimension, 1);
					}
					break;
				}
				default: {
					printf("Bad case for transformFloat\n");
					break;
				}
			}
		}
	} else {
		transformDouble<<<launchDims.x, launchDims.y, launchDims.z * 3, *stream>>> (
				opNum,
						xPointer,
						xShapeInfoPointer,
						extraParamsPointer,
						resultPointer, resultShapeInfoPointer, allocPointer, reductionPointer);
	}
	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer resultIndexes) {
	double *xPointer = reinterpret_cast<double *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[14], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	transformDoubleIndexes<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultIndexesPointer, allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
float   NativeOps::execIndexReduceScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[13], deviceProperties[(int) extraPointers[2]]);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	float *resultPointer = reinterpret_cast<float *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	indexReduceFloat<<<launchDims.x,launchDims.y, launchDims.z * 2, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1, allocationPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	float result = resultPointer[0];
	return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension,
		int dimensionLength){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[13], deviceProperties[(int) extraPointers[2]]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	indexReduceFloat<<<launchDims.x,launchDims.y,launchDims.z * 2, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer dimension, int dimensionLength){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[12], deviceProperties[(int) extraPointers[2]]);

	broadcastFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		int xStride,
		Nd4jPointer y,
		int yStride,
		Nd4jPointer result,
		int resultStride,
		Nd4jPointer extraParams, Nd4jIndex n){
	float *xPointer = reinterpret_cast<float *>(dx);
	float *yPointer = reinterpret_cast<float *>(y);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[11], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	pairWiseTransformStridedFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			xPointer,
			yPointer,
			xStride,
			yStride,
			extraParamsPointer,
			resultPointer,
			resultStride, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer yIndexes,
		Nd4jPointer resultIndexes){
	float *xPointer = reinterpret_cast<float *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
	int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[10], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	pairWiseTransformFloatIndex<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			yPointer,
			extraParamsPointer,
			resultPointer,
			xShapeInfoPointer,
			yShapeInfoPointer,
			resultShapeInfoPointer,
			xIndexesPointer,
			yIndexesPointer,
			resultIndexesPointer, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer  xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer  yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer  resultShapeInfo,
		Nd4jPointer extraParams){
	float *xPointer = reinterpret_cast<float *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[9], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	pairWiseTransformFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			yPointer,
			extraParamsPointer,
			resultPointer,
			xShapeInfoPointer,
			yShapeInfoPointer,
			resultShapeInfoPointer, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo) {
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[8], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	reduceFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer
			,extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			nullptr,
			1,
			1,
			allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer dimension,int dimensionLength){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[8], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	reduceFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer
			,extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1,
			allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
float NativeOps::execReduceScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[8], deviceProperties[(int) extraPointers[2]]);

	float *resultPointer = reinterpret_cast<float *>(extraPointers[5]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	reduceFloat<<< launchDims.x,launchDims.y, launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer
			,extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1,
			allocPointer,
			reductionPointer
	);


	checkCudaErrors(hipStreamSynchronize(*stream));

	float result = resultPointer[0];
	return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Float(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[7], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	reduce3Float<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			nullptr,
			1,
			1, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
float   NativeOps::execReduce3ScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo) {
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[7], deviceProperties[(int) extraPointers[2]]);

	float *resultPointer = reinterpret_cast<float *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	reduce3Float<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1, allocationPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result  = resultPointer[0];
	return result;

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Float(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension,
		int dimensionLength){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[7], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	reduce3Float<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		int xStride,
		Nd4jPointer result,
		int resultStride,
		double scalar,
		Nd4jPointer extraParams,
		Nd4jIndex n){
	float *xPointer = reinterpret_cast<float *>(x);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[6], deviceProperties[(int) extraPointers[2]]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	scalarFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			scalar,
			xPointer,
			xStride,
			extraParamsPointer,
			resultPointer,resultStride, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		float scalar,
		Nd4jPointer extraParams){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);
	Nd4jIndex n = shape::length(hostXShapeInfo);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[5], deviceProperties[(int) extraPointers[2]]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	scalarFloat<<<launchDims.x, launchDims.y,launchDims.z, *stream>>>(
			opNum,
			scalar,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,resultShapeInfoPointer, allocPointer );

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		double scalar,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer resultIndexes){
	float *xPointer = reinterpret_cast<float *>(x);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	int *hostShapeInfo = reinterpret_cast<int *>(extraPointers[0]);
	Nd4jIndex n = shape::length(hostShapeInfo);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[4], deviceProperties[(int) extraPointers[2]]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	scalarFloatIndexes<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			scalar,
			xPointer,
			extraParamsPointer,
			resultPointer,
			resultIndexesPointer, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
float   NativeOps::execSummaryStatsScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,bool biasCorrected){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[3], deviceProperties[(int) extraPointers[2]]);

	float *resultPointer = reinterpret_cast<float *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	summaryStatsReduceFloat<<<launchDims.x,launchDims.y,launchDims.z * 3, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1,biasCorrected, allocationPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	float result = resultPointer[0];
	return result;
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,bool biasCorrected){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[3], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	summaryStatsReduceFloat<<<launchDims.x,launchDims.y,launchDims.z * 3, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			nullptr,
			1,
			1,biasCorrected, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension,
		int dimensionLength,bool biasCorrected){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);
	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[3], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	summaryStatsReduceFloat<<<launchDims.x,launchDims.y,launchDims.z * 3, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1,biasCorrected, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		int xStride,
		Nd4jPointer result,
		int resultStride,
		Nd4jPointer extraParams,
		Nd4jIndex n) {
	float *xPointer = reinterpret_cast<float *>(dx);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[2], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	transformFloat<<<launchDims.x,launchDims.y,launchDims.z * 2, *stream>>>(
			opNum,
			n,
			xPointer,
			xStride,
			extraParamsPointer,
			resultPointer,resultStride, allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(Nd4jPointer *extraPointers,int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams) {
	float *xPointer = reinterpret_cast<float *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[1], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	// special pointer for special buffer for special ops
	float *specialPointer = reinterpret_cast<float *>(extraPointers[6]);

	int *dimension = (int *) specialPointer;
	int *maxDimension = dimension + 1;
	int *maxShapeBuffer = (int *) maxDimension + 1;
	float * special = (float *) maxShapeBuffer + 8;

	// simple trick to get workaround over reductions into scalar
	if (opNum >= 38 && opNum <= 41) {
		if (shape::isVector(hostXShapeInfo) && opNum != 41) {
			// if that's vector, we just go directly to op in 1 block
			transformFloat <<< 1, launchDims.y, launchDims.z * 3, *stream >> > (
					opNum,
					xPointer,
					xShapeInfoPointer,
					extraParamsPointer,
					resultPointer, resultShapeInfoPointer, allocPointer, reductionPointer);
		} else {
			// going for blockwise specials
			//float *xpf = reinterpret_cast<float *>(dx);

			int *shape = shape::shapeOf(hostXShapeInfo);
			//printf("Rows num: %i\n", shape[0]);
			switch (opNum) {
				case 40: // LogSoftMax
				case 39: // SoftMax Derivative
				case 38: {// softmax
					prepareShapeBuffer << < 1, 1, 128, *stream >> > (dimension, maxDimension, maxShapeBuffer, shape[0]);

					//checkCudaErrors(hipStreamSynchronize(*stream));

					// max 3
					execReduceFloat(extraPointers, 3, dx, xShapeInfo, extraParams, (Nd4jPointer) special,
									(Nd4jPointer) maxShapeBuffer, (Nd4jPointer) maxDimension, 1);

					// sub 1
					execBroadcastFloat(extraPointers, 1, dx, xShapeInfo, (Nd4jPointer) special,
									   (Nd4jPointer) maxShapeBuffer, dx, xShapeInfo, (Nd4jPointer) dimension, 1);

					// exp 3
					execTransformFloat(extraPointers, 3, dx, xShapeInfo, dx, xShapeInfo, extraParams);

					//sum 1
					execReduceFloat(extraPointers, 1, dx, xShapeInfo, extraParams, (Nd4jPointer) special,
									(Nd4jPointer) maxShapeBuffer, (Nd4jPointer) maxDimension, 1);

					// divide 3
					execBroadcastFloat(extraPointers, 3, dx, xShapeInfo, (Nd4jPointer) special,
									   (Nd4jPointer) maxShapeBuffer, dx, xShapeInfo, (Nd4jPointer) dimension, 1);

					// log 3
					if (opNum == 40)
						execTransformFloat(extraPointers, 5, dx, xShapeInfo, dx, xShapeInfo, extraParams);
					else if (opNum == 39)
						execTransformFloat(extraPointers, 42, dx, xShapeInfo, dx, xShapeInfo, extraParams);

					break;
				}
				case 41: {
					// IsMax along all dimensions
					if (extraParamsPointer == nullptr) {
						int maxIdx = (int) execIndexReduceScalarFloat(extraPointers, 0, dx, xShapeInfo, extraParams);
						int targetIdx = 0;

						if (shape::order(hostXShapeInfo) == 'c' || shape::order(hostXShapeInfo) == 'f' && maxIdx * shape::stride(hostXShapeInfo)[shape::rank(hostXShapeInfo) - 1] >= shape::length(hostXShapeInfo))
							targetIdx = maxIdx;
						else
							targetIdx = maxIdx * shape::stride(hostXShapeInfo)[shape::rank(hostXShapeInfo) - 1];

						fillIsMaxFloat<<< 256, 256, 0, *stream >>>(resultPointer, shape::length(hostXShapeInfo), targetIdx);
					} else {
						// going for dimension-based IsMax
						execIndexReduceFloat(extraPointers,0, dx, xShapeInfo, extraParams, result, resultShapeInfo, (Nd4jPointer) dimension, 1);
					}
					break;
				}
				default: {
					printf("Bad case for transformFloat\n");
					break;
				}
			}
		}
	} else {
		transformFloat <<<launchDims.x, launchDims.y, launchDims.z * 3, *stream>>> (
				opNum,
				xPointer,
				xShapeInfoPointer,
				extraParamsPointer,
				resultPointer, resultShapeInfoPointer, allocPointer, reductionPointer);
	}

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer resultIndexes) {
	float *xPointer = reinterpret_cast<float *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[0], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	transformFloatIndexes<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultIndexesPointer, allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));


}


template <typename T>
__device__ void flattenKernelGeneric(int dOffset,
					char order,
					T *result,
					int *resultShapeInfo,
					T *input,
					int *inputShapeInfo, int *allocationPointer) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int *zShape = shape::shapeOf(resultShapeInfo);
	int *zStride = shape::stride(resultShapeInfo);


	int *yShape = shape::shapeOf(inputShapeInfo);
	int *yStride = shape::stride(inputShapeInfo);
	char yOrder = shape::order(inputShapeInfo);

	int len = shape::length(inputShapeInfo);

	int resultEWS = shape::elementWiseStride(resultShapeInfo);
	int inputEWS = shape::elementWiseStride(inputShapeInfo);
	if (yOrder == order) {
		if (resultEWS >= 1 && inputEWS >= 1) {
			for (int i = tid; i < len; i+= gridDim.x * blockDim.x) {
				result[i * resultEWS + dOffset] = input[i * inputEWS];
			}
		} else {
			int rank = shape::rank(inputShapeInfo);
			long allocSize = sizeof(int) * rank;
			int *coord = shape::cuMalloc(allocationPointer, allocSize);

			if(order == 'f') {
				for(int i = tid; i < len; i+= gridDim.x * blockDim.x) {
					shape::ind2sub(rank,yShape,i,coord);
					int offset = shape::getOffset(0,yShape,yStride,coord,rank);
					result[i + dOffset] = input[offset];
				}
			}
			else {
				for(int i = tid; i < len; i+= gridDim.x * blockDim.x) {
					shape::ind2subC(rank,yShape,i,coord);
					int offset = shape::getOffset(0,yShape,yStride,coord,rank);
					result[i + dOffset] = input[offset];
				}
			}

			if (tid * allocSize > PREALLOC_SIZE - allocSize) {
				free(coord);
			}
		}
	} else {
		int rank = shape::rank(inputShapeInfo);
		long allocSize = sizeof(int) * rank;
		int *coord = shape::cuMalloc(allocationPointer, allocSize);
		if(order == 'f') {
			for(int i = tid; i < len; i+= gridDim.x * blockDim.x) {
				shape::ind2sub(rank,yShape,i,coord);
				int offset = shape::getOffset(0,yShape,yStride,coord,rank);
				result[i+dOffset] = input[offset];
			}
		}
		else {
			for(int i = tid; i < len; i+= gridDim.x * blockDim.x) {
				shape::ind2subC(rank,yShape,i,coord);
				int offset = shape::getOffset(0,yShape,yStride,coord,rank);
				result[i+dOffset] = input[offset];
			}
		}
		if (tid * allocSize > PREALLOC_SIZE - allocSize) {
			free(coord);
		}
	}

}

extern "C" __global__ void flattenKernelDouble(int offset,
											  char order,
											  double *result,
											  int *resultShapeInfo,
											  double *input,
											  int *inputShapeInfo, int *allocationPointer) {
	flattenKernelGeneric<double>(offset, order, result, resultShapeInfo, input, inputShapeInfo, allocationPointer);
}

extern "C" __global__ void flattenKernelFloat(int offset,
											  char order,
											  float *result,
											  int *resultShapeInfo,
											  float *input,
											  int *inputShapeInfo, int *allocationPointer) {

	flattenKernelGeneric<float>(offset, order, result, resultShapeInfo, input, inputShapeInfo, allocationPointer);
}

/**
 * Append an input array
 * to the end of a flat array
 * in a particular order
 * @param offset the offset of the array to start at
 * @param order the order
 * @param result the result array
 * @param resultShapeInfo the shape info for te array
 * @param input the input for the array
 * @param inputShapeInfo the shape information for that array
 */
void NativeOps::flattenFloat(
		Nd4jPointer *extraPointers,
		int offset,
		char order,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer input,
		Nd4jPointer inputShapeInfo) {
	float *xPointer = reinterpret_cast<float *>(result);
	int *xShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *yPointer = reinterpret_cast<float *>(input);
	int *yShapeInfoPointer = reinterpret_cast<int *>(inputShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[5], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	flattenKernelFloat<<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(offset, order, xPointer, xShapeInfoPointer, yPointer, yShapeInfoPointer, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 * Append an input array
 * to the end of a flat array
 * in a particular order
 * @param offset the offset of the array to start at
 * @param order the order
 * @param result the result array
 * @param resultShapeInfo the shape info for te array
 * @param input the input for the array
 * @param inputShapeInfo the shape information for that array
 */
void NativeOps::flattenDouble(
		Nd4jPointer *extraPointers,
		int offset,
		char order,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer input,
		Nd4jPointer inputShapeInfo) {
	double *xPointer = reinterpret_cast<double *>(result);
	int *xShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *yPointer = reinterpret_cast<double *>(input);
	int *yShapeInfoPointer = reinterpret_cast<int *>(inputShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[5], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	flattenKernelDouble<<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(offset, order, xPointer, xShapeInfoPointer, yPointer, yShapeInfoPointer, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

void NativeOps::initializeDevicesAndFunctions() {
	int devCnt = 0;
	hipGetDeviceCount(&devCnt);
	deviceProperties = new hipDeviceProp_t[devCnt];
	for (int i = 0; i < devCnt; i++) {
		hipSetDevice(i);
		hipGetDeviceProperties(&deviceProperties[i], i);

		hipDeviceSetLimit(hipLimitStackSize, 10000);
		hipDeviceSetLimit(hipLimitMallocHeapSize , 10000);
	}

	hipSetDevice(0);

	hipFuncGetAttributes(&funcAttributes[0], reinterpret_cast<const void*>((void *))transformFloatIndexes);

	void (*transformFloatPointer1)(int opNum, float *dy,int *shapeInfo, float *params, float *result,int *resultShapeInfo, int *allocationPointer, float *reductionPointer) = transformFloat;
	hipFuncGetAttributes(&funcAttributes[1], reinterpret_cast<const void*>(transformFloatPointer1));

	void (*transformFloatPointer2)(int opNum, Nd4jIndex n, float *dy, int incy, float *params, float *result,int resultStride, int *allocationPointer, float *reductionPointer) = transformFloat;
	hipFuncGetAttributes(&funcAttributes[2], reinterpret_cast<const void*>(transformFloatPointer2));

	hipFuncGetAttributes(&funcAttributes[3], reinterpret_cast<const void*>((void *))summaryStatsReduceFloat);

	hipFuncGetAttributes(&funcAttributes[4], reinterpret_cast<const void*>((void *))scalarFloatIndexes);

	void (*scalarFloatPointer1)(int opNum, float dx,float *dy, int *shapeInfo,float *params, float *result,int *resultShapeInfo, int *allocPointer) = scalarFloat;
	hipFuncGetAttributes(&funcAttributes[5], reinterpret_cast<const void*>(scalarFloatPointer1));

	void (*scalarFloatPointer2)(int opNum, Nd4jIndex n,float dx, float *dy, int incy, float *params, float *result,int resultStride, int *allocPointer) = scalarFloat;
	hipFuncGetAttributes(&funcAttributes[6], reinterpret_cast<const void*>(scalarFloatPointer2));

	hipFuncGetAttributes(&funcAttributes[7], reinterpret_cast<const void*>(reduce3Float));

	hipFuncGetAttributes(&funcAttributes[8], reinterpret_cast<const void*>(reduceFloat));

	hipFuncGetAttributes(&funcAttributes[9], reinterpret_cast<const void*>(pairWiseTransformFloat));

	hipFuncGetAttributes(&funcAttributes[10], reinterpret_cast<const void*>(pairWiseTransformFloatIndex));

	hipFuncGetAttributes(&funcAttributes[11], reinterpret_cast<const void*>(pairWiseTransformStridedFloat));

	hipFuncGetAttributes(&funcAttributes[12], reinterpret_cast<const void*>(broadcastFloat));

	hipFuncGetAttributes(&funcAttributes[13], reinterpret_cast<const void*>(indexReduceFloat));

	///////////////////////////////////////// Doubles are separate, just in case of...

	hipFuncGetAttributes(&funcAttributes[14], reinterpret_cast<const void*>(transformDoubleIndexes));

	void (*transformDoublePointer1)(int opNum, double *dy, int *shapeInfo, double *params, double *result,int *resultShapeInfo, int *allocationPointer, double *reductionPointer) = transformDouble;
	hipFuncGetAttributes(&funcAttributes[15], reinterpret_cast<const void*>(transformDoublePointer1));

	void (*transformDoublePointer2)(int opNum, Nd4jIndex n, double *dy, int incy, double *params, double *result,int resultStride, int *allocationPointer, double *reductionPointer) = transformDouble;
	hipFuncGetAttributes(&funcAttributes[16], reinterpret_cast<const void*>(transformDoublePointer2));

	hipFuncGetAttributes(&funcAttributes[17], reinterpret_cast<const void*>(summaryStatsReduceDouble));

	hipFuncGetAttributes(&funcAttributes[18], reinterpret_cast<const void*>(scalarDoubleIndexes));

	void (*scalarDoublePointer1)(int opNum, double dx,double *dy, int *shapeInfo,double *params, double *result,int *resultShapeInfo, int *allocPointer) = scalarDouble;
	hipFuncGetAttributes(&funcAttributes[19], reinterpret_cast<const void*>(scalarDoublePointer1));


	void (*scalarDoublePointer2)(int opNum, Nd4jIndex n,double dx, double *dy, int incy, double *params, double *result,int resultStride, int *allocPointer) = scalarDouble;
	hipFuncGetAttributes(&funcAttributes[20], reinterpret_cast<const void*>(scalarDoublePointer2));

	hipFuncGetAttributes(&funcAttributes[21], reinterpret_cast<const void*>(reduce3Double));

	hipFuncGetAttributes(&funcAttributes[22], reinterpret_cast<const void*>(reduceDouble));

	hipFuncGetAttributes(&funcAttributes[23], reinterpret_cast<const void*>(pairWiseTransformDouble));

	hipFuncGetAttributes(&funcAttributes[24], reinterpret_cast<const void*>(pairWiseTransformDoubleIndex));

	hipFuncGetAttributes(&funcAttributes[25], reinterpret_cast<const void*>(pairWiseTransformStridedDouble));

	hipFuncGetAttributes(&funcAttributes[26], reinterpret_cast<const void*>(broadcastDouble));

	hipFuncGetAttributes(&funcAttributes[27], reinterpret_cast<const void*>(indexReduceDouble));
}


/**
 * This method acquires memory chunk of requested size on host side
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param flags optional parameter
 */
Nd4jPointer NativeOps::mallocHost(long memorySize, int flags) {
	Nd4jPointer pointer;
	hipError_t res = hipHostAlloc((void **)&pointer, memorySize, hipHostMallocMapped |hipHostMallocPortable );
	if (res != 0)
		pointer = 0L;
	return pointer;
}

/**
 * This method acquires memory chunk of requested size on specified device
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param ptrToDeviceId pointer to deviceId. For cuda that's just and int, for OpenCL that's pointer to device_id, etc
 * @param flags optional parameter
 */
Nd4jPointer NativeOps::mallocDevice(long memorySize, Nd4jPointer ptrToDeviceId, int flags) {
	Nd4jPointer pointer;
	hipError_t res = hipMalloc((void **)&pointer, memorySize);
	if (res != 0)
		pointer = 0L;
	return pointer;
}

/**
 * This method releases previously allocated host memory space
 *
 * @param pointer pointer that'll be freed
 */
Nd4jPointer NativeOps::freeHost(Nd4jPointer pointer) {
	hipError_t res = hipHostFree((void *) pointer);
	if (res != 0)
		pointer = 0L;
	return 1L;
}

/**
 * This method releases previously allocated memory space on device
 *
 * @param pointer pointer that'll be freed
 * @param ptrToDeviceId pointer to deviceId.
 */
Nd4jPointer NativeOps::freeDevice(Nd4jPointer pointer, Nd4jPointer ptrToDeviceId) {
	hipError_t res = hipFree((void *)pointer);
	if (res != 0)
		pointer = 0L;
	return 1L;
}


Nd4jPointer NativeOps::createContext() {
	return 0L;
}

Nd4jPointer NativeOps::createStream() {
	Nd4jPointer nativeStream = 0;
	hipError_t result = hipStreamCreate((hipStream_t *) &nativeStream);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return nativeStream;
}

Nd4jPointer NativeOps::createEvent() {
	Nd4jPointer nativeEvent= 0;
	hipError_t result = hipEventCreateWithFlags((hipEvent_t *) &nativeEvent, hipEventBlockingSync | hipEventDisableTiming);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return nativeEvent;
}

Nd4jPointer NativeOps::createBlasHandle() {
	Nd4jPointer nativeHandle= 0;
	hipblasStatus_t result = hipblasCreate((hipblasHandle_t *) &nativeHandle);
	if (result != 0)
		return 0L;
	else return nativeHandle;
}

Nd4jPointer NativeOps::registerEvent(Nd4jPointer event, Nd4jPointer stream) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&stream);

	hipError_t result = hipEventRecord(*pEvent, *pStream);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

Nd4jPointer NativeOps::setBlasStream(Nd4jPointer handle, Nd4jPointer stream) {
	hipblasHandle_t *pHandle = reinterpret_cast<hipblasHandle_t *>(&handle);
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&stream);

	hipblasStatus_t result = hipblasSetStream(*pHandle, *pStream);
	if (result != 0)
		return 0L;
	else return 1L;
}

Nd4jPointer NativeOps::setDevice(Nd4jPointer ptrToDeviceId) {
	int deviceId = (int) ptrToDeviceId;
	hipError_t result = hipSetDevice(deviceId);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

long NativeOps::getDeviceFreeMemory(Nd4jPointer ptrToDeviceId) {
	int device = (int) ptrToDeviceId;

	if (device >= 0) {
		setDevice(ptrToDeviceId);
	}
	size_t memFree = 0;
	size_t memTotal = 0;

	hipMemGetInfo(&memFree, &memTotal);

	return (long) memFree;
}

Nd4jPointer NativeOps::memcpy(Nd4jPointer dst, Nd4jPointer src, long size, int flags, Nd4jPointer reserved) {

	return memcpyAsync(dst, src, size, flags, reserved);
}

Nd4jPointer NativeOps::memcpyAsync(Nd4jPointer dst, Nd4jPointer src, long size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&reserved);

	hipMemcpyKind 	kind;

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*pStream));

	switch (flags) {
		case 0: {
				kind = hipMemcpyHostToHost;
			}
			break;
		case 1: {
				kind = hipMemcpyHostToDevice;
			}
			break;
		case 2: {
				kind = hipMemcpyDeviceToHost;
			}
		case 3: {
			kind = hipMemcpyDeviceToDevice;
		}
			break;
	}

	hipError_t result = hipMemcpyAsync((void *) dst, (const void *) src, (size_t) size, kind, *pStream);
	checkCudaErrors(result);
	if (result != 0) {
		printf("Failed on [%lu] -> [%lu], size: [%i], direction: [%i]\n", src, dst, size, flags );
		return 0L;
	}
	else return 1;
}

Nd4jPointer NativeOps::memset(Nd4jPointer dst, int value, long size, int flags, Nd4jPointer reserved) {
	//hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&reserved);

	hipError_t result = hipMemset((void *) dst, value, (size_t) size);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

Nd4jPointer NativeOps::memsetAsync(Nd4jPointer dst, int value, long size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&reserved);

	hipError_t result = hipMemsetAsync((void *) dst, value, (size_t) size, *pStream);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

Nd4jPointer NativeOps::destroyEvent(Nd4jPointer event) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);
	hipError_t result = hipEventDestroy(*pEvent);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

Nd4jPointer NativeOps::streamSynchronize(Nd4jPointer stream) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&stream);

	hipError_t result = hipStreamSynchronize(*pStream);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1L;
}

Nd4jPointer NativeOps::eventSynchronize(Nd4jPointer event) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);

	hipError_t result = hipEventSynchronize(*pEvent);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1L;
}

Nd4jPointer NativeOps::getAvailableDevices() {
	int devCnt = 0;
	hipGetDeviceCount(&devCnt);
	return (Nd4jPointer) devCnt;
}

void NativeOps::enableDebugMode(bool reallyEnable) {
	debug = reallyEnable;
}

void NativeOps::setGridLimit(int gridSize) {
	blockLimit = gridSize;
}


int NativeOps::ompGetNumThreads() {
	return maxThreads;
}

void NativeOps::setOmpNumThreads(int threads) {
	maxThreads = threads;
}
