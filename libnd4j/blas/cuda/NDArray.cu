//
// CUDA-compatible NDArray implementation
//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include "../NDArray.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <pointercast.h>

namespace nd4j {


////////////////////////////////////////////////////////////////////////
template<typename T>
void* NDArray<T>::operator new(size_t i) {
	
	return nullptr; 
}


////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::operator delete(void* p) {
    
}

////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::getView() {

	return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
template <typename T>
template <typename N>
NDArray<N>* NDArray<T>::asT() {	

        return new NDArray<N>();
}

////////////////////////////////////////////////////////////////////////
// default constructor, do not allocate memory, memory for array is passed from outside 
template <typename T>
NDArray<T>::NDArray(T *buffer, Nd4jLong *shapeInfo, nd4j::memory::Workspace* workspace) {

}

////////////////////////////////////////////////////////////////////////
//constructor, create empty array at given workspace
template <typename T>
NDArray<T>::NDArray(nd4j::memory::Workspace* workspace) {

}

////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>::NDArray(std::initializer_list<Nd4jLong> s, nd4j::memory::Workspace* workspace) {

}

////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>::NDArray(T scalar) {

}

////////////////////////////////////////////////////////////////////////
#ifndef __JAVACPP_HACK__
template <typename T>
NDArray<T>::NDArray(std::initializer_list<T> v, nd4j::memory::Workspace* workspace) {

}

////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>::NDArray(std::vector<T> &values, nd4j::memory::Workspace* workspace) {

}
#endif

////////////////////////////////////////////////////////////////////////
// creates new NDArray using shape information from "shapeInfo" array, set all elements in new array to be zeros
template <typename T>
NDArray<T>::NDArray(const Nd4jLong* shapeInfo, const bool copyStrides, nd4j::memory::Workspace* workspace) {
   
}

////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>::NDArray(const char order, const std::vector<Nd4jLong> &shape, const std::vector<T> &data, nd4j::memory::Workspace* workspace) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>::NDArray(const char order, const std::vector<Nd4jLong> &shape, nd4j::memory::Workspace* workspace) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>::NDArray(T* buffer, const char order, const std::vector<Nd4jLong> &shape, nd4j::memory::Workspace* workspace) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
std::string NDArray<T>::toStringValue(T value) {

	return "";
}

////////////////////////////////////////////////////////////////////////
template<>
std::string NDArray<float16>::toStringValue(float16 value) {

	return "";
}

////////////////////////////////////////////////////////////////////////
template<typename T>
std::string NDArray<T>::asIndexedString(Nd4jLong limit) {

	return "";
}

////////////////////////////////////////////////////////////////////////
template<typename T>
std::string NDArray<T>::asString(Nd4jLong limit) {

	return "";
}

////////////////////////////////////////////////////////////////////////
template<typename T>
std::vector<T> NDArray<T>::getBufferAsVector() {

	return std::vector<T>();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
std::vector<Nd4jLong> NDArray<T>::getShapeAsVector() {

 	return std::vector<Nd4jLong >();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
std::vector<int64_t> NDArray<T>::getShapeInfoAsFlatVector() {

 	return std::vector<int64_t>();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
std::vector<Nd4jLong> NDArray<T>::getShapeInfoAsVector() {

	return std::vector<Nd4jLong>();
}

////////////////////////////////////////////////////////////////////////
#ifndef __JAVACPP_HACK__
template<typename T>
void NDArray<T>::applyTriplewiseLambda(NDArray<T>* second, NDArray<T> *third, const std::function<T(T, T, T)>& func, NDArray<T>* target) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::applyPairwiseLambda(NDArray<T>* other, const std::function<T(T, T)>& func, NDArray<T>* target) {
        
}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::applyLambda(const std::function<T(T)>& func, NDArray<T>* target) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::applyIndexedLambda(const std::function<T(Nd4jLong, T)>& func, NDArray<T>* target) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::applyIndexedPairwiseLambda(NDArray<T>* other, const std::function<T(Nd4jLong, T, T)>& func, NDArray<T>* target) {

}
#endif

////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>::NDArray(const NDArray<T> *other, const bool copyStrides, nd4j::memory::Workspace* workspace) {

}

////////////////////////////////////////////////////////////////////////
template <typename T>
std::vector<int8_t> NDArray<T>::asByteVector() {

	return std::vector<int8_t>();
}

////////////////////////////////////////////////////////////////////////
// copy constructor
template <typename T>
NDArray<T>::NDArray(const NDArray<T>& other) {

}

////////////////////////////////////////////////////////////////////////
// move constructor
template <typename T>
NDArray<T>::NDArray(NDArray<T>&& other) noexcept {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
T* NDArray<T>::getBuffer() {
	return new T();
}

template<typename T>
T* NDArray<T>::buffer() {
	return new T();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
Nd4jLong* NDArray<T>::getShapeInfo() const{
	return new Nd4jLong();
}

template<typename T>
Nd4jLong* NDArray<T>::shapeInfo() {
	return new Nd4jLong();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
T* NDArray<T>::specialBuffer() {
	return new T();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
Nd4jLong* NDArray<T>::specialShapeInfo() {

	return new Nd4jLong();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::setSpecialBuffers(T * buffer, Nd4jLong *shape) {

}

////////////////////////////////////////////////////////////////////////
// assignment operator
template<typename T>
NDArray<T>& NDArray<T>::operator=(const NDArray<T>& other) {

	return *this;
}

////////////////////////////////////////////////////////////////////////
// move assignment operator
template <typename T>
NDArray<T>& NDArray<T>::operator=(NDArray<T>&& other) noexcept {

    return *this;
}

////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>& NDArray<T>::operator=(const T scalar) {
    
    return *this;
}

////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::replacePointers(T *buffer, Nd4jLong *shapeInfo, const bool releaseExisting ) {

}

////////////////////////////////////////////////////////////////////////
// This method assigns values of given NDArray to this one, wrt order
template<typename T>
void NDArray<T>::assign(const NDArray<T> *other) {

}

////////////////////////////////////////////////////////////////////////
// This method assigns values of given NDArray to this one
template<typename T>
void NDArray<T>::assign(const NDArray<T>& other) {

}

////////////////////////////////////////////////////////////////////////
// This method assigns given value to all elements in this NDArray
template<typename T>
void NDArray<T>::assign(const T value) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>* NDArray<T>::detach() {

	return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// This method returns new copy of this NDArray, optionally in different order
template <typename T>
NDArray<T>* NDArray<T>::dup(const char newOrder) {

    return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
T NDArray<T>::varianceNumber(bool biasCorrected) {

	return T();
}

//////////////////////////////////////////////////////////////////////////
// This method returns sum of all elements of this NDArray
template<typename T>
T NDArray<T>::sumNumber() const {
	return T();
}

//////////////////////////////////////////////////////////////////////////
// This method returns mean number of this NDArray
template<typename T>
T NDArray<T>::meanNumber() const {
    return T();
}

//////////////////////////////////////////////////////////////////////////
// method calculates sum along dimension(s) in this array and save it to row: as new NDArray with dimensions 1xN
template<typename T>
NDArray<T>* NDArray<T>::sum(const std::vector<int> &dimensions) const {

	return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
bool NDArray<T>::isContiguous() {

	return true;
}

//////////////////////////////////////////////////////////////////////////
// eventually method reduces array by excluding its shapes along axes present in dimensions vector
template<typename T>
template<typename OpName>
NDArray<T> *NDArray<T>::reduceAlongDimension(const std::vector<int>& dimensions, const bool keepDims, const bool supportOldShapes) const {
            
	return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
// eventually method reduces array by excluding its shapes along axes present in dimensions vector
template<typename T>
template<typename OpName>
NDArray<T> NDArray<T>::reduceAlongDims(const std::vector<int>& dimensions, const bool keepDims, const bool supportOldShapes) const {
                
    return NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
// method reduces array by excluding its shapes along axes present in dimensions vector
template<typename T>
template<typename OpName>
void NDArray<T>::reduceAlongDimension(NDArray<T>* target, const std::vector<int>& dimensions, const bool keepDims, const bool supportOldShapes, T *extras) const {

}

//////////////////////////////////////////////////////////////////////////
// method reduces array by excluding its shapes along axes present in dimensions vector
template<typename T>
template<typename OpName>
NDArray<T>* NDArray<T>::reduceAlongDimension(const std::initializer_list<int>& dimensions, const bool keepDims, const bool supportOldShapes) const {
		        
    return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
T NDArray<T>::reduceNumber(T *extraParams) const {

	return T();
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
Nd4jLong NDArray<T>::indexReduceNumber(T *extraParams) {

	return Nd4jLong();
}

//////////////////////////////////////////////////////////////////////////
// perform array transformation
template<typename T>
template<typename OpName>
void NDArray<T>::applyTransform(NDArray<T> *target, T *extraParams) {

}

//////////////////////////////////////////////////////////////////////////
// perform array transformation
template<typename T>
template<typename OpName>
void NDArray<T>::applyTransform(T *extraParams) {

}

//////////////////////////////////////////////////////////////////////////
// perform array transformation
template<typename T>
template<typename OpName>
NDArray<T> NDArray<T>::transform(T *extraParams) {
    
	return NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
// perform pairwise transformation
template<typename T>
template<typename OpName>
void NDArray<T>::applyPairwiseTransform(NDArray<T> *other, T *extraParams) {

}

//////////////////////////////////////////////////////////////////////////
// perform pairwise transformation
template<typename T>
template<typename OpName>
void NDArray<T>::applyPairwiseTransform(NDArray<T> *other, NDArray<T> *target, T *extraParams) {
                                       
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
void NDArray<T>::applyRandom(nd4j::random::RandomBuffer *buffer, NDArray<T>* y, NDArray<T>* z, T* extraArgs) {

}

//////////////////////////////////////////////////////////////////////////
template <typename T>
Nd4jLong NDArray<T>::tensorsAlongDimension(std::initializer_list<int> dimensions) const {

	return Nd4jLong();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
Nd4jLong NDArray<T>::tensorsAlongDimension(const std::vector<int>& dimensions) const {
        
	return Nd4jLong();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::tensorAlongDimension(Nd4jLong index, const std::initializer_list<int>& dimensions) const {

	return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::printShapeInfo(const char * msg) const {

}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::printBuffer(const char* msg, Nd4jLong limit) {
        
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::printIndexedBuffer(const char* msg, Nd4jLong limit) const {

}

//////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::tensorAlongDimension(Nd4jLong index, const std::vector<int>& dimensions) const {

	return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
// method makes copy of this array and applies to the copy transpose operation, this array remains unaffected 
template <typename T>
NDArray<T>* NDArray<T>::transpose() const {
	return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// method performs transpose operation based on this array and store result in target, this array remains unaffected 
template <typename T>
void NDArray<T>::transpose(NDArray<T>& target) const {

}

////////////////////////////////////////////////////////////////////////
// This method applies in-place transpose to this array, so this array becomes transposed 
template <typename T>
void NDArray<T>::transposei() {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
bool NDArray<T>::equalsTo(NDArray<T> &other, T eps) const {
    
    return true;
}

////////////////////////////////////////////////////////////////////////
// This method returns true if two arrays are equal, with custom or default Eps value of 1e-5, false otherwise
template<typename T>
bool NDArray<T>::equalsTo(const NDArray<T> *other, T eps) const {

	return true;
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::addRowVector(const NDArray<T> *row, NDArray<T>* target) const {

}

//////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::subRowVector(const NDArray<T> *row, NDArray<T>* target) const {

}

//////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::mulRowVector(const NDArray<T> *row, NDArray<T>* target) const {

}

//////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::divRowVector(const NDArray<T> *row, NDArray<T>* target) const {

}

//////////////////////////////////////////////////////////////////////////
// This method adds given row to all rows in this NDArray, this array becomes affected
template<typename T>
void NDArray<T>::addiRowVector(const NDArray<T> *row) {

}

//////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::addColumnVector(const NDArray<T> *column, NDArray<T>* target) const {

}

//////////////////////////////////////////////////////////////////////////
// This method adds given column to all columns in this NDArray, this array becomes affected
template<typename T>
void NDArray<T>::addiColumnVector(const NDArray<T> *column) {

}

//////////////////////////////////////////////////////////////////////////
// This method multiplies each column of this array by given argument-column, this array becomes affected
template<typename T>
void NDArray<T>::muliColumnVector(const NDArray<T> *column) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
void NDArray<T>::applyScalar(T scalar, NDArray<T>* target, T *extraParams) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
void NDArray<T>::applyScalar(NDArray<T>& scalar, NDArray<T>* target, T *extraParams) {

}

//////////////////////////////////////////////////////////////////////////
// calculate strides 
template <typename T>
void NDArray<T>::updateStrides(const char order) {
	
}

//////////////////////////////////////////////////////////////////////////
// set new order and shape in case of suitable array length 
template <typename T>
bool NDArray<T>::reshapei(const char order, const std::initializer_list<Nd4jLong>& shape) {
	return true;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
bool NDArray<T>::reshapei(const std::initializer_list<Nd4jLong>& shape) {
    return true;
}	

//////////////////////////////////////////////////////////////////////////
template <typename T>
bool NDArray<T>::reshapei(const std::vector<Nd4jLong>& shape) {
    return true;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::enforce(const std::initializer_list<Nd4jLong> &dimensions, char order) {

}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::enforce(std::vector<Nd4jLong> &dimensions, char o) {

}

//////////////////////////////////////////////////////////////////////////
// set new order and shape in case of suitable array length 
template <typename T>
bool NDArray<T>::reshapei(const char order, const std::vector<Nd4jLong>& cshape) {

    return true;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
Nd4jLong NDArray<T>::argMax(std::initializer_list<int> dimensions) {

	return Nd4jLong();
}

//////////////////////////////////////////////////////////////////////////
// create new array with corresponding order and shape, new array will point to the same _buffer as this array
template <typename T>
NDArray<T>* NDArray<T>::reshape(const char order, const std::vector<Nd4jLong>& shape) const {

	return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
// change an array by repeating it the number of times given by reps.
template <typename T>
void NDArray<T>::tilei(const std::vector<Nd4jLong>& reps) {
	
}

//////////////////////////////////////////////////////////////////////////
// change an array by repeating it the number of times given by reps.
template <typename T>
NDArray<T> NDArray<T>::tile(const std::vector<Nd4jLong>& reps) const {
    
    return NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
// change an array by repeating it the number of times given by reps.
template <typename T>
void NDArray<T>::tile(const std::vector<Nd4jLong>& reps, NDArray<T>& target) const {

}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::tile(NDArray<T>& target) const {

}

//////////////////////////////////////////////////////////////////////////
template<typename T>
Nd4jLong NDArray<T>::sizeAt(int dim) const {

	return Nd4jLong();
}

//////////////////////////////////////////////////////////////////////////
// create new  array by repeating it the number of times given by reps
template<typename T>
NDArray<T>* NDArray<T>::repeat(int dimension, const std::vector<Nd4jLong>& repeats) const {

    return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
// fill array by repeating it the number of times given by reps
template<typename T>
void NDArray<T>::repeat(int dimension, NDArray<T>& target) const {

}


//////////////////////////////////////////////////////////////////////////
template <typename T>
bool NDArray<T>::permutei(const int* dimensions, const int rank) {

    return true;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
bool NDArray<T>::permutei(const Nd4jLong* dimensions, const int rank) {

	return true;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
bool NDArray<T>::permutei(const std::initializer_list<int>& dimensions) {
    
    return true;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
bool NDArray<T>::permutei(const std::vector<int>& dimensions) {
    
    return true;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
bool NDArray<T>::permutei(const std::initializer_list<Nd4jLong>& dimensions) {

    return true;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
bool NDArray<T>::permutei(const std::vector<Nd4jLong>& dimensions) {

    return true;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::permute(const int* dimensions, const int rank) const {

    return new NDArray<T>();
}

/////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::permute(const Nd4jLong* dimensions, const int rank) const {

	return new NDArray<T>();
}


//////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::permute(const std::vector<int>& dimensions) const {

	return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::permute(const std::vector<Nd4jLong>& dimensions) const {

	return new NDArray<T>();
}


//////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::permute(const std::initializer_list<int>& dimensions) const {
    
    std::vector<int> vec(dimensions);
    return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::permute(const std::initializer_list<Nd4jLong>& dimensions) const {
    
    std::vector<Nd4jLong> vec(dimensions);
		return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::permute(const int* dimensions, const int rank, NDArray<T>& target) const {

}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::permute(const Nd4jLong *dimensions, const int rank, NDArray<T>& target) const {

}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::permute(const std::vector<int>& dimensions, NDArray<T>& target) const {

}

//////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::permute(const std::vector<Nd4jLong>& dimensions, NDArray<T>& target) const {
	
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
void NDArray<T>::applyBroadcast(std::initializer_list<int> dimensions, const NDArray<T>* tadArray, NDArray<T>* target, T* extraArgs) {

}

//////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
void NDArray<T>::applyBroadcast(std::vector<int>& dimensions, const NDArray<T>* tadArray, NDArray<T>* target, T* extraArgs) {

}

//////////////////////////////////////////////////////////////////////////
template<typename T>
template <typename OpName>
void NDArray<T>::applyTrueBroadcast(const NDArray<T>* other, NDArray<T>* target, const bool checkTargetShape, T *extraArgs) const {

}

//////////////////////////////////////////////////////////////////////////
template<typename T>
template <typename OpName>
NDArray<T>* NDArray<T>::applyTrueBroadcast(const NDArray<T>* other, T *extraArgs) const {
 
    return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
template <typename OpName>
NDArray<T> NDArray<T>::applyTrueBroadcast(const NDArray<T>& other, T *extraArgs) const {

    return NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
// return array which is broadcasted from this and argument array  
template<typename T>
NDArray<T>* NDArray<T>::broadcast(const NDArray<T>& other) {	

    return new NDArray<T>();
}

//////////////////////////////////////////////////////////////////////////
// check whether array's rows (arg=0) or columns (arg=1) create orthogonal basis
template<typename T>
bool NDArray<T>::hasOrthonormalBasis(const int arg) {
    
    return true;
}

//////////////////////////////////////////////////////////////////////////
// check whether array is identity matrix
template<typename T>
bool NDArray<T>::isIdentityMatrix() {

	return true;
}

//////////////////////////////////////////////////////////////////////////
// check whether array is unitary matrix
template<typename T>
bool NDArray<T>::isUnitary() {
    
    return true;
}

////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>* NDArray<T>::subarray(IndicesList& idx, std::vector<Nd4jLong>& strides) const {

    return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>* NDArray<T>::subarray(IndicesList& idx) const {
    
    return new NDArray<T>();
}
    
////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>* NDArray<T>::subarray(const std::initializer_list<NDIndex*>& idx) const {

    return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>* NDArray<T>::subarray(const Intervals& idx) const {

    return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
template <typename T>
NDArray<T>* NDArray<T>::cast(DataType dtype) {

	return nullptr;
}

////////////////////////////////////////////////////////////////////////
template <typename T>
void NDArray<T>::cast(NDArray<T>* target, DataType dtype) {
	
}

////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
void NDArray<T>::applyIndexReduce(const NDArray<T>* target, const std::vector<int>& dimensions, const T *extraParams) const {

}

////////////////////////////////////////////////////////////////////////
// reduce dimensions in this array relying on index operations
template<typename T>
template<typename OpName>
NDArray<T>* NDArray<T>::applyIndexReduce(const std::vector<int>& dimensions, const T* extraParams ) const {
        
	return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// apply reduce3 operations to this and other array, return result in new output array
template<typename T>
template<typename OpName>
NDArray<T>* NDArray<T>::applyReduce3(const NDArray<T>* other, const T* extraParams) const {
	
	return new NDArray<T>();
}
    
////////////////////////////////////////////////////////////////////////
// apply reduce3 (execAll) operations to this and other array, return result in new output array
template<typename T>
template<typename OpName>
NDArray<T>*  NDArray<T>::applyAllReduce3(const NDArray<T>* other, const std::vector<int>& dimensions, const T* extraParams) const {

	return new NDArray<T>();
}
 
////////////////////////////////////////////////////////////////////////
// apply reduce3 (exec) operations to this and other array, return result in new output array
template<typename T>
template<typename OpName>
NDArray<T>* NDArray<T>::applyReduce3(const NDArray<T>* other, const std::vector<int>& dimensions, const T* extraParams) const {

	return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
NDArray<T>* NDArray<T>::varianceAlongDimension(const bool biasCorrected, const std::vector<int>& dimensions) const {

	return new NDArray<T>();    
}
    
////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
NDArray<T>* NDArray<T>::varianceAlongDimension(const bool biasCorrected, const std::initializer_list<int>& dimensions) const {
    
	return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
void NDArray<T>::varianceAlongDimension(const NDArray<T> *target, const bool biasCorrected, const std::vector<int>& dimensions) {
	
}

////////////////////////////////////////////////////////////////////////
template<typename T>
template<typename OpName>
void NDArray<T>::varianceAlongDimension(const NDArray<T> *target, const bool biasCorrected, const std::initializer_list<int>& dimensions) {

}

////////////////////////////////////////////////////////////////////////
// operator returns sub-array with buffer pointing at this->_buffer + certain offset
template<typename T>
NDArray<T> NDArray<T>::operator()(const int* idx, bool keepUnitiesInShape)  const {

	return NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// operator returns sub-array with buffer pointing at this->_buffer + certain offset
template<typename T>
NDArray<T> NDArray<T>::operator()(const Intervals& idx, bool keepUnitiesInShape)  const {

	return NDArray<T>();
}
        
////////////////////////////////////////////////////////////////////////
// addition operator array + array
template<typename T>
NDArray<T> NDArray<T>::operator+(const NDArray<T>& other) const {

	return NDArray<T>();

}

////////////////////////////////////////////////////////////////////////
// addition operator array + scalar
template<typename T>
NDArray<T> NDArray<T>::operator+(const T scalar) const {

	return NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// addition operator scalar + array
// template<typename T>
// NDArray<T> operator+(const T scalar, const NDArray<T>& arr) {
//     return arr + scalar;
// }
ND4J_EXPORT NDArray<float16> operator+(const float16 scalar, const NDArray<float16>& arr) {
	return NDArray<float16>();
}
ND4J_EXPORT NDArray<float> operator+(const float scalar, const NDArray<float>& arr) {
    return NDArray<float>();
}
ND4J_EXPORT NDArray<double> operator+(const double scalar, const NDArray<double>& arr) {
	return NDArray<double>();        
}

////////////////////////////////////////////////////////////////////////
// subtraction operator scalar - array
// template<typename T>
// NDArray<T> operator-(const T scalar, const NDArray<T>& arr) {

//     NDArray<T> result(arr._shapeInfo, false, arr._workspace);
//     functions::scalar::ScalarTransform<T>::template transform<simdOps::ReverseSubtract<T>>(arr._buffer, arr._shapeInfo, result._buffer, result._shapeInfo, scalar, nullptr);

//     return result;
// }    
ND4J_EXPORT NDArray<float16> operator-(const float16 scalar, const NDArray<float16>& arr) {
	return NDArray<float16>();
}        
ND4J_EXPORT NDArray<float> operator-(const float scalar, const NDArray<float>& arr) {
	return NDArray<float>();
}        
ND4J_EXPORT NDArray<double> operator-(const double scalar, const NDArray<double>& arr) {
	return NDArray<double>();
}    
    
////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::operator+=(const NDArray<T>& other) {    

}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::operator-=(const NDArray<T>& other) {    

}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::operator+=(const T other) {

}

////////////////////////////////////////////////////////////////////////    
template<typename T>
void NDArray<T>::operator-=(const T other) {  

}

////////////////////////////////////////////////////////////////////////
// subtraction operator array - array
template<typename T>
NDArray<T> NDArray<T>::operator-(const NDArray<T>& other) const {

	return NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// subtraction operator array - scalar
template<typename T>
NDArray<T> NDArray<T>::operator-(const T& scalar) const {

	return NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// negative operator, it makes all array elements = -elements
template<typename T>
NDArray<T> NDArray<T>::operator-() const {

	return NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// multiplication operator array*array
template<typename T>
NDArray<T> NDArray<T>::operator*(const NDArray<T>& other) const {
        
	return NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// multiplication operator array*scalar
template<typename T>
NDArray<T> NDArray<T>::operator*(const T scalar) const {
        
	return NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
// multiplication operator array1 *= array2
template<typename T>
void NDArray<T>::operator*=(const NDArray<T>& other) {    

}

////////////////////////////////////////////////////////////////////////
// multiplication operator array*scalar
template<typename T>
void NDArray<T>::operator*=(const T scalar) {

}


////////////////////////////////////////////////////////////////////////
// division operator array/array
template<typename T>
NDArray<T> NDArray<T>::operator/(const NDArray<T>& other) const {
	NDArray<T> cs;
	return cs;
}

////////////////////////////////////////////////////////////////////////
// division operator array / scalar
template<typename T>
NDArray<T> NDArray<T>::operator/(const T scalar) const {
	NDArray<T> cs;
	return cs;
}

////////////////////////////////////////////////////////////////////////
// division operator array1 /= array2
template<typename T>
void NDArray<T>::operator/=(const NDArray<T>& other) {

}

////////////////////////////////////////////////////////////////////////
// division operator array /= scalar
template<typename T>
void NDArray<T>::operator/=(const T scalar) {
        	
}

////////////////////////////////////////////////////////////////////////
// mathematical multiplication of two arrays
template<typename T>
NDArray<T> mmul(const NDArray<T>& left, const NDArray<T>& right) {

	return NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
DataType NDArray<T>::dataType() const {
	
	return DataType_INHERIT;
}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::assign(const NDArray<T>& other, const Intervals& idx) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::setIdentity() {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::swapUnsafe(NDArray<T>& other) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
NDArray<T>* NDArray<T>::diagonal(const char type) const {        
	
	return new NDArray<T>();
}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::setValueInDiagMatrix(const T& value, const int diag, const char direction) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::streamline(char o) {

}


////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::tileToShape(const std::vector<Nd4jLong>& shape, NDArray<T>* target) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
void NDArray<T>::tileToShape(const std::initializer_list<Nd4jLong>& shape, NDArray<T>* target) {

}

////////////////////////////////////////////////////////////////////////
template<typename T>
T NDArray<T>::getTrace() const {
    
    return T();
}
////////////////////////////////////////////////////////////////////////
// default destructor
template<typename T>
NDArray<T>::~NDArray() noexcept {

}

	template class ND4J_EXPORT NDArray<float>;
	template class ND4J_EXPORT NDArray<float16>;
	template class ND4J_EXPORT NDArray<double>;


	template NDArray<float>* NDArray<float>::asT<float>();
	template NDArray<float16>* NDArray<float>::asT<float16>();
	template NDArray<double>* NDArray<float>::asT<double>();

	template NDArray<float>* NDArray<float16>::asT<float>();
	template NDArray<float16>* NDArray<float16>::asT<float16>();
	template NDArray<double>* NDArray<float16>::asT<double>();

	template NDArray<float>* NDArray<double>::asT<float>();
	template NDArray<float16>* NDArray<double>::asT<float16>();
	template NDArray<double>* NDArray<double>::asT<double>();


#ifndef __CLION_IDE__
#include "../cpu/NDArray.macro"
#endif
}