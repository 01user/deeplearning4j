//
//  @author raver119@gmail.com
//

#include <array/LaunchContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <helpers/logger.h>

//static hipStream_t defaultStream;
namespace nd4j {
    LaunchContext::LaunchContext() {
        // default constructor, just to make clang/ranlib happy
        _stream = new hipStream_t;
        hipStreamCreate(_stream);
    }

    LaunchContext::~LaunchContext() {
        // default constructor, just to make clang/ranlib happy
        hipStreamDestroy(*_stream);
        delete _stream;
    }

    void* LaunchContext::reductionPointer() {
        return reinterpret_cast<void *>(_reductionBuffer);
    }

    void* LaunchContext::allocationBuffer() {
        return reinterpret_cast<void *>(_allocationBuffer);
    }

    hipStream_t* LaunchContext::stream() {
        return _stream;
    }

    LaunchContext* LaunchContext::setCudaStream(hipStream_t *stream) {
        _stream = stream;
        return this;
    }
    LaunchContext* LaunchContext::defaultContext() {
        /**
         * defaultContext should be platform-specific
         */
        return nullptr;
    }

}