//
//  @author raver119@gmail.co
//

#include <array/LaunchContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace nd4j {
    void* LaunchContext::reductionPointer() {
        return reinterpret_cast<void *>(_reductionBuffer);
    }

    hipStream_t* LaunchContext::stream() {
        return _stream;
    }

    LaunchContext* LaunchContext::setStream(hipStream_t *stream) {
        _stream = stream;
        return this;
    }
}