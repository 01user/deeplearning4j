//
//  @author raver119@gmail.com
//

#include <array/LaunchContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace nd4j {
    void* LaunchContext::reductionPointer() {
        return reinterpret_cast<void *>(_reductionBuffer);
    }

    void* LaunchContext::allocationBuffer() {
        return reinterpret_cast<void *>(_allocationBuffer);
    }

    hipStream_t* LaunchContext::stream() {
        return _stream;
    }

    LaunchContext* LaunchContext::setCudaStream(hipStream_t *stream) {
        _stream = stream;
        return this;
    }
}