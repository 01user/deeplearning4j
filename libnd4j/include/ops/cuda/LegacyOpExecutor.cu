//
//
//

#include <ops/LegacyOpExecutor.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <loops/scalar.h>
#include <helpers/TadMigrationHelper.h>
#include <helpers/VectorMigrationHelper.h>


namespace nd4j {

    template <typename T>
    void LegacyOpExecutor<T>::execScalarOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, T scalar, std::vector<T> &extras) {
        Nd4jPointer extraPtrs[] = {nullptr, reinterpret_cast<Nd4jPointer>(ctx.stream()), nullptr, nullptr};
        dim3 launchDims = {128, 1024, 2048};

        functions::scalar::ScalarTransform<T>::executeCudaShaped(launchDims, extraPtrs, opNum, x->specialBuffer(), x->specialShapeInfo(), z->specialBuffer(),  z->specialShapeInfo(), scalar, extras.data());
        hipStreamSynchronize(*ctx.stream());
    }

    template <typename T>
    void LegacyOpExecutor<T>::execReduceOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, std::vector<int> &axis, std::vector<T> &extras) {
        shape::TAD tad(x->getShapeInfo(), axis.data(), static_cast<int>(axis.size()));
        tad.createTadOnlyShapeInfo();
        tad.createOffsets();

        TadMigrationHelper helper(tad);

        //NativeOpExcutioner<T>::execReduce(opNum, x->buffer(), x->shapeInfo(), extras.data(), z->buffer(), z->shapeInfo(), axis.data(), static_cast<int>(axis.size()), tad.tadOnlyShapeInfo, tad.tadOffsets);

        dim3 launchDims = {128, 1024, 8192};

        VectorMigrationHelper<int> _axis(axis);
        VectorMigrationHelper<T> _extras(extras);

        functions::reduce::ReduceFunction<T>::execReduceXD(launchDims, ctx.stream(), opNum, x->rankOf(), x->specialBuffer(), x->specialShapeInfo(), _extras.data(), z->specialBuffer(), z->specialShapeInfo(), _axis.data(), axis.size(), reinterpret_cast<T *>(ctx.reductionPointer()), helper.tadShapeInfo(), helper.tadOffsets());

        hipStreamSynchronize(*ctx.stream());
    }


    template class ND4J_EXPORT LegacyOpExecutor<float>;
    template class ND4J_EXPORT LegacyOpExecutor<float16>;
    template class ND4J_EXPORT LegacyOpExecutor<double>;
}