//
//
//

#include <ops/LegacyOpExecutor.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <loops/scalar.h>
#include <loops/transform.h>
#include <helpers/TadMigrationHelper.h>
#include <helpers/VectorMigrationHelper.h>


namespace nd4j {

    template <typename T>
    void LegacyOpExecutor<T>::execScalarOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, T scalar, std::vector<T> &extras) {
        Nd4jPointer extraPtrs[] = {nullptr, reinterpret_cast<Nd4jPointer>(ctx.stream()), nullptr, nullptr};
        dim3 launchDims = {128, 1024, 2048};

        functions::scalar::ScalarTransform<T>::executeCudaShaped(launchDims, extraPtrs, opNum, x->specialBuffer(), x->specialShapeInfo(), z->specialBuffer(),  z->specialShapeInfo(), scalar, extras.data());
        hipStreamSynchronize(*ctx.stream());
    }

    template <typename T>
    void LegacyOpExecutor<T>::execSummaryStatsScalarOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, std::vector<T> &extras, bool biasCorrected) {
//        T res = NativeOpExcutioner<T>::execSummaryStatsScalar(opNum, x->getBuffer(), x->getShapeInfo(), extras.data(),  biasCorrected);
//        z->putScalar(0, res);

        Nd4jPointer extraPtrs[] = {nullptr, reinterpret_cast<Nd4jPointer>(ctx.stream()), nullptr, nullptr};
        dim3 launchDims = {128, 1024, 2048};
//dim3& launchDims, Nd4jPointer *extraPointers, int opNum, double *x, Nd4jLong *xShapeInfo, double *extraParams, bool biasCorrected
        (*z)(0) = functions::summarystats::SummaryStatsReduce<T>::execSummaryStatsReduceScalar(launchDims, extraPtrs, opNum, x->specialBuffer(), x->specialShapeInfo(), extras.data(), biasCorrected);
        hipStreamSynchronize(*ctx.stream());

    }

    template <typename T>
    void LegacyOpExecutor<T>::execSummaryStatsOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, std::vector<int> &axis, std::vector<T> &extras, bool biasCorrected) {
        Nd4jPointer extraPtrs[] = {nullptr, reinterpret_cast<Nd4jPointer>(ctx.stream()), nullptr, nullptr};
        dim3 launchDims = {128, 1024, 2048};
//dim3& launchDims, Nd4jPointer *extraPointers, int opNum, double *x, Nd4jLong *xShapeInfo, double *extraParams, bool biasCorrected
        functions::summarystats::SummaryStatsReduce<T>::execSummaryStatsReduce(launchDims, extraPtrs, opNum, x->specialBuffer(), x->specialShapeInfo(), extras.data(), z->specialBuffer(),  z->specialShapeInfo(), biasCorrected);

//        _CUDA_H void SummaryStatsReduce<float>::execSummaryStatsReduce(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, float *x, Nd4jLong *xShapeInfo, float *extraParams, float *result, Nd4jLong *resultShapeInfo,bool biasCorrected) {
          
//        NativeOpExcutioner<T>::execSummaryStats(opNum, x->getBuffer(), x->getShapeInfo(), extras.data(), z->getBuffer(), z->getShapeInfo(), axis.data(), static_cast<int>(axis.size()), biasCorrected);
        hipStreamSynchronize(*ctx.stream());
    }

    template <typename T>
    void LegacyOpExecutor<T>::execTransformOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, std::vector<T> &extras) {
        Nd4jPointer extraPtrs[] = {nullptr, reinterpret_cast<Nd4jPointer>(ctx.stream()), nullptr, nullptr};
        dim3 launchDims = {128, 1024, 2048};

//        NativeOpExcutioner<T>::execTransform(opNum, x->buffer(), x->shapeInfo(), z->getBuffer(), z->getShapeInfo(), extras.data(), nullptr, nullptr);
//	executeTransformShaped(dim3 launchDims, hipStream_t *stream, int opNum, T *x, Nd4jLong *xShape, int xRank, T *extraParams, T *z, Nd4jLong *zShape, int zRank, int *allocationPointer, T *reductionPointer,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets);

        functions::transform::Transform<T>::executeTransformShaped(launchDims, ctx.stream(), opNum, x->specialBuffer(), x->specialShapeInfo(), x->rankOf(), extras.data(), z->specialBuffer(),  z->specialShapeInfo(), z->rankOf(), nullptr, nullptr, nullptr, nullptr);
        hipStreamSynchronize(*ctx.stream());

    }

    template <typename T>
    void LegacyOpExecutor<T>::execReduceScalarOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, std::vector<T> &extras) {
//        T res = NativeOpExcutioner<T>::execReduceScalar(opNum, x->getBuffer(), x->getShapeInfo(), extras.data());
//        z->putScalar(0, res);
    }

    template <typename T>
    void LegacyOpExecutor<T>::execReduceOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, std::vector<int> &axis, std::vector<T> &extras) {
        shape::TAD tad(x->getShapeInfo(), axis.data(), static_cast<int>(axis.size()));
        tad.createTadOnlyShapeInfo();
        tad.createOffsets();

        TadMigrationHelper helper(tad);

        //NativeOpExcutioner<T>::execReduce(opNum, x->buffer(), x->shapeInfo(), extras.data(), z->buffer(), z->shapeInfo(), axis.data(), static_cast<int>(axis.size()), tad.tadOnlyShapeInfo, tad.tadOffsets);

        dim3 launchDims = {128, 1024, 8192};

        VectorMigrationHelper<int> _axis(axis);
        VectorMigrationHelper<T> _extras(extras);

        functions::reduce::ReduceFunction<T>::execReduceXD(launchDims, ctx.stream(), opNum, x->rankOf(), x->specialBuffer(), x->specialShapeInfo(), _extras.data(), z->specialBuffer(), z->specialShapeInfo(), _axis.data(), axis.size(), reinterpret_cast<T *>(ctx.reductionPointer()), helper.tadShapeInfo(), helper.tadOffsets());

        hipStreamSynchronize(*ctx.stream());
    }

    template <typename T>
    void LegacyOpExecutor<T>::execBroadcastOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *y, NDArray<T> *z, std::vector<int> &axis) {
        // only skeleton
    }

    template <typename T>
    void LegacyOpExecutor<T>::execIndexReduceScalarOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, std::vector<T> &extras) {
//        T res = NativeOpExcutioner<T>::execIndexReduceScalar(opNum, x->getBuffer(), x->getShapeInfo(), extras.data());
//        z->putScalar(0, res);
    }

    template <typename T>
    void LegacyOpExecutor<T>::execIndexReduceOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *z, std::vector<int> &axis, std::vector<T> &extras) {

    }

    template <typename T>
    void LegacyOpExecutor<T>::execPairwiseOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *y, NDArray<T> *z, std::vector<T> &extras) {
//        NativeOpExcutioner<T>::execPairwiseTransform(opNum, x->getBuffer(), x->getShapeInfo(), y->getBuffer(), y->getShapeInfo(), z->getBuffer(), z->getShapeInfo(), extras.data());
    }

    template <typename T>
    void LegacyOpExecutor<T>::execReduce3ScalarOp(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *y, NDArray<T> *z, std::vector<T> &extras) {
//        T scalar = NativeOpExcutioner<T>::execReduce3Scalar(opNum, x->buffer(), x->shapeInfo(), extras.data(), y->buffer(), y->shapeInfo());
//        z->putScalar(0, scalar);
    }

    template <typename T>
    void LegacyOpExecutor<T>::execReduce3Op(nd4j::LaunchContext &ctx, int opNum, NDArray<T> *x, NDArray<T> *y, NDArray<T> *z, std::vector<int> &axis, std::vector<T> &extras) {
//        NativeOpExcutioner<T>::execReduce3(opNum, x->buffer(), x->shapeInfo(), extras.data(), y->buffer(), y->shapeInfo(), z->buffer(), z->shapeInfo(), axis.data(), static_cast<int>(axis.size()));
    }

    template class ND4J_EXPORT LegacyOpExecutor<float>;
    template class ND4J_EXPORT LegacyOpExecutor<float16>;
    template class ND4J_EXPORT LegacyOpExecutor<double>;
}