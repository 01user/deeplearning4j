
#include <hip/hip_runtime.h>
extern "C"
__global__ void div_strided_double(int n, double *dx, double *dy) {
       int  dxIdx = blockDim.x * blockIdx.x + threadIdx.x;
          int  incx = blockDim.x * gridDim.x;
          int incy = blockDim.y * gridDim.y;
          int dyIdx = blockDim.y * gridDim.y + threadIdx.y;

      for (int c = 0, xi = dxIdx, yi = dyIdx; c < n; c++, xi += incx, yi += incy) {
                          dy[yi] /= dx[xi];
       }
 }


