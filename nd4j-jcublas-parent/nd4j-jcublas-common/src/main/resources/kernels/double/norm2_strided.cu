#include "hip/hip_runtime.h"
#include <reduce.h>


__device__ double update(double old,double opOutput,double *extraParams) {
       return opOutput + old;
 }


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
*/
__device__ double op(double d1,double d2,double *extraParams) {
      return op(d1,extraParams);
}

__device__ double op(double d1,double *extraParams) {
      return pow(d1,2);
}



__device__ double postProcess(double reduction,int n,int xOffset,double *dx,int incx,double *extraParams,double *result) {
             return sqrt(reduction);
}

extern "C"
__global__ void norm2_strided_double(int n, int xOffset,double *dx,int incx,double *extraParams,double *result) {
             transform(n,xOffset,dx,incx,extraParams,result);
}


