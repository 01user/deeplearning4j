#include "hip/hip_runtime.h"
#include "transform.h"


__device__ float op(float d1,float *params) {
       float min = params[0];
       float max = params[1];
       if(origin >= min && origin <= max)
                return d1;
        if(min == 0 && max == 1) {
               float val = 1 / (1 + expf(-origin));
               return (floorf(val * (max - min)) + min);
         }

         float ret =  (floorf(origin * (max - min)) + min);
         return ret;

}

extern "C"
__global__ void setrange_strided_float(int n,int idx,float *dy,int incy,float *params,float *result) {
       transform(n,idx,dy,incy,params,result);

 }
